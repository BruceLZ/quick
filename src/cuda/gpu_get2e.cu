#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *  
 *  Yipu Miao 9/15/11:  the first draft is released. And the GPUGP QM compuation can 
 *                      achieve as much as 15x faster at double precision level compared with CPU.
 */

#include "gpu.h"
#include <hip/hip_runtime.h>

#ifdef CUDA_SPDF
#include "int.h"
#endif

#include "int2.h"

/*
 Constant Memory in GPU is fast but quite limited and hard to operate, usually not allocatable and 
 readonly. So we put the following variables into constant memory:
 devSim: a gpu simluation type variable. which is to store to location of basic information about molecule and basis
 set. Note it only store the location, so it's mostly a set of pointer to GPU memory. and with some non-pointer
 value like the number of basis set. See gpu_type.h for details.
 devTrans : arrays to save the mapping index, will be elimited by hand writing unrolling code.
 Sumindex: a array to store refect how many temp variable needed in VRR. can be elimited by hand writing code.
 */
static __constant__ gpu_simulation_type devSim;
static __constant__ int devTrans[TRANSDIM*TRANSDIM*TRANSDIM];
static __constant__ int Sumindex[10]={0,0,1,4,10,20,35,56,84,120};


/*
 upload gpu simulation type to constant memory
 */
void upload_sim_to_constant(_gpu_type gpu){
    hipError_t status;
	status = hipMemcpyToSymbol(HIP_SYMBOL(devSim), &gpu->gpu_sim, sizeof(gpu_simulation_type));
	PRINTERROR(status, " hipMemcpyToSymbol, sim copy to constants failed")
}


#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#include "gpu_get2e_subs.h"

#ifdef CUDA_SPDF
#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#include "gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#include "gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#include "gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#include "gpu_get2e_subs.h"
#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4


// totTime is the timer for GPU 2e time. Only on under debug mode
#ifdef DEBUG
static float totTime;
#endif

// =======   INTERFACE SECTION ===========================
// interface to call Kernel subroutine
void getAOInt(_gpu_type gpu, QUICKULL intStart, QUICKULL intEnd, hipStream_t streamI, int streamID,  ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((getAOInt_kernel<<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>>(intStart, intEnd, aoint_buffer, streamID)));
#ifdef CUDA_SPDF
    // Part f-1
    QUICK_SAFE_CALL((getAOInt_kernel_spdf<<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>>( intStart, intEnd, aoint_buffer, streamID)));
    // Part f-2
    QUICK_SAFE_CALL((getAOInt_kernel_spdf2<<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>>( intStart, intEnd, aoint_buffer, streamID)));
    // Part f-3
    QUICK_SAFE_CALL((getAOInt_kernel_spdf3<<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>>( intStart, intEnd, aoint_buffer, streamID)));
    // Part f-4
    QUICK_SAFE_CALL((getAOInt_kernel_spdf4<<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>>( intStart, intEnd, aoint_buffer, streamID)));
#endif 
}

// interface to call Kernel subroutine
void get2e(_gpu_type gpu)
{
    // Part spd
    QUICK_SAFE_CALL((get2e_kernel<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
#ifdef CUDA_SPDF
    // Part f-1
    QUICK_SAFE_CALL((get2e_kernel_spdf<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
    // Part f-2
    QUICK_SAFE_CALL((get2e_kernel_spdf2<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
    // Part f-3
    QUICK_SAFE_CALL((get2e_kernel_spdf3<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
    // Part f-4
    QUICK_SAFE_CALL((get2e_kernel_spdf4<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
#endif 
}


// interface to call Kernel subroutine
void getAddInt(_gpu_type gpu, int bufferSize, ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((getAddInt_kernel<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>(bufferSize, aoint_buffer)));
}



// =======   KERNEL SECTION ===========================
__global__ void getAddInt_kernel(int bufferSize, ERI_entry* aoint_buffer){
    unsigned int offside = blockIdx.x*blockDim.x+threadIdx.x;
    int totalThreads = blockDim.x*gridDim.x;
    int const batchSize = 20;
    ERI_entry a[batchSize];
    int j = 0;
    
    QUICKULL myInt = (QUICKULL) (bufferSize) / totalThreads;
    if ((bufferSize - myInt*totalThreads)> offside) myInt++;
    
    for (QUICKULL i = 1; i<=myInt; i++) {
        
        QUICKULL currentInt = totalThreads * (i-1) + offside;
        a[j] = aoint_buffer[currentInt];
        j++;
        if (j == batchSize || i == myInt) {
            
            for (int k = 0; k<j; k++) {
                int III = a[k].IJ / devSim.nbasis + 1;
                int JJJ = a[k].IJ % devSim.nbasis + 1;
                int KKK = a[k].KL / devSim.nbasis + 1;
                int LLL = a[k].KL % devSim.nbasis + 1;
                
                if (III <= devSim.nbasis && III >= 1 && JJJ <= devSim.nbasis && JJJ >= 1 && KKK <= devSim.nbasis && KKK >= 1 && LLL <= devSim.nbasis && LLL >= 1){
                    QUICKDouble hybrid_coeff = 0.0;
                    if (devSim.method == HF){
                        hybrid_coeff = 1.0;
                    }else if (devSim.method == B3LYP){
                        hybrid_coeff = 0.2;
                    }else if (devSim.method == DFT){
                        hybrid_coeff = 0.0;
                    }
                    
                    addint(devSim.oULL, a[k].value, III, JJJ, KKK, LLL, hybrid_coeff, devSim.dense, devSim.nbasis);
                }
            }
            j = 0;
        }
        
    }
    
}


__device__ __forceinline__ void addint(QUICKULL* oULL, QUICKDouble Y, int III, int JJJ, int KKK, int LLL,QUICKDouble hybrid_coeff,  QUICKDouble* dense, int nbasis){
    
    QUICKDouble DENSEKI = (QUICKDouble) LOC2(dense, KKK-1, III-1, nbasis, nbasis);
    QUICKDouble DENSEKJ = (QUICKDouble) LOC2(dense, KKK-1, JJJ-1, nbasis, nbasis);
    QUICKDouble DENSELJ = (QUICKDouble) LOC2(dense, LLL-1, JJJ-1, nbasis, nbasis);
    QUICKDouble DENSELI = (QUICKDouble) LOC2(dense, LLL-1, III-1, nbasis, nbasis);
    QUICKDouble DENSELK = (QUICKDouble) LOC2(dense, LLL-1, KKK-1, nbasis, nbasis);
    QUICKDouble DENSEJI = (QUICKDouble) LOC2(dense, JJJ-1, III-1, nbasis, nbasis);
    
    
    // ATOMIC ADD VALUE 1
    QUICKDouble _tmp = 2.0;
    if (KKK==LLL) {
        _tmp = 1.0;
    }
    
    QUICKDouble val1d = _tmp*DENSELK*Y;
    QUICKULL val1 = (QUICKULL) (fabs(val1d*OSCALE) + (QUICKDouble)0.5);
    if ( val1d < (QUICKDouble)0.0) val1 = 0ull - val1;
    QUICKADD(LOC2(oULL, JJJ-1, III-1, nbasis, nbasis), val1);
    
    
    // ATOMIC ADD VALUE 2
    if ((LLL != JJJ) || (III!=KKK)) {
        _tmp = 2.0;
        if (III==JJJ) {
            _tmp = 1.0;
        }
        
        QUICKDouble val2d = _tmp*DENSEJI*Y;
        QUICKULL val2 = (QUICKULL) (fabs(val2d*OSCALE) + (QUICKDouble)0.5);
        if ( val2d < (QUICKDouble)0.0) val2 = 0ull - val2;
        QUICKADD(LOC2(oULL, LLL-1, KKK-1, nbasis, nbasis), val2);
    }
    
    
    // ATOMIC ADD VALUE 3
    QUICKDouble val3d = hybrid_coeff*0.5*DENSELJ*Y;
    
    QUICKULL val3 = (QUICKULL) (fabs(val3d*OSCALE) + (QUICKDouble)0.5);
    if (((III == KKK) && (III <  JJJ) && (JJJ < LLL))) {
        val3 = (QUICKULL) (fabs(2*val3d*OSCALE) + (QUICKDouble)0.5);
    }
    if ( DENSELJ*Y < (QUICKDouble)0.0) val3 = 0ull - val3;
    QUICKADD(LOC2(oULL, KKK-1, III-1, nbasis, nbasis), 0ull-val3);
    
    // ATOMIC ADD VALUE 4
    if (KKK != LLL) {
        QUICKDouble val4d = hybrid_coeff*0.5*DENSEKJ*Y;
        
        QUICKULL val4 = (QUICKULL) (fabs(val4d*OSCALE) + (QUICKDouble)0.5);
        if ( val4d < (QUICKDouble)0.0) val4 = 0ull - val4;
        QUICKADD(LOC2(oULL, LLL-1, III-1, nbasis, nbasis), 0ull-val4);
    }
    
    
    
    // ATOMIC ADD VALUE 5
    QUICKDouble val5d = hybrid_coeff*0.5*DENSELI*Y;
    
    QUICKULL val5 = (QUICKULL) (fabs(val5d*OSCALE) + (QUICKDouble)0.5);
    if ( val5d < (QUICKDouble)0.0) val5 = 0ull - val5;
    
    if ((III != JJJ && III<KKK) || ((III == JJJ) && (III == KKK) && (III < LLL)) || ((III == KKK) && (III <  JJJ) && (JJJ < LLL))) {
        QUICKADD(LOC2(oULL, MAX(JJJ,KKK)-1, MIN(JJJ,KKK)-1, nbasis, nbasis), 0ull-val5);
    }
    
    
    // ATOMIC ADD VALUE 5 - 2
    if ( III != JJJ && JJJ == KKK) {
        QUICKADD(LOC2(oULL, JJJ-1, KKK-1, nbasis, nbasis), 0ull-val5);
    }
    
    // ATOMIC ADD VALUE 6
    if (III != JJJ) {
        if (KKK != LLL) {
            QUICKDouble val6d = hybrid_coeff*0.5*DENSEKI*Y;
            QUICKULL val6 = (QUICKULL) (fabs(val6d*OSCALE) + (QUICKDouble)0.5);
            if ( val6d < (QUICKDouble)0.0) val6 = 0ull - val6;
            
            QUICKADD(LOC2(oULL, MAX(JJJ,LLL)-1, MIN(JJJ,LLL)-1, devSim.nbasis, devSim.nbasis), 0ull-val6);
            
            // ATOMIC ADD VALUE 6 - 2
            if (JJJ == LLL && III!= KKK) {
                QUICKADD(LOC2(oULL, LLL-1, JJJ-1, nbasis, nbasis), 0ull-val6);
            }
        }
    }
}

__device__ __forceinline__ void FmT(int MaxM, QUICKDouble X, QUICKDouble* YVerticalTemp)
{
    
    const QUICKDouble PIE4 = (QUICKDouble) PI/4.0 ;
    
    const QUICKDouble XINV = (QUICKDouble) 1.0 /X;
    const QUICKDouble E = (QUICKDouble) exp(-X);
    QUICKDouble WW1;
    
    if (X > 5.0 ) {
        if (X>15.0 ) {
            if (X>33.0 ) {
                WW1 = sqrt(PIE4 * XINV);
            }else {
                WW1 = (( 1.9623264149430E-01 *XINV-4.9695241464490E-01 )*XINV - \
                       6.0156581186481E-05 )*E + sqrt(PIE4*XINV);
            }
        }else if (X>10.0 ) {
            WW1 = (((-1.8784686463512E-01 *XINV+2.2991849164985E-01 )*XINV - \
                    4.9893752514047E-01 )*XINV-2.1916512131607E-05 )*E + sqrt(PIE4*XINV);
        }else {
            WW1 = (((((( 4.6897511375022E-01  *XINV-6.9955602298985E-01 )*XINV + \
                       5.3689283271887E-01 )*XINV-3.2883030418398E-01 )*XINV + \
                     2.4645596956002E-01 )*XINV-4.9984072848436E-01 )*XINV - \
                   3.1501078774085E-06 )*E + sqrt(PIE4*XINV);
        }
    }else if (X >1.0 ) {
        if (X>3.0 ) {
            QUICKDouble Y = (QUICKDouble) X - 4.0 ;
            QUICKDouble F1 = ((((((((((-2.62453564772299E-11 *Y+3.24031041623823E-10  )*Y- \
                                      3.614965656163E-09 )*Y+3.760256799971E-08 )*Y- \
                                    3.553558319675E-07 )*Y+3.022556449731E-06 )*Y- \
                                  2.290098979647E-05 )*Y+1.526537461148E-04 )*Y- \
                                8.81947375894379E-04 )*Y+4.33207949514611E-03 )*Y- \
                              1.75257821619926E-02 )*Y+5.28406320615584E-02 ;
            WW1 = (X+X)*F1+E;
        }else {
            QUICKDouble Y = (QUICKDouble) X - 2.0 ;
            QUICKDouble F1 = ((((((((((-1.61702782425558E-10 *Y+1.96215250865776E-09  )*Y- \
                                      2.14234468198419E-08  )*Y+2.17216556336318E-07  )*Y- \
                                    1.98850171329371E-06  )*Y+1.62429321438911E-05  )*Y- \
                                  1.16740298039895E-04  )*Y+7.24888732052332E-04  )*Y- \
                                3.79490003707156E-03  )*Y+1.61723488664661E-02  )*Y- \
                              5.29428148329736E-02  )*Y+1.15702180856167E-01 ;
            WW1 = (X+X)*F1+E;
        }
        
    }else if (X > 3.0E-7 ) {
        QUICKDouble F1 =(((((((( -8.36313918003957E-08 *X+1.21222603512827E-06  )*X- \
                               1.15662609053481E-05  )*X+9.25197374512647E-05  )*X- \
                             6.40994113129432E-04  )*X+3.78787044215009E-03  )*X- \
                           1.85185172458485E-02  )*X+7.14285713298222E-02  )*X- \
                         1.99999999997023E-01  )*X+3.33333333333318E-01 ;
        WW1 = (X+X)*F1+E;
    }else {
        WW1 = (1.0 -X)/(QUICKDouble)(2.0 * MaxM+1);
    }
    if (X > 3.0E-7 ) {
        LOC3(YVerticalTemp, 0, 0, 0, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = 1; m<= MaxM; m++) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (((2*m-1)*LOC3(YVerticalTemp, 0, 0, m-1, VDIM1, VDIM2, VDIM3))- E)*0.5*XINV;
        }
    }else {
        LOC3(YVerticalTemp, 0, 0, MaxM, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = MaxM-1; m >=0; m--) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (2.0 * X * LOC3(YVerticalTemp, 0, 0, m+1, VDIM1, VDIM2, VDIM3) + E) / (QUICKDouble)(m*2+1);
        }
    }
    return;
}

/*
 sqr for double precision. there no internal function to do that in fast-math-lib of CUDA
 */
__device__ __forceinline__ QUICKDouble quick_dsqr(QUICKDouble a)
{
    return a*a;
}


__device__ __forceinline__ QUICKDouble hrrwhole(int I, int J, int K, int L, \
                     int III, int JJJ, int KKK, int LLL, int IJKLTYPE, QUICKDouble* store, \
                     QUICKDouble RAx,QUICKDouble RAy,QUICKDouble RAz, \
                     QUICKDouble RBx,QUICKDouble RBy,QUICKDouble RBz, \
                     QUICKDouble RCx,QUICKDouble RCy,QUICKDouble RCz, \
                     QUICKDouble RDx,QUICKDouble RDy,QUICKDouble RDz)
{
    QUICKDouble Y;
#ifdef CUDA_SP
    int NAx = LOC2(devSim.KLMN,0,III-1,3,devSim.nbasis);
    int NAy = LOC2(devSim.KLMN,1,III-1,3,devSim.nbasis);
    int NAz = LOC2(devSim.KLMN,2,III-1,3,devSim.nbasis);
    
    int NBx = LOC2(devSim.KLMN,0,JJJ-1,3,devSim.nbasis);
    int NBy = LOC2(devSim.KLMN,1,JJJ-1,3,devSim.nbasis);
    int NBz = LOC2(devSim.KLMN,2,JJJ-1,3,devSim.nbasis);
    
    int NCx = LOC2(devSim.KLMN,0,KKK-1,3,devSim.nbasis);
    int NCy = LOC2(devSim.KLMN,1,KKK-1,3,devSim.nbasis);
    int NCz = LOC2(devSim.KLMN,2,KKK-1,3,devSim.nbasis);
    
    int NDx = LOC2(devSim.KLMN,0,LLL-1,3,devSim.nbasis);
    int NDy = LOC2(devSim.KLMN,1,LLL-1,3,devSim.nbasis);
    int NDz = LOC2(devSim.KLMN,2,LLL-1,3,devSim.nbasis);
    
    
    int MA = LOC3(devTrans, NAx, NAy, NAz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MB = LOC3(devTrans, NBx, NBy, NBz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MC = LOC3(devTrans, NCx, NCy, NCz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MD = LOC3(devTrans, NDx, NDy, NDz, TRANSDIM, TRANSDIM, TRANSDIM);
    
    switch (IJKLTYPE) {
        case 0:
        case 10:
        case 1000:
        case 1010:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            break;
        }
        case 2000:
        case 20:
        case 2010:
        case 1020:
        case 2020:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM) * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
            break;
        }
        case 100:
        {
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 110:
        {
            
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 101:
        {
            QUICKDouble Y1,Y2;
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 111:
        {
            QUICKDouble Y1,Y2;
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1100:
        {
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1110:
        {   
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1101:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1111:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            
            break;
        }
        case 1:
        {
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 11:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1001:
        {   
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
        }
        case 1011:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        default:
            break;
    }
#else
    
    int angularL[12], angularR[12];
    QUICKDouble coefAngularL[12], coefAngularR[12];
    Y = (QUICKDouble) 0.0;
    
    int numAngularL = lefthrr(RAx, RAy, RAz, RBx, RBy, RBz, 
                              LOC2(devSim.KLMN,0,III-1,3,devSim.nbasis), LOC2(devSim.KLMN,1,III-1,3,devSim.nbasis), LOC2(devSim.KLMN,2,III-1,3,devSim.nbasis),
                              LOC2(devSim.KLMN,0,JJJ-1,3,devSim.nbasis), LOC2(devSim.KLMN,1,JJJ-1,3,devSim.nbasis), LOC2(devSim.KLMN,2,JJJ-1,3,devSim.nbasis),
                              J, coefAngularL, angularL);
    int numAngularR = lefthrr(RCx, RCy, RCz, RDx, RDy, RDz,
                              LOC2(devSim.KLMN,0,KKK-1,3,devSim.nbasis), LOC2(devSim.KLMN,1,KKK-1,3,devSim.nbasis), LOC2(devSim.KLMN,2,KKK-1,3,devSim.nbasis),
                              LOC2(devSim.KLMN,0,LLL-1,3,devSim.nbasis), LOC2(devSim.KLMN,1,LLL-1,3,devSim.nbasis), LOC2(devSim.KLMN,2,LLL-1,3,devSim.nbasis),
                              L, coefAngularR, angularR);
    
    for (int i = 0; i<numAngularL; i++) {
        for (int j = 0; j<numAngularR; j++) {
            Y += coefAngularL[i] * coefAngularR[j] * LOC2(store, angularL[i]-1, angularR[j]-1 , STOREDIM, STOREDIM);
        }
    }
    //if (K == 2 && L == 3) Y = coefAngularL[0] * coefAngularR[2] * LOC2(store, angularL[0]-1, angularR[2]-1 , STOREDIM, STOREDIM);
    
    Y = Y * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
#endif
    return Y;
}  


#ifndef CUDA_SP
__device__ __forceinline__ int lefthrr(QUICKDouble RAx, QUICKDouble RAy, QUICKDouble RAz, 
            QUICKDouble RBx, QUICKDouble RBy, QUICKDouble RBz,
            int KLMNAx, int KLMNAy, int KLMNAz,
            int KLMNBx, int KLMNBy, int KLMNBz,
            int IJTYPE,QUICKDouble* coefAngularL, int* angularL)
{           
    int numAngularL;
    switch (IJTYPE) {
        
        case 0:
        {
            numAngularL = 1;
            coefAngularL[0] = 1.0;
            angularL[0] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            break;
        }
        case 1:
        {
            coefAngularL[0] = 1.0;
            numAngularL = 2;
            angularL[0] = (int) LOC3(devTrans, KLMNAx + KLMNBx, KLMNAy + KLMNBy, KLMNAz + KLMNBz, TRANSDIM, TRANSDIM, TRANSDIM);
            angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (KLMNBx != 0) {
                coefAngularL[1] = RAx-RBx;
            }else if(KLMNBy !=0 ){
                coefAngularL[1] = RAy-RBy;
            }else if (KLMNBz != 0) {
                coefAngularL[1] = RAz-RBz;
            }
            break;
        }
        case 2:
        {
            coefAngularL[0] = 1.0;
            angularL[0] = (int) LOC3(devTrans, KLMNAx + KLMNBx, KLMNAy + KLMNBy, KLMNAz + KLMNBz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (KLMNBx == 2) {
                numAngularL = 3;
                QUICKDouble tmp = RAx - RBx;
                coefAngularL[1] = 2 * tmp;
                angularL[1] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                coefAngularL[2]= tmp * tmp;
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if(KLMNBy == 2) {
                numAngularL = 3;
                QUICKDouble tmp = RAy - RBy;
                coefAngularL[1] = 2 * tmp;
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                coefAngularL[2]= tmp * tmp;
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBz == 2 ){
                numAngularL = 3;
                QUICKDouble tmp = RAz - RBz;
                coefAngularL[1] = 2 * tmp;
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                coefAngularL[2]= tmp * tmp;
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBy == 1){
                numAngularL = 4;
                coefAngularL[1] = RAx - RBx;
                coefAngularL[2] = RAy - RBy;
                coefAngularL[3] = (RAx - RBx) * (RAy - RBy);
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                
            }else if (KLMNBx == 1 && KLMNBz == 1) {
                numAngularL = 4;
                coefAngularL[1] = RAx - RBx;
                coefAngularL[2] = RAz - RBz;
                coefAngularL[3] = (RAx - RBx) * (RAz - RBz);
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 1 && KLMNBz == 1) {
                numAngularL = 4;
                coefAngularL[1] = RAy - RBy;
                coefAngularL[2] = RAz - RBz;
                coefAngularL[3] = (RAy - RBy) * (RAz - RBz);
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }
            break;
        }
        case 3:
        {
            coefAngularL[0] = 1.0;
            angularL[0] = (int) LOC3(devTrans, KLMNAx + KLMNBx, KLMNAy + KLMNBy, KLMNAz + KLMNBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (KLMNBx == 3) {
                numAngularL = 4;
                QUICKDouble tmp = RAx - RBx;
                
                coefAngularL[1] = 3 * tmp;
                coefAngularL[2] = 3 * tmp * tmp;
                coefAngularL[3] = tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 3) {
                numAngularL = 4;
                QUICKDouble tmp = RAy - RBy;
                coefAngularL[1] = 3 * tmp;
                coefAngularL[2] = 3 * tmp * tmp;
                coefAngularL[3] = tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBz == 3) {
                numAngularL = 4;
                
                QUICKDouble tmp = RAz - RBz;
                coefAngularL[1] = 3 * tmp;
                coefAngularL[2] = 3 * tmp * tmp;
                coefAngularL[3] = tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBy ==2) { // case 120
                numAngularL = 6;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAy - RBy;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);   
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBz ==2) { // case 102
                numAngularL = 6;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAz - RBz;
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);   
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 1 && KLMNBz ==2) { // case 012
                numAngularL = 6;
                QUICKDouble tmp = RAy - RBy;
                QUICKDouble tmp2 = RAz - RBz;
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);   
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy== 2 && KLMNBz == 1) { // case 021
                numAngularL = 6;
                QUICKDouble tmp = RAz - RBz;
                QUICKDouble tmp2 = RAy - RBy;
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 2 && KLMNBy == 1) { // case 210
                numAngularL = 6;
                QUICKDouble tmp = RAy - RBy;
                QUICKDouble tmp2 = RAx - RBx;
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 2 && KLMNBz ==1) { // case 201
                numAngularL = 6;
                QUICKDouble tmp = RAz - RBz;
                QUICKDouble tmp2 = RAx - RBx;
                coefAngularL[1] = tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 2 * tmp * tmp2;
                coefAngularL[4] = tmp2 * tmp2;
                coefAngularL[5] = tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBy == 1) {
                numAngularL = 8;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAy - RBy;
                QUICKDouble tmp3 = RAz - RBz;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = tmp2;
                coefAngularL[3] = tmp3;
                coefAngularL[4] = tmp * tmp2;
                coefAngularL[5] = tmp * tmp3;                
                coefAngularL[6] = tmp2 * tmp3;
                coefAngularL[7] = tmp * tmp2 * tmp3;

                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }
            break;
            
        }
        case 4:
        {
            coefAngularL[0] = 1.0;
            angularL[0] = (int) LOC3(devTrans, KLMNAx + KLMNBx, KLMNAy + KLMNBy, KLMNAz + KLMNBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (KLMNBx == 4) {
                numAngularL = 5;
                QUICKDouble tmp = RAx - RBx;
                
                coefAngularL[1] = 4 * tmp;
                coefAngularL[2] = 6 * tmp * tmp;
                coefAngularL[3] = 4 * tmp * tmp * tmp;
                coefAngularL[4] = tmp * tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+3, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 4) {
                numAngularL = 5;
                QUICKDouble tmp = RAy - RBy;
                coefAngularL[1] = 4 * tmp;
                coefAngularL[2] = 6 * tmp * tmp;
                coefAngularL[3] = 4 * tmp * tmp * tmp;
                coefAngularL[4] = tmp * tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+3, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                
            }else if (KLMNBz == 4) {
                numAngularL = 5;
                
                QUICKDouble tmp = RAz - RBz;
                coefAngularL[1] = 4 * tmp;
                coefAngularL[2] = 6 * tmp * tmp;
                coefAngularL[3] = 4 * tmp * tmp * tmp;
                coefAngularL[4] = tmp * tmp * tmp * tmp;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+3, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                
            }else if (KLMNBx == 1 && KLMNBy ==3) {
                numAngularL = 8;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAy - RBy;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = 3 * tmp2;
                coefAngularL[3] = 3 * tmp * tmp2;
                coefAngularL[4] = 3 * tmp2 * tmp2;
                coefAngularL[5] = 3 * tmp * tmp2 * tmp2;
                coefAngularL[6] = tmp2 * tmp2 * tmp2;
                coefAngularL[7] = tmp * tmp2 * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx,   KLMNAy+3, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx,   KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBz ==3) {
                numAngularL = 8;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAz - RBz;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = 3 * tmp2;
                coefAngularL[3] = 3 * tmp * tmp2;
                coefAngularL[4] = 3 * tmp2 * tmp2;
                coefAngularL[5] = 3 * tmp * tmp2 * tmp2;
                coefAngularL[6] = tmp2 * tmp2 * tmp2;
                coefAngularL[7] = tmp * tmp2 * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+3, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                
            }else if (KLMNBy == 1 && KLMNBz ==3) {
                numAngularL = 8;
                QUICKDouble tmp = RAy - RBy;
                QUICKDouble tmp2 = RAz - RBz;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = 3 * tmp2;
                coefAngularL[3] = 3 * tmp * tmp2;
                coefAngularL[4] = 3 * tmp2 * tmp2;
                coefAngularL[5] = 3 * tmp * tmp2 * tmp2;
                coefAngularL[6] = tmp2 * tmp2 * tmp2;
                coefAngularL[7] = tmp * tmp2 * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+3, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                
            }else if (KLMNBx == 2 && KLMNBy == 2) {
                numAngularL = 9;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAy - RBy;
                
                coefAngularL[1] = 2 * tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 4 * tmp * tmp2;
                coefAngularL[4] = tmp * tmp;
                coefAngularL[5] = tmp2 * tmp2;
                coefAngularL[6] = 2 * tmp * tmp2 * tmp2;
                coefAngularL[7] = 2 * tmp * tmp * tmp2;
                coefAngularL[8] = tmp * tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx,   KLMNAy+2, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 2 && KLMNBz == 2) {
                numAngularL = 9;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAz - RBz;
                
                coefAngularL[1] = 2 * tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 4 * tmp * tmp2;
                coefAngularL[4] = tmp * tmp;
                coefAngularL[5] = tmp2 * tmp2;
                coefAngularL[6] = 2 * tmp * tmp2 * tmp2;
                coefAngularL[7] = 2 * tmp * tmp * tmp2;
                coefAngularL[8] = tmp * tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx+2, KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 2 && KLMNBz == 2) {
                numAngularL = 9;
                QUICKDouble tmp = RAy - RBy;
                QUICKDouble tmp2 = RAz - RBz;
                
                coefAngularL[1] = 2 * tmp;
                coefAngularL[2] = 2 * tmp2;
                coefAngularL[3] = 4 * tmp * tmp2;
                coefAngularL[4] = tmp * tmp;
                coefAngularL[5] = tmp2 * tmp2;
                coefAngularL[6] = 2 * tmp * tmp2 * tmp2;
                coefAngularL[7] = 2 * tmp * tmp * tmp2;
                coefAngularL[8] = tmp * tmp * tmp2 * tmp2;
                
                angularL[1] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int) LOC3(devTrans, KLMNAx, KLMNAy+2,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int) LOC3(devTrans, KLMNAx, KLMNAy+1,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int) LOC3(devTrans, KLMNAx,   KLMNAy, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz, TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBy == 1) {
                numAngularL = 12;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAy - RBy;
                QUICKDouble tmp3 = RAz - RBz;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = tmp2;
                coefAngularL[3] = 2 * tmp3;
                coefAngularL[4] = tmp * tmp2;
                coefAngularL[5] = 2 * tmp * tmp3;
                coefAngularL[6] = 2 * tmp2 * tmp3;
                coefAngularL[7] = tmp3 * tmp3;
                coefAngularL[8] = 2 * tmp * tmp2 * tmp3;
                coefAngularL[9] = tmp * tmp3 * tmp3;
                coefAngularL[10] = tmp * tmp3 * tmp3;
                coefAngularL[11] = tmp * tmp2 * tmp3 * tmp3;
                
                angularL[1] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz+2, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[9] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[10] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[11] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBx == 1 && KLMNBz == 1) {
                numAngularL = 12;
                QUICKDouble tmp = RAx - RBx;
                QUICKDouble tmp2 = RAz - RBz;
                QUICKDouble tmp3 = RAy - RBy;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = tmp2;
                coefAngularL[3] = 2 * tmp3;
                coefAngularL[4] = tmp * tmp2;
                coefAngularL[5] = 2 * tmp * tmp3;
                coefAngularL[6] = 2 * tmp2 * tmp3;
                coefAngularL[7] = tmp3 * tmp3;
                coefAngularL[8] = 2 * tmp * tmp2 * tmp3;
                coefAngularL[9] = tmp * tmp3 * tmp3;
                coefAngularL[10] = tmp * tmp3 * tmp3;
                coefAngularL[11] = tmp * tmp2 * tmp3 * tmp3;
                
                angularL[1] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+2, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+2, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+2, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[9] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[10] = (int) LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[11] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
            }else if (KLMNBy == 1 && KLMNBz == 1) {
                numAngularL = 12;
                QUICKDouble tmp = RAy - RBy;
                QUICKDouble tmp2 = RAz - RBz;
                QUICKDouble tmp3 = RAx - RBx;
                
                coefAngularL[1] = tmp;
                coefAngularL[2] = tmp2;
                coefAngularL[3] = 2 * tmp3;
                coefAngularL[4] = tmp * tmp2;
                coefAngularL[5] = 2 * tmp * tmp3;
                coefAngularL[6] = 2 * tmp2 * tmp3;
                coefAngularL[7] = tmp3 * tmp3;
                coefAngularL[8] = 2 * tmp * tmp2 * tmp3;
                coefAngularL[9] = tmp * tmp3 * tmp3;
                coefAngularL[10] = tmp * tmp3 * tmp3;
                coefAngularL[11] = tmp * tmp2 * tmp3 * tmp3;
                
                angularL[1] = (int)  LOC3(devTrans, KLMNAx+2, KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[2] = (int)  LOC3(devTrans, KLMNAx+2, KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[3] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[4] = (int)  LOC3(devTrans, KLMNAx+2, KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[5] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[6] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy+1, KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[7] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[8] = (int)  LOC3(devTrans, KLMNAx+1, KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[9] = (int)  LOC3(devTrans, KLMNAx,   KLMNAy+1, KLMNAz+1, TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[10] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
                angularL[11] = (int) LOC3(devTrans, KLMNAx,   KLMNAy,   KLMNAz,   TRANSDIM, TRANSDIM, TRANSDIM);
            }
            
            
            break;

            
        }
    }
    return numAngularL;
}

#endif

void upload_para_to_const(){
    
    int trans[TRANSDIM*TRANSDIM*TRANSDIM];
    // Data to trans
    {
        LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   1;
        LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   4;
        LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  10;
        LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  20;
        LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  35;
        LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  56;
        LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) =  84;
        LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 120;
        LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   3;
        LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   6;
        LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  17;
        LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  32;
        LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  48;
        LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  67;
        LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
        LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   9;
        LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  16;
        LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  23;
        LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  42;
        LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  73;
        LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
        LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  19;
        LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  31;
        LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  43;
        LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  79;
        LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
        LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  34;
        LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  49;
        LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  74;
        LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
        LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  55;
        LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  68;
        LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 107;
        LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  83;
        LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
        LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
        LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   2;
        LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   7;
        LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  15;
        LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  28;
        LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  50;
        LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  69;
        LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
        LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   5;
        LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  11;
        LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  26;
        LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  41;
        LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  59;
        LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  87;
        LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  13;
        LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  25;
        LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  36;
        LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  60;
        LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  88;
        LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  30;
        LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  40;
        LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  61;
        LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  94;
        LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  52;
        LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  58;
        LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  89;
        LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  71;
        LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  86;
        LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
        LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   8;
        LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  14;
        LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  22;
        LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  44;
        LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  75;
        LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
        LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  12;
        LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  24;
        LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  37;
        LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  62;
        LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  90;
        LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  21;
        LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  38;
        LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  66;
        LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  99;
        LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  46;
        LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  64;
        LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  98;
        LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  77;
        LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  92;
        LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
        LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  18;
        LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  27;
        LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  45;
        LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  80;
        LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
        LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  29;
        LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  39;
        LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  63;
        LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  95;
        LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  47;
        LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  65;
        LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  97;
        LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  81;
        LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  96;
        LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
        LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  33;
        LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  51;
        LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  76;
        LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
        LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  53;
        LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  57;
        LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  91;
        LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  78;
        LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  93;
        LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;
        LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  54;
        LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  70;
        LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
        LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  72;
        LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  85;
        LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
        LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  82;
        LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
        LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
        LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    }
    // upload to trans device location
    hipError_t status;

    status = hipMemcpyToSymbol(HIP_SYMBOL(devTrans), trans, sizeof(int)*TRANSDIM*TRANSDIM*TRANSDIM);
    PRINTERROR(status, " hipMemcpyToSymbol, Trans copy to constants failed")

}

