#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *
 */

#include "gpu.h"
#include <hip/hip_runtime.h>

static 
#ifndef TEST
__constant__
#endif
 gpu_simulation_type devSim;

static
#ifndef TEST
__constant__
#endif
int devTrans[TRANSDIM*TRANSDIM*TRANSDIM];

static
#ifndef TEST
__constant__
#endif
int devMcal[MCALDIM*3];

static
#ifndef TEST
__constant__
#endif
int Sumindex[10]={0,0,1,4,10,20,35,56,84,120};

void upload_sim_to_constant(_gpu_type gpu){
    hipError_t status;
#ifdef TEST
    memcpy(&devSim, &gpu->gpu_sim, sizeof(gpu_simulation_type));
#else    
    status = hipMemcpyToSymbol(HIP_SYMBOL("devSim"), &gpu->gpu_sim, sizeof(gpu_simulation_type), 0, hipMemcpyHostToDevice);
    PRINTERROR(status, " hipMemcpyToSymbol, sim copy to constants failed")
#endif
}

void upload_para_to_const(){

    int trans[TRANSDIM*TRANSDIM*TRANSDIM];
    // Data to trans
    {
        LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   1;
        LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   4;
        LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  10;
        LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  20;
        LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  35;
        LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  56;
        LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) =  84;
        LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 120;
        LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   3;
        LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   6;
        LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  17;
        LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  32;
        LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  48;
        LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  67;
        LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
        LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   9;
        LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  16;
        LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  23;
        LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  42;
        LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  73;
        LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
        LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  19;
        LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  31;
        LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  43;
        LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  79;
        LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
        LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  34;
        LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  49;
        LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  74;
        LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
        LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  55;
        LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  68;
        LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 107;
        LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  83;
        LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
        LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
        LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   2;
        LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   7;
        LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  15;
        LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  28;
        LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  50;
        LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  69;
        LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
        LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   5;
        LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  11;
        LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  26;
        LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  41;
        LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  59;
        LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  87;
        LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  13;
        LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  25;
        LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  36;
        LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  60;
        LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  88;
        LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  30;
        LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  40;
        LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  61;
        LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  94;
        LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  52;
        LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  58;
        LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  89;
        LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  71;
        LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  86;
        LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
        LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   8;
        LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  14;
        LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  22;
        LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  44;
        LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  75;
        LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
        LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  12;
        LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  24;
        LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  37;
        LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  62;
        LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  90;
        LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  21;
        LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  38;
        LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  66;
        LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  99;
        LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  46;
        LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  64;
        LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  98;
        LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  77;
        LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  92;
        LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
        LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  18;
        LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  27;
        LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  45;
        LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  80;
        LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
        LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  29;
        LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  39;
        LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  63;
        LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  95;
        LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  47;
        LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  65;
        LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  97;
        LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  81;
        LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  96;
        LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
        LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  33;
        LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  51;
        LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  76;
        LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
        LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  53;
        LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  57;
        LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  91;
        LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  78;
        LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  93;
        LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;
        LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  54;
        LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  70;
        LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
        LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  72;
        LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  85;
        LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
        LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  82;
        LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
        LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
        LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    }
    
    int Mcal[3*MCALDIM];
    {
        LOC2(Mcal, 0,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   1, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   2, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   3, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   4, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   5, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   6, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   7, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,   8, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   9, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  11, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  12, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  13, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  14, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  15, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  16, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  17, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  18, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  19, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  20, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  21, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  22, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  23, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  24, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  25, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  26, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  27, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  28, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  29, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  30, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  31, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  32, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  33, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  34, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  35, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  36, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  37, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  38, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  39, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  40, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  41, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  42, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  43, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  44, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  45, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  46, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  47, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  48, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  49, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  50, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  51, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  52, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  53, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  54, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  55, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  56, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  57, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  58, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  59, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  60, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  61, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  62, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  63, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  64, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  66, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  67, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  68, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  69, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  70, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  71, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  72, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  73, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  74, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  75, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  76, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  77, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  78, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  79, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  80, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  81, 3, MCALDIM) =   6;
        LOC2(Mcal, 0,  82, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  83, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  84, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  85, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  86, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  87, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  88, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  89, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  90, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  91, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  92, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  93, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  94, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  95, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  96, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  97, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  98, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  99, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 100, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 101, 3, MCALDIM) =   1;
        LOC2(Mcal, 0, 102, 3, MCALDIM) =   6;
        LOC2(Mcal, 0, 103, 3, MCALDIM) =   1;
        LOC2(Mcal, 0, 104, 3, MCALDIM) =   6;
        LOC2(Mcal, 0, 105, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 106, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 107, 3, MCALDIM) =   2;
        LOC2(Mcal, 0, 108, 3, MCALDIM) =   5;
        LOC2(Mcal, 0, 109, 3, MCALDIM) =   2;
        LOC2(Mcal, 0, 110, 3, MCALDIM) =   5;
        LOC2(Mcal, 0, 111, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 112, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 113, 3, MCALDIM) =   3;
        LOC2(Mcal, 0, 114, 3, MCALDIM) =   4;
        LOC2(Mcal, 0, 115, 3, MCALDIM) =   3;
        LOC2(Mcal, 0, 116, 3, MCALDIM) =   4;
        LOC2(Mcal, 0, 117, 3, MCALDIM) =   7;
        LOC2(Mcal, 0, 118, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 119, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   1, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   2, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   3, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   4, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   5, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   6, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   7, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   8, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,   9, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  11, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  12, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  13, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  14, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  15, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  16, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  17, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  18, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  19, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  20, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  21, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  22, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  23, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  24, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  25, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  26, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  27, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  28, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  29, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  30, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  31, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  32, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  33, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  34, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  35, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  36, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  37, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  38, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  39, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  40, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  41, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  42, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  43, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  44, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  45, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  46, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  47, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  48, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  49, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  50, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  51, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  52, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  53, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  54, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  55, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  56, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  57, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  58, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  59, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  60, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  61, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  62, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  63, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  64, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  66, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  67, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  68, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  69, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  70, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  71, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  72, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  73, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  74, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  75, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  76, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  77, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  78, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  79, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  80, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  81, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  82, 3, MCALDIM) =   6;
        LOC2(Mcal, 1,  83, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  84, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  85, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  86, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  87, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  88, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  89, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  90, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  91, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  92, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  93, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  94, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  95, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  96, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  97, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  98, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  99, 3, MCALDIM) =   1;
        LOC2(Mcal, 1, 100, 3, MCALDIM) =   6;
        LOC2(Mcal, 1, 101, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 102, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 103, 3, MCALDIM) =   6;
        LOC2(Mcal, 1, 104, 3, MCALDIM) =   1;
        LOC2(Mcal, 1, 105, 3, MCALDIM) =   2;
        LOC2(Mcal, 1, 106, 3, MCALDIM) =   5;
        LOC2(Mcal, 1, 107, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 108, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 109, 3, MCALDIM) =   5;
        LOC2(Mcal, 1, 110, 3, MCALDIM) =   2;
        LOC2(Mcal, 1, 111, 3, MCALDIM) =   3;
        LOC2(Mcal, 1, 112, 3, MCALDIM) =   4;
        LOC2(Mcal, 1, 113, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 114, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 115, 3, MCALDIM) =   4;
        LOC2(Mcal, 1, 116, 3, MCALDIM) =   3;
        LOC2(Mcal, 1, 117, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 118, 3, MCALDIM) =   7;
        LOC2(Mcal, 1, 119, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   1, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   2, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   3, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   4, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   5, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   6, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   7, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   8, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   9, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  11, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  12, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  13, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  14, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  15, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  16, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  17, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  18, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  19, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  20, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  21, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  22, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  23, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  24, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  25, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  26, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  27, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  28, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  29, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  30, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  31, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  32, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  33, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  34, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  35, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  36, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  37, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  38, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  39, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  40, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  41, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  42, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  43, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  44, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  45, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  46, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  47, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  48, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  49, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  50, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  51, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  52, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  53, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  54, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  55, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  56, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  57, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  58, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  59, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  60, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  61, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  62, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  63, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  64, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  66, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  67, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  68, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  69, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  70, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  71, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  72, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  73, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  74, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  75, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  76, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  77, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  78, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  79, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  80, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  81, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  82, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  83, 3, MCALDIM) =   6;
        LOC2(Mcal, 2,  84, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  85, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  86, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  87, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  88, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  89, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  90, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  91, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  92, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  93, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  94, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  95, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  96, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  97, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  98, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  99, 3, MCALDIM) =   6;
        LOC2(Mcal, 2, 100, 3, MCALDIM) =   1;
        LOC2(Mcal, 2, 101, 3, MCALDIM) =   6;
        LOC2(Mcal, 2, 102, 3, MCALDIM) =   1;
        LOC2(Mcal, 2, 103, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 104, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 105, 3, MCALDIM) =   5;
        LOC2(Mcal, 2, 106, 3, MCALDIM) =   2;
        LOC2(Mcal, 2, 107, 3, MCALDIM) =   5;
        LOC2(Mcal, 2, 108, 3, MCALDIM) =   2;
        LOC2(Mcal, 2, 109, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 110, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 111, 3, MCALDIM) =   4;
        LOC2(Mcal, 2, 112, 3, MCALDIM) =   3;
        LOC2(Mcal, 2, 113, 3, MCALDIM) =   4;
        LOC2(Mcal, 2, 114, 3, MCALDIM) =   3;
        LOC2(Mcal, 2, 115, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 116, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 117, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 118, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 119, 3, MCALDIM) =   7;
    }
    
    // upload to trans device location
    hipError_t status;
#ifdef TEST
    memcpy(devTrans, trans, sizeof(QUICKDouble)*TRANSDIM*TRANSDIM*TRANSDIM);
    memcpy(devMcal, Mcal, sizeof(QUICKDouble)*3*MCALDIM);
#else    
    status = hipMemcpyToSymbol(HIP_SYMBOL(devTrans), trans, sizeof(int)*TRANSDIM*TRANSDIM*TRANSDIM);
    PRINTERROR(status, " hipMemcpyToSymbol, Trans copy to constants failed")
    
    status = hipMemcpyToSymbol(HIP_SYMBOL(devMcal), Mcal, sizeof(int)*3*MCALDIM);
    PRINTERROR(status, " hipMemcpyToSymbol, Mcal copy to constants failed")
#endif
}

#ifdef DEBUG
static float totTime;
#endif

void get2e(_gpu_type gpu)
{
//    dim3 blocks(64,64);
//    gpu->threadsPerBlock = 1;
#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif
	get2e_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    totTime+=time;
    printf("this cycle:%f ms total time:%f ms\n", time, totTime);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif
    
}

__global__ void 
get2e_kernel()
{
    unsigned int offside = blockIdx.x*blockDim.x+threadIdx.x;
    int totalThreads = blockDim.x*gridDim.x;

    QUICKULL jshell   = (QUICKULL) devSim.sqrQshell;
    QUICKULL myInt    = (QUICKULL) jshell*jshell / totalThreads;

    if ((jshell*jshell - myInt*totalThreads)> offside) myInt++;

    for (QUICKULL i = 1; i<=myInt; i++) {

        QUICKULL currentInt = totalThreads * (i-1)+offside;        
        QUICKULL a = (QUICKULL) currentInt/jshell;
        QUICKULL b = (QUICKULL) (currentInt - a*jshell);
        
        /*
        QUICKULL a, b;
        double aa = (double)((currentInt+1)*1E-4);
        QUICKULL t = (QUICKULL)(sqrt(aa)*1E2);
        if ((currentInt+1)==t*t) {
            t--;
        }
        
        QUICKULL k = currentInt-t*t;
        if (k<=t) {
            a = k;
            b = t;
        }else {
            a = t;
            b = 2*t-k;
        }*/

        
        
        int II = devSim.sorted_YCutoffIJ[a].x;
        int JJ = devSim.sorted_YCutoffIJ[a].y;
        int KK = devSim.sorted_YCutoffIJ[b].x;
        int LL = devSim.sorted_YCutoffIJ[b].y;        
        
        int ii = devSim.sorted_Q[II];
        int jj = devSim.sorted_Q[JJ];
        int kk = devSim.sorted_Q[KK];
        int ll = devSim.sorted_Q[LL];
        
        if (ii<=kk){
            int nshell = devSim.nshell;
            QUICKDouble DNMax = MAX(MAX(4.0*LOC2(devSim.cutMatrix, ii, jj, nshell, nshell), 4.0*LOC2(devSim.cutMatrix, kk, ll, nshell, nshell)),
                                    MAX(MAX(LOC2(devSim.cutMatrix, ii, ll, nshell, nshell),     LOC2(devSim.cutMatrix, ii, kk, nshell, nshell)),
                                        MAX(LOC2(devSim.cutMatrix, jj, kk, nshell, nshell),     LOC2(devSim.cutMatrix, jj, ll, nshell, nshell))));
            
            if ((LOC2(devSim.YCutoff, kk, ll, nshell, nshell) * LOC2(devSim.YCutoff, ii, jj, nshell, nshell))> devSim.integralCutoff && \
                (LOC2(devSim.YCutoff, kk, ll, nshell, nshell) * LOC2(devSim.YCutoff, ii, jj, nshell, nshell) * DNMax) > devSim.integralCutoff) {
                
                int iii = devSim.sorted_Qnumber[II];
                int jjj = devSim.sorted_Qnumber[JJ];
                int kkk = devSim.sorted_Qnumber[KK];
                int lll = devSim.sorted_Qnumber[LL];
                
                iclass(iii, jjj, kkk, lll, ii, jj, kk, ll, DNMax);
                
            }
        }
    }
}

__device__ QUICKDouble quick_dsqr(QUICKDouble a)
{
    return a*a;
}



#ifndef TEST
__device__
#endif
void iclass(int I, int J, int K, int L, unsigned int II, unsigned int JJ, unsigned int KK, unsigned int LL, QUICKDouble DNMax)
{

    QUICKDouble RAx, RAy, RAz;
    QUICKDouble RBx, RBy, RBz;
    QUICKDouble RCx, RCy, RCz;
    QUICKDouble RDx, RDy, RDz;

    int kPrimI, kPrimJ, kPrimL, kPrimK;
    int kStartI, kStartJ, kStartK, kStartL;

    /* 
     kAtom A, B, C ,D is the coresponding atom for shell ii, jj, kk, ll
     and be careful with the index difference between Fortran and C++, 
     Fortran starts array index with 1 and C++ starts 0.
     */
    /*
     NII1 is the starting angular momenta for shell i and NII2 is the ending
     angular momenta.So it is with other varibles
     */
     
    /*RA, RB, RC, and RD are the coordinates for atom katomA, katomB, katomC and katomD, 
     which means they are corrosponding coorinates for shell II, JJ, KK, and LL.
     */
    RAx = LOC2(devSim.xyz, 0 , devSim.katom[II]-1, 3, devSim.natom);
    RAy = LOC2(devSim.xyz, 1 , devSim.katom[II]-1, 3, devSim.natom);
    RAz = LOC2(devSim.xyz, 2 , devSim.katom[II]-1, 3, devSim.natom);
    
    RCx = LOC2(devSim.xyz, 0 , devSim.katom[KK]-1, 3, devSim.natom);
    RCy = LOC2(devSim.xyz, 1 , devSim.katom[KK]-1, 3, devSim.natom);
    RCz = LOC2(devSim.xyz, 2 , devSim.katom[KK]-1, 3, devSim.natom);
    
    
    kPrimI = devSim.kprim[II];
    kPrimJ = devSim.kprim[JJ];
    kPrimK = devSim.kprim[KK];
    kPrimL = devSim.kprim[LL];
    
    kStartI = devSim.kstart[II];
    kStartJ = devSim.kstart[JJ];
    kStartK = devSim.kstart[KK];
    kStartL = devSim.kstart[LL];
    
    
    
    QUICKDouble store[STOREDIM*STOREDIM];
	memset(store, 0, STOREDIM*STOREDIM*sizeof(QUICKDouble));
    
    for (int JJJ = 0; JJJ < kPrimJ; JJJ++) {
        for (int III = 0; III < kPrimI; III++) {
			/* In the following comments, we have I, J, K, L denote the primitive gaussian function we use, and
             for example, expo(III, ksumtype(II)) stands for the expo for the IIIth primitive guassian function for II shell, 
             we use I to express the corresponding index.
             AA = expo(I)
             BB = expo(J)
             AB = expo(I)+expo(J)
                            1
             ABtemp = -------------------
                      2(expo(I) + expo(J))
             */
            QUICKDouble AB = LOC4(devSim.expoSum, III, JJJ, II, JJ, 6, 6, devSim.jshell, devSim.jshell);
            
			/*
                              --->                --->
             ->     expo(I) * xyz (I) + expo(J) * xyz(J)
             P  = ---------------------------------------
                              expo(I) + expo(J)
             
                                    -->             -->
             ----->        expo(I)*xyz(I) + expo(J)*xyz(J)                                 -->            -->
             AAtemp = ----------------------------------- * (expo(I) + expo(J)) = expo(I)*xyz(I)+expo(J)*xyz(J)
                                  expo(I) + expo(J)
             
             ----->   ->  ->
             Ptemp  = P - A
             */            
             
             QUICKDouble cutoffPrim = DNMax * LOC2(devSim.cutPrim, kStartI+III-1, kStartJ+JJJ-1, devSim.jbasis, devSim.jbasis);
            
            QUICKDouble X1 = LOC4(devSim.Xcoeff, kStartI+III-1, kStartJ+JJJ-1, I, J, devSim.jbasis, devSim.jbasis, 4, 4);
            
            QUICKDouble Px = LOC4(devSim.weightedCenterX, III, JJJ, II, JJ, 6, 6, devSim.jshell, devSim.jshell);
            QUICKDouble Py = LOC4(devSim.weightedCenterY, III, JJJ, II, JJ, 6, 6, devSim.jshell, devSim.jshell);
            QUICKDouble Pz = LOC4(devSim.weightedCenterZ, III, JJJ, II, JJ, 6, 6, devSim.jshell, devSim.jshell);
            
                
            for (int LLL = 0 ; LLL < kPrimL; LLL++) {
                for (int KKK = 0; KKK < kPrimK; KKK++) {
                    if (cutoffPrim * LOC2(devSim.cutPrim, kStartK+KKK-1, kStartL+LLL-1, devSim.jbasis, devSim.jbasis) > devSim.primLimit) {
                        /*
                         CC = expo(L)
                         DD = expo(K)
                         CD = expo(L)+expo(K)
                                            1
                         CDtemp = ----------------------
                                    2(expo(I) + expo(J))
                         ABCD = AB + CD = expo(I)+expo(J)+expo(K)+expo(L)
                                                 AB * CD      (expo(I)+expo(J))*(expo(K)+expo(L))
                         Rou(Greek Letter) =   ----------- = ------------------------------------
                                                 AB + CD         expo(I)+expo(J)+expo(K)+expo(L)

                                      expo(I)+expo(J)                        expo(K)+expo(L)
                         ABcom = --------------------------------  CDcom = --------------------------------
                                  expo(I)+expo(J)+expo(K)+expo(L)           expo(I)+expo(J)+expo(K)+expo(L)
                         
                         ABCDtemp = 1/2(expo(I)+expo(J)+expo(K)+expo(L))                    
                         */                        
                        QUICKDouble CD = LOC4(devSim.expoSum, KKK, LLL, KK, LL, 6, 6, devSim.jshell, devSim.jshell);
                        QUICKDouble ABCD = 1/(AB+CD);

                        /*
                         Q' is the weighting center of K and L
                                                   --->           --->
                         ->  ------>       expo(K)*xyz(K)+expo(L)*xyz(L)
                         Q = P'(K,L)  = ------------------------------
                                                 expo(K) + expo(L)
                         
                         W' is the weight center for I, J, K, L
                         
                                        --->             --->             --->            --->
                         ->     expo(I)*xyz(I) + expo(J)*xyz(J) + expo(K)*xyz(K) +expo(L)*xyz(L)
                         W = -------------------------------------------------------------------
                                             expo(I) + expo(J) + expo(K) + expo(L)                                            
                               ->  ->  2
                         RPQ =| P - Q | 
                         
                        ---->   ->  ->
                         Qtemp = Q - K
                         ----->   ->  ->
                         WQtemp = W - Q
                         ----->   ->  ->
                         WPtemp = W - P

                         ->  -> 2
                         T = ROU * | P - Q|
                         */
                         
                        QUICKDouble Qx = LOC4(devSim.weightedCenterX, KKK, LLL, KK, LL, 6, 6, devSim.jshell, devSim.jshell);
                        QUICKDouble Qy = LOC4(devSim.weightedCenterY, KKK, LLL, KK, LL, 6, 6, devSim.jshell, devSim.jshell);
                        QUICKDouble Qz = LOC4(devSim.weightedCenterZ, KKK, LLL, KK, LL, 6, 6, devSim.jshell, devSim.jshell);
                                                                        
                        QUICKDouble T = AB * CD * ABCD * ( quick_dsqr(Px-Qx) + quick_dsqr(Py-Qy) + quick_dsqr(Pz-Qz));

                        QUICKDouble YVerticalTemp[VDIM1*VDIM2*VDIM3];
                        FmT(I+J+K+L, T, YVerticalTemp);
                        QUICKDouble X2 = sqrt(ABCD) * X1 * LOC4(devSim.Xcoeff, kStartK+KKK-1, kStartL+LLL-1, K, L, devSim.jbasis, devSim.jbasis, 4, 4);
                        
                        for (int i = 0; i<=I+J+K+L; i++) {
                            VY(0, 0, i) = VY(0, 0, i) * X2;
                        }
                        
                        /*
                         X2 is the multiplication of four indices normalized coeffecient
                         */
                        
                       /* 
                        if (I+J+K+L != 0) {
                       
                            QUICKDouble tempx = (Px*AB+Qx*CD)*ABCD;
                            QUICKDouble tempy = (Py*AB+Qy*CD)*ABCD;
                            QUICKDouble tempz = (Pz*AB+Qz*CD)*ABCD;
                       
                            ABCD = ABCD * 0.5;
                            QUICKDouble ABtemp, CDtemp;
                            
                            if (I+J>0) {
                                //PSSS(0)
                                VY( 1, 0, 0) = (Px-RAx) * VY( 0, 0, 0) + (tempx - Px) * VY( 0, 0, 1);
                                VY( 2, 0, 0) = (Py-RAy) * VY( 0, 0, 0) + (tempy - Py) * VY( 0, 0, 1);
                                VY( 3, 0, 0) = (Pz-RAz) * VY( 0, 0, 0) + (tempz - Pz) * VY( 0, 0, 1);
                            }
                            
                            if (K+L>0) {
                                //SSPS(0)
                                VY( 0, 1, 0) = (Qx-RCx) * VY( 0, 0, 0) + (tempx - Qx) * VY( 0, 0, 1);
                                VY( 0, 2, 0) = (Qy-RCy) * VY( 0, 0, 0) + (tempy - Qy) * VY( 0, 0, 1);
                                VY( 0, 3, 0) = (Qz-RCz) * VY( 0, 0, 0) + (tempz - Qz) * VY( 0, 0, 1);
                            }
                            
                            if ((I+J>0 && K+L>0) || K+L>1) {
                                //SSPS(1, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                VY( 0, 1, 1) = (Qx-RCx) * VY( 0, 0, 1) + (tempx - Qx) * VY( 0, 0, 2);
                                VY( 0, 2, 1) = (Qy-RCy) * VY( 0, 0, 1) + (tempy - Qy) * VY( 0, 0, 2);
                                VY( 0, 3, 1) = (Qz-RCz) * VY( 0, 0, 1) + (tempz - Qz) * VY( 0, 0, 2);
                            }
                            
                            if (I+J>0 && K+L>0) {
                                
                                //PSPS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                VY( 1, 1, 0) = (Px-RAx) * VY( 0, 1, 0) + (tempx - Px) * VY( 0, 1, 1) + ABCD * VY( 0, 0, 1);
                                VY( 2, 1, 0) = (Py-RAy) * VY( 0, 1, 0) + (tempy - Py) * VY( 0, 1, 1);
                                VY( 3, 1, 0) = (Pz-RAz) * VY( 0, 1, 0) + (tempz - Pz) * VY( 0, 1, 1);

                                VY( 1, 2, 0) = (Px-RAx) * VY( 0, 2, 0) + (tempx - Px) * VY( 0, 2, 1);
                                VY( 2, 2, 0) = (Py-RAy) * VY( 0, 2, 0) + (tempy - Py) * VY( 0, 2, 1) + ABCD * VY( 0, 0, 1);
                                VY( 3, 2, 0) = (Pz-RAz) * VY( 0, 2, 0) + (tempz - Pz) * VY( 0, 2, 1);

                                VY( 1, 3, 0) = (Px-RAx) * VY( 0, 3, 0) + (tempx - Px) * VY( 0, 3, 1);
                                VY( 2, 3, 0) = (Py-RAy) * VY( 0, 3, 0) + (tempy - Py) * VY( 0, 3, 1);
                                VY( 3, 3, 0) = (Pz-RAz) * VY( 0, 3, 0) + (tempz - Pz) * VY( 0, 3, 1) + ABCD * VY( 0, 0, 1);
                            }
                            
                            if (I+J>1) {
                                ABtemp = 0.5/AB;
                                //PSSS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                VY( 1, 0, 1) = (Px-RAx) * VY( 0, 0, 1) + (tempx - Px) * VY( 0, 0, 2);
                                VY( 2, 0, 1) = (Py-RAy) * VY( 0, 0, 1) + (tempy - Py) * VY( 0, 0, 2);
                                VY( 3, 0, 1) = (Pz-RAz) * VY( 0, 0, 1) + (tempz - Pz) * VY( 0, 0, 2);
                                
                                //DSSS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5/AB, CD*ABCD);
                                VY( 4, 0, 0) = (Px-RAx) * VY( 2, 0, 0) + (tempx - Px) * VY( 2, 0, 1);
                                VY( 5, 0, 0) = (Py-RAy) * VY( 3, 0, 0) + (tempy - Py) * VY( 3, 0, 1);
                                VY( 6, 0, 0) = (Px-RAx) * VY( 3, 0, 0) + (tempx - Px) * VY( 3, 0, 1);
                                
                                VY( 7, 0, 0) = (Px-RAx) * VY( 1, 0, 0) + (tempx - Px) * VY( 1, 0, 1) + ABtemp *(VY( 0, 0, 0) - CD * 2 * ABCD * VY( 0, 0, 1));
                                VY( 8, 0, 0) = (Py-RAy) * VY( 2, 0, 0) + (tempy - Py) * VY( 2, 0, 1) + ABtemp *(VY( 0, 0, 0) - CD * 2 * ABCD * VY( 0, 0, 1));
                                VY( 9, 0, 0) = (Pz-RAz) * VY( 3, 0, 0) + (tempz - Pz) * VY( 3, 0, 1) + ABtemp *(VY( 0, 0, 0) - CD * 2 * ABCD * VY( 0, 0, 1));
                            }
                            
                            if (K+L>1) {
                                CDtemp = 1/CD;
                                //SSPS(1, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                VY( 0, 1, 1) = (Qx-RCx) * VY( 0, 0, 1) + (tempx - Qx) * VY( 0, 0, 2);
                                VY( 0, 2, 1) = (Qy-RCy) * VY( 0, 0, 1) + (tempy - Qy) * VY( 0, 0, 2);
                                VY( 0, 3, 1) = (Qz-RCz) * VY( 0, 0, 1) + (tempz - Qz) * VY( 0, 0, 2);
                                
                                //SSDS(0, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                VY( 0, 4, 0) = (Qx-RCx) * VY( 0, 2, 0) + (tempx - Qx) * VY( 0, 2, 1);
                                VY( 0, 5, 0) = (Qy-RCy) * VY( 0, 3, 0) + (tempy - Qy) * VY( 0, 3, 1);
                                VY( 0, 6, 0) = (Qx-RCx) * VY( 0, 3, 0) + (tempx - Qx) * VY( 0, 3, 1);
                                
                                VY( 0, 7, 0) = (Qx-RCx) * VY( 0, 1, 0) + (tempx - Qx) * VY( 0, 1, 1) + 0.5 * CDtemp *(VY( 0, 0, 0) - AB*2 * ABCD * VY( 0, 0, 1)) ;
                                VY( 0, 8, 0) = (Qy-RCy) * VY( 0, 2, 0) + (tempy - Qy) * VY( 0, 2, 1) + 0.5 * CDtemp *(VY( 0, 0, 0) - AB*2 * ABCD * VY( 0, 0, 1)) ;
                                VY( 0, 9, 0) = (Qz-RCz) * VY( 0, 3, 0) + (tempz - Qz) * VY( 0, 3, 1) + 0.5 * CDtemp *(VY( 0, 0, 0) - AB*2 * ABCD * VY( 0, 0, 1)) ;
                            }
                            
                            if (I+J>1 && K+L>0) {
                                //PSSS(2, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                VY( 1, 0, 2) = (Px-RAx) * VY( 0, 0, 2) + (tempx - Px) * VY( 0, 0, 3);
                                VY( 2, 0, 2) = (Py-RAy) * VY( 0, 0, 2) + (tempy - Py) * VY( 0, 0, 3);
                                VY( 3, 0, 2) = (Pz-RAz) * VY( 0, 0, 2) + (tempz - Pz) * VY( 0, 0, 3);
                                
                                //DSSS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5/AB, CD*ABCD);
                                VY( 4, 0, 1) = (Px-RAx) * VY( 2, 0, 1) + (tempx - Px) * VY( 2, 0, 2);
                                VY( 5, 0, 1) = (Py-RAy) * VY( 3, 0, 1) + (tempy - Py) * VY( 3, 0, 2);
                                VY( 6, 0, 1) = (Px-RAx) * VY( 3, 0, 1) + (tempx - Px) * VY( 3, 0, 2);
                                
                                VY( 7, 0, 1) = (Px-RAx) * VY( 1, 0, 1) + (tempx - Px) * VY( 1, 0, 2) + ABtemp *( VY( 0, 0, 1) - 2 * CD * ABCD * VY( 0, 0, 2));
                                VY( 8, 0, 1) = (Py-RAy) * VY( 2, 0, 1) + (tempy - Py) * VY( 2, 0, 2) + ABtemp *( VY( 0, 0, 1) - 2 * CD * ABCD * VY( 0, 0, 2));
                                VY( 9, 0, 1) = (Pz-RAz) * VY( 3, 0, 1) + (tempz - Pz) * VY( 3, 0, 2) + ABtemp *( VY( 0, 0, 1) - 2 * CD * ABCD * VY( 0, 0, 2));
                                
                                //DSPS(0, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5*ABCD);
                                VY( 4, 1, 0) = (Qx-RCx) * VY( 4, 0, 0) + (tempx - Qx) * VY( 4, 0, 1) + ABCD * VY( 2, 0, 1);
                                VY( 4, 2, 0) = (Qy-RCy) * VY( 4, 0, 0) + (tempy - Qy) * VY( 4, 0, 1) + ABCD * VY( 1, 0, 1);
                                VY( 4, 3, 0) = (Qz-RCz) * VY( 4, 0, 0) + (tempz - Qz) * VY( 4, 0, 1);
                                
                                VY( 5, 1, 0) = (Qx-RCx) * VY( 5, 0, 0) + (tempx - Qx) * VY( 5, 0, 1);
                                VY( 5, 2, 0) = (Qy-RCy) * VY( 5, 0, 0) + (tempy - Qy) * VY( 5, 0, 1) + ABCD * VY( 3, 0, 1);
                                VY( 5, 3, 0) = (Qz-RCz) * VY( 5, 0, 0) + (tempz - Qz) * VY( 5, 0, 1) + ABCD * VY( 2, 0, 1);
                                
                                VY( 6, 1, 0) = (Qx-RCx) * VY( 6, 0, 0) + (tempx - Qx) * VY( 6, 0, 1) + ABCD * VY( 3, 0, 1);
                                VY( 6, 2, 0) = (Qy-RCy) * VY( 6, 0, 0) + (tempy - Qy) * VY( 6, 0, 1);
                                VY( 6, 3, 0) = (Qz-RCz) * VY( 6, 0, 0) + (tempz - Qz) * VY( 6, 0, 1) + ABCD * VY( 1, 0, 1);
                                
                                VY( 7, 1, 0) = (Qx-RCx) * VY( 7, 0, 0) + (tempx - Qx) * VY( 7, 0, 1) + 2 * ABCD * VY( 1, 0, 1);
                                VY( 7, 2, 0) = (Qy-RCy) * VY( 7, 0, 0) + (tempy - Qy) * VY( 7, 0, 1);
                                VY( 7, 3, 0) = (Qz-RCz) * VY( 7, 0, 0) + (tempz - Qz) * VY( 7, 0, 1);
                                
                                VY( 8, 1, 0) = (Qx-RCx) * VY( 8, 0, 0) + (tempx - Qx) * VY( 8, 0, 1);
                                VY( 8, 2, 0) = (Qy-RCy) * VY( 8, 0, 0) + (tempy - Qy) * VY( 8, 0, 1) + 2 * ABCD * VY( 2, 0, 1);
                                VY( 8, 3, 0) = (Qz-RCz) * VY( 8, 0, 0) + (tempz - Qz) * VY( 8, 0, 1);
                                
                                VY( 9, 1, 0) = (Qx-RCx) * VY( 9, 0, 0) + (tempx - Qx) * VY( 9, 0, 1);
                                VY( 9, 2, 0) = (Qy-RCy) * VY( 9, 0, 0) + (tempy - Qy) * VY( 9, 0, 1);
                                VY( 9, 3, 0) = (Qz-RCz) * VY( 9, 0, 0) + (tempz - Qz) * VY( 9, 0, 1) + 2 * ABCD * VY( 3, 0, 1);
                            }
                            
                            if (I+J>0 && K+L>1) {
                            
                                //SSPS(2, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                VY( 0, 1, 2) = (Qx-RCx) * VY( 0, 0, 2) + (tempx - Qx) * VY( 0, 0, 3);
                                VY( 0, 2, 2) = (Qy-RCy) * VY( 0, 0, 2) + (tempy - Qy) * VY( 0, 0, 3);
                                VY( 0, 3, 2) = (Qz-RCz) * VY( 0, 0, 2) + (tempz - Qz) * VY( 0, 0, 3);
                                
                                //SSDS(1, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                VY( 0, 4, 1) = (Qx-RCx) * VY( 0, 2, 1) + (tempx - Qx) * VY( 0, 2, 2);
                                VY( 0, 5, 1) = (Qy-RCy) * VY( 0, 3, 1) + (tempy - Qy) * VY( 0, 3, 2);
                                VY( 0, 6, 1) = (Qx-RCx) * VY( 0, 3, 1) + (tempx - Qx) * VY( 0, 3, 2);
                                
                                VY( 0, 7, 1) = (Qx-RCx) * VY( 0, 1, 1) + (tempx - Qx) * VY( 0, 1, 2) + 1 * CDtemp *( 0.5 * VY( 0, 0, 1) - AB*ABCD * VY( 0, 0, 2)) ;
                                VY( 0, 8, 1) = (Qy-RCy) * VY( 0, 2, 1) + (tempy - Qy) * VY( 0, 2, 2) + 1 * CDtemp *( 0.5 * VY( 0, 0, 1) - AB*ABCD * VY( 0, 0, 2)) ;
                                VY( 0, 9, 1) = (Qz-RCz) * VY( 0, 3, 1) + (tempz - Qz) * VY( 0, 3, 2) + 1 * CDtemp *( 0.5 * VY( 0, 0, 1) - AB*ABCD * VY( 0, 0, 2)) ;
                                
                                //PSDS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                VY( 1, 4, 0) = (Px-RAx) * VY( 0, 4, 0) + (tempx - Px) * VY( 0, 4, 1) + ABCD * VY( 0, 2, 1);
                                VY( 2, 4, 0) = (Py-RAy) * VY( 0, 4, 0) + (tempy - Py) * VY( 0, 4, 1) + ABCD * VY( 0, 1, 1);
                                VY( 3, 4, 0) = (Pz-RAz) * VY( 0, 4, 0) + (tempz - Pz) * VY( 0, 4, 1) ;
                                
                                VY( 1, 5, 0) = (Px-RAx) * VY( 0, 5, 0) + (tempx - Px) * VY( 0, 5, 1);
                                VY( 2, 5, 0) = (Py-RAy) * VY( 0, 5, 0) + (tempy - Py) * VY( 0, 5, 1) + ABCD * VY( 0, 3, 1);
                                VY( 3, 5, 0) = (Pz-RAz) * VY( 0, 5, 0) + (tempz - Pz) * VY( 0, 5, 1) + ABCD * VY( 0, 2, 1);
                                
                                VY( 1, 6, 0) = (Px-RAx) * VY( 0, 6, 0) + (tempx - Px) * VY( 0, 6, 1) + ABCD * VY( 0, 3, 1);
                                VY( 2, 6, 0) = (Py-RAy) * VY( 0, 6, 0) + (tempy - Py) * VY( 0, 6, 1);
                                VY( 3, 6, 0) = (Pz-RAz) * VY( 0, 6, 0) + (tempz - Pz) * VY( 0, 6, 1) + ABCD * VY( 0, 1, 1);
                                
                                VY( 1, 7, 0) = (Px-RAx) * VY( 0, 7, 0) + (tempx - Px) * VY( 0, 7, 1) + 2 * ABCD * VY( 0, 1, 1);
                                VY( 2, 7, 0) = (Py-RAy) * VY( 0, 7, 0) + (tempy - Py) * VY( 0, 7, 1);
                                VY( 3, 7, 0) = (Pz-RAz) * VY( 0, 7, 0) + (tempz - Pz) * VY( 0, 7, 1);

                                VY( 1, 8, 0) = (Px-RAx) * VY( 0, 8, 0) + (tempx - Px) * VY( 0, 8, 1);
                                VY( 2, 8, 0) = (Py-RAy) * VY( 0, 8, 0) + (tempy - Py) * VY( 0, 8, 1) + 2 * ABCD * VY( 0, 2, 1);
                                VY( 3, 8, 0) = (Pz-RAz) * VY( 0, 8, 0) + (tempz - Pz) * VY( 0, 8, 1);
                                
                                VY( 1, 9, 0) = (Px-RAx) * VY( 0, 9, 0) + (tempx - Px) * VY( 0, 9, 1);
                                VY( 2, 9, 0) = (Py-RAy) * VY( 0, 9, 0) + (tempy - Py) * VY( 0, 9, 1);
                                VY( 3, 9, 0) = (Pz-RAz) * VY( 0, 9, 0) + (tempz - Pz) * VY( 0, 9, 1) + 2 * ABCD * VY( 0, 3, 1);                                

                            }

                            if (I+J>1 && K+L>1) {
                                //SSPS(3, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                VY( 0, 1, 3) = (Qx-RCx) * VY( 0, 0, 3) + (tempx - Qx) * VY( 0, 0, 4);
                                VY( 0, 2, 3) = (Qy-RCy) * VY( 0, 0, 3) + (tempy - Qy) * VY( 0, 0, 4);
                                VY( 0, 3, 3) = (Qz-RCz) * VY( 0, 0, 3) + (tempz - Qz) * VY( 0, 0, 4);
                                
                                //SSDS(2, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                VY( 0, 4, 2) = (Qx-RCx) * VY( 0, 2, 2) + (tempx - Qx) * VY( 0, 2, 3);
                                VY( 0, 5, 2) = (Qy-RCy) * VY( 0, 3, 2) + (tempy - Qy) * VY( 0, 3, 3);
                                VY( 0, 6, 2) = (Qx-RCx) * VY( 0, 3, 2) + (tempx - Qx) * VY( 0, 3, 3);
                                
                                VY( 0, 7, 2) = (Qx-RCx) * VY( 0, 1, 2) + (tempx - Qx) * VY( 0, 1, 3) + CDtemp *( 0.5 * VY( 0, 0, 2) - AB*ABCD * VY( 0, 0, 3)) ;
                                VY( 0, 8, 2) = (Qy-RCy) * VY( 0, 2, 2) + (tempy - Qy) * VY( 0, 2, 3) + CDtemp *( 0.5 * VY( 0, 0, 2) - AB*ABCD * VY( 0, 0, 3)) ;
                                VY( 0, 9, 2) = (Qz-RCz) * VY( 0, 3, 2) + (tempz - Qz) * VY( 0, 3, 3) + CDtemp *( 0.5 * VY( 0, 0, 2) - AB*ABCD * VY( 0, 0, 3)) ;
                                
                                //PSDS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                VY( 1, 4, 1) = (Px-RAx) * VY( 0, 4, 1) + (tempx - Px) * VY( 0, 4, 2) + ABCD * VY( 0, 2, 2);
                                VY( 2, 4, 1) = (Py-RAy) * VY( 0, 4, 1) + (tempy - Py) * VY( 0, 4, 2) + ABCD * VY( 0, 1, 2);
                                VY( 3, 4, 1) = (Pz-RAz) * VY( 0, 4, 1) + (tempz - Pz) * VY( 0, 4, 2) ;
                                
                                VY( 1, 5, 1) = (Px-RAx) * VY( 0, 5, 1) + (tempx - Px) * VY( 0, 5, 2);
                                VY( 2, 5, 1) = (Py-RAy) * VY( 0, 5, 1) + (tempy - Py) * VY( 0, 5, 2) + ABCD * VY( 0, 3, 2);
                                VY( 3, 5, 1) = (Pz-RAz) * VY( 0, 5, 1) + (tempz - Pz) * VY( 0, 5, 2) + ABCD * VY( 0, 2, 2);
                                
                                VY( 1, 6, 1) = (Px-RAx) * VY( 0, 6, 1) + (tempx - Px) * VY( 0, 6, 2) + ABCD * VY( 0, 3, 2);
                                VY( 2, 6, 1) = (Py-RAy) * VY( 0, 6, 1) + (tempy - Py) * VY( 0, 6, 2);
                                VY( 3, 6, 1) = (Pz-RAz) * VY( 0, 6, 1) + (tempz - Pz) * VY( 0, 6, 2) + ABCD * VY( 0, 1, 2);
                                
                                VY( 1, 7, 1) = (Px-RAx) * VY( 0, 7, 1) + (tempx - Px) * VY( 0, 7, 2) + 2 * ABCD * VY( 0, 1, 2);
                                VY( 2, 7, 1) = (Py-RAy) * VY( 0, 7, 1) + (tempy - Py) * VY( 0, 7, 2);
                                VY( 3, 7, 1) = (Pz-RAz) * VY( 0, 7, 1) + (tempz - Pz) * VY( 0, 7, 2);

                                VY( 1, 8, 1) = (Px-RAx) * VY( 0, 8, 1) + (tempx - Px) * VY( 0, 8, 2);
                                VY( 2, 8, 1) = (Py-RAy) * VY( 0, 8, 1) + (tempy - Py) * VY( 0, 8, 2) + 2 * ABCD * VY( 0, 2, 2);
                                VY( 3, 8, 1) = (Pz-RAz) * VY( 0, 8, 1) + (tempz - Pz) * VY( 0, 8, 2);
                                
                                VY( 1, 9, 1) = (Px-RAx) * VY( 0, 9, 1) + (tempx - Px) * VY( 0, 9, 2);
                                VY( 2, 9, 1) = (Py-RAy) * VY( 0, 9, 1) + (tempy - Py) * VY( 0, 9, 2);
                                VY( 3, 9, 1) = (Pz-RAz) * VY( 0, 9, 1) + (tempz - Pz) * VY( 0, 9, 2) + 2 * ABCD * VY( 0, 3, 2);
                                
                                //PSPS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                VY( 1, 1, 1) = (Px-RAx) * VY( 0, 1, 1) + (tempx - Px) * VY( 0, 1, 2) + ABCD * VY( 0, 0, 2);
                                VY( 2, 1, 1) = (Py-RAy) * VY( 0, 1, 1) + (tempy - Py) * VY( 0, 1, 2);
                                VY( 3, 1, 1) = (Pz-RAz) * VY( 0, 1, 1) + (tempz - Pz) * VY( 0, 1, 2);

                                VY( 1, 2, 1) = (Px-RAx) * VY( 0, 2, 1) + (tempx - Px) * VY( 0, 2, 2);
                                VY( 2, 2, 1) = (Py-RAy) * VY( 0, 2, 1) + (tempy - Py) * VY( 0, 2, 2) + ABCD * VY( 0, 0, 2);
                                VY( 3, 2, 1) = (Pz-RAz) * VY( 0, 2, 1) + (tempz - Pz) * VY( 0, 2, 2);

                                VY( 1, 3, 1) = (Px-RAx) * VY( 0, 3, 1) + (tempx - Px) * VY( 0, 3, 2);
                                VY( 2, 3, 1) = (Py-RAy) * VY( 0, 3, 1) + (tempy - Py) * VY( 0, 3, 2);
                                VY( 3, 3, 1) = (Pz-RAz) * VY( 0, 3, 1) + (tempz - Pz) * VY( 0, 3, 2) + ABCD * VY( 0, 0, 2);
                                
                                //DSDS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD, 0.5/AB, CD*ABCD); 
                                VY( 4, 4, 0) = (Px-RAx) * VY( 2, 4, 0) + (tempx - Px) * VY( 2, 4, 1) + ABCD * VY( 2, 2, 1);
                                VY( 4, 5, 0) = (Px-RAx) * VY( 2, 5, 0) + (tempx - Px) * VY( 2, 5, 1);
                                VY( 4, 6, 0) = (Px-RAx) * VY( 2, 6, 0) + (tempx - Px) * VY( 2, 6, 1) + ABCD * VY( 2, 3, 1);
                                VY( 4, 7, 0) = (Px-RAx) * VY( 2, 7, 0) + (tempx - Px) * VY( 2, 7, 1) + 2 * ABCD * VY( 2, 1, 1);
                                VY( 4, 8, 0) = (Px-RAx) * VY( 2, 8, 0) + (tempx - Px) * VY( 2, 8, 1);
                                VY( 4, 9, 0) = (Px-RAx) * VY( 2, 9, 0) + (tempx - Px) * VY( 2, 9, 1);
                                
                                VY( 5, 4, 0) = (Py-RAy) * VY( 3, 4, 0) + (tempy - Py) * VY( 3, 4, 1) + ABCD * VY( 3, 1, 1);
                                VY( 5, 5, 0) = (Py-RAy) * VY( 3, 5, 0) + (tempy - Py) * VY( 3, 5, 1) + ABCD * VY( 3, 3, 1);
                                VY( 5, 6, 0) = (Py-RAy) * VY( 3, 6, 0) + (tempy - Py) * VY( 3, 6, 1);
                                VY( 5, 7, 0) = (Py-RAy) * VY( 3, 7, 0) + (tempy - Py) * VY( 3, 7, 1);
                                VY( 5, 8, 0) = (Py-RAy) * VY( 3, 8, 0) + (tempy - Py) * VY( 3, 8, 1) + 2 * ABCD * VY( 3, 2, 1);
                                VY( 5, 9, 0) = (Py-RAy) * VY( 3, 9, 0) + (tempy - Py) * VY( 3, 9, 1);
                                
                                VY( 6, 4, 0) = (Px-RAx) * VY( 3, 4, 0) + (tempx - Px) * VY( 3, 4, 1) + ABCD * VY( 3, 2, 1);
                                VY( 6, 5, 0) = (Px-RAx) * VY( 3, 5, 0) + (tempx - Px) * VY( 3, 5, 1);
                                VY( 6, 6, 0) = (Px-RAx) * VY( 3, 6, 0) + (tempx - Px) * VY( 3, 6, 1) + ABCD * VY( 3, 3, 1);
                                VY( 6, 7, 0) = (Px-RAx) * VY( 3, 7, 0) + (tempx - Px) * VY( 3, 7, 1) + 2 * ABCD * VY( 3, 1, 1);
                                VY( 6, 8, 0) = (Px-RAx) * VY( 3, 8, 0) + (tempx - Px) * VY( 3, 8, 1);
                                VY( 6, 9, 0) = (Px-RAx) * VY( 3, 9, 0) + (tempx - Px) * VY( 3, 9, 1);
                                
                                VY( 7, 4, 0) = (Px-RAx) * VY( 1, 4, 0) + (tempx - Px) * VY( 1, 4, 1) +  ABtemp * (VY( 0, 4,0)-2 * CD*ABCD*VY( 0, 4,1)) + ABCD * VY( 1, 2, 1);
                                VY( 7, 5, 0) = (Px-RAx) * VY( 1, 5, 0) + (tempx - Px) * VY( 1, 5, 1) +  ABtemp * (VY( 0, 5,0)-2 * CD*ABCD*VY( 0, 5,1));
                                VY( 7, 6, 0) = (Px-RAx) * VY( 1, 6, 0) + (tempx - Px) * VY( 1, 6, 1) +  ABtemp * (VY( 0, 6,0)-2 * CD*ABCD*VY( 0, 6,1)) + ABCD * VY( 1, 3, 1);
                                VY( 7, 7, 0) = (Px-RAx) * VY( 1, 7, 0) + (tempx - Px) * VY( 1, 7, 1) +  ABtemp * (VY( 0, 7,0)-2 * CD*ABCD*VY( 0, 7,1)) + 2*ABCD * VY( 1, 1, 1);
                                VY( 7, 8, 0) = (Px-RAx) * VY( 1, 8, 0) + (tempx - Px) * VY( 1, 8, 1) +  ABtemp * (VY( 0, 8,0)-2 * CD*ABCD*VY( 0, 8,1));
                                VY( 7, 9, 0) = (Px-RAx) * VY( 1, 9, 0) + (tempx - Px) * VY( 1, 9, 1) +  ABtemp * (VY( 0, 9,0)-2 * CD*ABCD*VY( 0, 9,1));
                                
                                
                                VY( 8, 4, 0) = (Py-RAy) * VY( 2, 4, 0) + (tempy - Py) * VY( 2, 4, 1) +  ABtemp * (VY( 0, 4,0)-2 * CD*ABCD*VY( 0, 4,1)) + ABCD * VY( 2, 1, 1);
                                VY( 8, 5, 0) = (Py-RAy) * VY( 2, 5, 0) + (tempy - Py) * VY( 2, 5, 1) +  ABtemp * (VY( 0, 5,0)-2 * CD*ABCD*VY( 0, 5,1)) + ABCD * VY( 2, 3, 1);
                                VY( 8, 6, 0) = (Py-RAy) * VY( 2, 6, 0) + (tempy - Py) * VY( 2, 6, 1) +  ABtemp * (VY( 0, 6,0)-2 * CD*ABCD*VY( 0, 6,1));
                                VY( 8, 7, 0) = (Py-RAy) * VY( 2, 7, 0) + (tempy - Py) * VY( 2, 7, 1) +  ABtemp * (VY( 0, 7,0)-2 * CD*ABCD*VY( 0, 7,1));
                                VY( 8, 8, 0) = (Py-RAy) * VY( 2, 8, 0) + (tempy - Py) * VY( 2, 8, 1) +  ABtemp * (VY( 0, 8,0)-2 * CD*ABCD*VY( 0, 8,1)) + 2*ABCD * VY( 2, 2, 1);
                                VY( 8, 9, 0) = (Py-RAy) * VY( 2, 9, 0) + (tempy - Py) * VY( 2, 9, 1) +  ABtemp * (VY( 0, 9,0)-2 * CD*ABCD*VY( 0, 9,1));
                                
                                VY( 9, 4, 0) = (Pz-RAz) * VY( 3, 4, 0) + (tempz - Pz) * VY( 3, 4, 1) +  ABtemp * (VY( 0, 4,0)-2 * CD*ABCD*VY( 0, 4,1));
                                VY( 9, 5, 0) = (Pz-RAz) * VY( 3, 5, 0) + (tempz - Pz) * VY( 3, 5, 1) +  ABtemp * (VY( 0, 5,0)-2 * CD*ABCD*VY( 0, 5,1)) + ABCD * VY( 3, 2, 1);
                                VY( 9, 6, 0) = (Pz-RAz) * VY( 3, 6, 0) + (tempz - Pz) * VY( 3, 6, 1) +  ABtemp * (VY( 0, 6,0)-2 * CD*ABCD*VY( 0, 6,1)) + ABCD * VY( 3, 1, 1);
                                VY( 9, 7, 0) = (Pz-RAz) * VY( 3, 7, 0) + (tempz - Pz) * VY( 3, 7, 1) +  ABtemp * (VY( 0, 7,0)-2 * CD*ABCD*VY( 0, 7,1));
                                VY( 9, 8, 0) = (Pz-RAz) * VY( 3, 8, 0) + (tempz - Pz) * VY( 3, 8, 1) +  ABtemp * (VY( 0, 8,0)-2 * CD*ABCD*VY( 0, 8,1));
                                VY( 9, 9, 0) = (Pz-RAz) * VY( 3, 9, 0) + (tempz - Pz) * VY( 3, 9, 1) +  ABtemp * (VY( 0, 9,0)-2 * CD*ABCD*VY( 0, 9,1)) + 2*ABCD * VY( 3, 3, 1);
                                
                            }
                        }*/
                        
                        LOC2(store, 0, 0, STOREDIM, STOREDIM) += VY( 0, 0, 0);
                        
                        //------------- FOR CASE I+J+K+L > 0 -----------------------------------
                        // includes NABCDTYPE = 1,10,2,20,11,12,21,22 situation
                        //----------------------------------------------------------------------
                        if (I+J+K+L != 0) {
                            int NABCDTYPE = (I+J)*10+K+L;
                            QUICKDouble tempx = (Px*AB+Qx*CD)*ABCD;
                            QUICKDouble tempy = (Py*AB+Qy*CD)*ABCD;
                            QUICKDouble tempz = (Pz*AB+Qz*CD)*ABCD;
                            
                            if (NABCDTYPE == 10) { //NABCDTYPE: 10
                                //PSSS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, (Px*AB+Qx*CD)*ABCD - Px, (Py*AB+Qy*CD)*ABCD - Py, tempz - Pz);
                                LOC2(store, 1, 0, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 0, 0) + (tempx - Px) * VY( 0, 0, 1);
                                LOC2(store, 2, 0, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 0, 0) + (tempy - Py) * VY( 0, 0, 1);
                                LOC2(store, 3, 0, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 0, 0) + (tempz - Pz) * VY( 0, 0, 1);
                                 
                            }
                            
                            if (NABCDTYPE == 1){ //NABCDTYPE: 10
                                //SSPS(0, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                LOC2(store, 0, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 0, 0, 0) + (tempx - Qx) * VY( 0, 0, 1);
                                LOC2(store, 0, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 0, 0, 0) + (tempy - Qy) * VY( 0, 0, 1);
                                LOC2(store, 0, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 0, 0, 0) + (tempz - Qz) * VY( 0, 0, 1);
                                 
                            }
                            
                            //------------- FOR CASE I+J+K+L > 1 -----------------------------------
                            // includes NABCDTYPE = 2,20,11,12,21,22 situation
                            //----------------------------------------------------------------------
                            if (I+J+K+L>1) { 
                                
                                if (K+L>0) {
                                    //SSPS(0, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                    VY( 0, 1, 0) = (Qx-RCx) * VY( 0, 0, 0) + (tempx - Qx) * VY( 0, 0, 1);
                                    VY( 0, 2, 0) = (Qy-RCy) * VY( 0, 0, 0) + (tempy - Qy) * VY( 0, 0, 1);
                                    VY( 0, 3, 0) = (Qz-RCz) * VY( 0, 0, 0) + (tempz - Qz) * VY( 0, 0, 1);
                                    if (I==0) {
                                    LOC2(store, 0, 1, STOREDIM, STOREDIM)+= VY( 0, 1, 0);
                                    LOC2(store, 0, 2, STOREDIM, STOREDIM)+= VY( 0, 2, 0);
                                    LOC2(store, 0, 3, STOREDIM, STOREDIM)+= VY( 0, 3, 0);
                                    }
                                }
                                
                                if (I+J>0) {
                                    //PSSS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                    VY( 1, 0, 0) = (Px-RAx) * VY( 0, 0, 0) + (tempx - Px) * VY( 0, 0, 1);
                                    VY( 2, 0, 0) = (Py-RAy) * VY( 0, 0, 0) + (tempy - Py) * VY( 0, 0, 1);
                                    VY( 3, 0, 0) = (Pz-RAz) * VY( 0, 0, 0) + (tempz - Pz) * VY( 0, 0, 1);
                                    if(K==0) {
                                        LOC2(store, 1, 0, STOREDIM, STOREDIM)+= (Px-RAx) * VY( 0, 0, 0) + (tempx - Px) * VY( 0, 0, 1);
                                        LOC2(store, 2, 0, STOREDIM, STOREDIM)+= (Py-RAy) * VY( 0, 0, 0) + (tempy - Py) * VY( 0, 0, 1);
                                        LOC2(store, 3, 0, STOREDIM, STOREDIM)+= (Pz-RAz) * VY( 0, 0, 0) + (tempz - Pz) * VY( 0, 0, 1);
                                    }
                                }
                                
                                if ((I+J)>0 && (K+L)>0 || (K+L)>1) {
                                    //SSPS(1, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, (Px*AB+Qx*CD)*ABCD - Qx, (Py*AB+Qy*CD)*ABCD - Qy, (Pz*AB+Qz*CD)*ABCD - Qz);
                                    VY( 0, 1, 1) = (Qx-RCx) * VY( 0, 0, 1) + (tempx - Qx) * VY( 0, 0, 2);
                                    VY( 0, 2, 1) = (Qy-RCy) * VY( 0, 0, 1) + (tempy - Qy) * VY( 0, 0, 2);
                                    VY( 0, 3, 1) = (Qz-RCz) * VY( 0, 0, 1) + (tempz - Qz) * VY( 0, 0, 2);
                                }
                                
                                if(NABCDTYPE == 2) {
                                    LOC2(store, 0, 4, STOREDIM, STOREDIM)+= (Qx-RCx) * VY( 0, 2, 0) + (tempx - Qx) * VY( 0, 2, 1);
                                    LOC2(store, 0, 5, STOREDIM, STOREDIM)+= (Qy-RCy) * VY( 0, 3, 0) + (tempy - Qy) * VY( 0, 3, 1);
                                    LOC2(store, 0, 6, STOREDIM, STOREDIM)+= (Qx-RCx) * VY( 0, 3, 0) + (tempx - Qx) * VY( 0, 3, 1);
                                    
                                    QUICKDouble tmp = 0.5 / CD *(VY( 0, 0, 0) - AB*ABCD * VY( 0, 0, 1));
                                    LOC2(store, 0, 7, STOREDIM, STOREDIM)+= (Qx-RCx) * VY( 0, 1, 0) + (tempx - Qx) * VY( 0, 1, 1) + tmp ;
                                    LOC2(store, 0, 8, STOREDIM, STOREDIM)+= (Qy-RCy) * VY( 0, 2, 0) + (tempy - Qy) * VY( 0, 2, 1) + tmp ;
                                    LOC2(store, 0, 9, STOREDIM, STOREDIM)+= (Qz-RCz) * VY( 0, 3, 0) + (tempz - Qz) * VY( 0, 3, 1) + tmp ;
                                     
                                }
                                
                                if ((I+J)>1) {
                                    //PSSS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                    VY( 1, 0, 1) = (Px-RAx) * VY( 0, 0, 1) + (tempx - Px) * VY( 0, 0, 2);
                                    VY( 2, 0, 1) = (Py-RAy) * VY( 0, 0, 1) + (tempy - Py) * VY( 0, 0, 2);
                                    VY( 3, 0, 1) = (Pz-RAz) * VY( 0, 0, 1) + (tempz - Pz) * VY( 0, 0, 2);
                                }
                                
                                if(NABCDTYPE == 20) {
                                    LOC2(store, 4, 0, STOREDIM, STOREDIM)+= (Px-RAx) * VY( 2, 0, 0) + (tempx - Px) * VY( 2, 0, 1);
                                    LOC2(store, 5, 0, STOREDIM, STOREDIM)+= (Py-RAy) * VY( 3, 0, 0) + (tempy - Py) * VY( 3, 0, 1);
                                    LOC2(store, 6, 0, STOREDIM, STOREDIM)+= (Px-RAx) * VY( 3, 0, 0) + (tempx - Px) * VY( 3, 0, 1);
                                    
                                    QUICKDouble tmp = 0.5 / AB *(VY( 0, 0, 0) - CD * ABCD * VY( 0, 0, 1));
                                    LOC2(store, 7, 0, STOREDIM, STOREDIM)+= (Px-RAx) * VY( 1, 0, 0) + (tempx - Px) * VY( 1, 0, 1) + tmp;
                                    LOC2(store, 8, 0, STOREDIM, STOREDIM)+= (Py-RAy) * VY( 2, 0, 0) + (tempy - Py) * VY( 2, 0, 1) + tmp;
                                    LOC2(store, 9, 0, STOREDIM, STOREDIM)+= (Pz-RAz) * VY( 3, 0, 0) + (tempz - Pz) * VY( 3, 0, 1) + tmp;
                                     
                                }
                                
                                if((I+J) != 0 && (K+L) != 0) {// for case 11, 21, 12
                                
                                    QUICKDouble tmp = 0.5*ABCD * VY( 0, 0, 1);
                                    LOC2(store, 1, 1, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 1, 0) + (tempx - Px) * VY( 0, 1, 1) + tmp;
                                    LOC2(store, 2, 1, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 1, 0) + (tempy - Py) * VY( 0, 1, 1);
                                    LOC2(store, 3, 1, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 1, 0) + (tempz - Pz) * VY( 0, 1, 1);
                                    
                                    LOC2(store, 1, 2, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 2, 0) + (tempx - Px) * VY( 0, 2, 1);
                                    LOC2(store, 2, 2, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 2, 0) + (tempy - Py) * VY( 0, 2, 1) + tmp;
                                    LOC2(store, 3, 2, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 2, 0) + (tempz - Pz) * VY( 0, 2, 1);
                                    
                                    LOC2(store, 1, 3, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 3, 0) + (tempx - Px) * VY( 0, 3, 1);
                                    LOC2(store, 2, 3, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 3, 0) + (tempy - Py) * VY( 0, 3, 1);
                                    LOC2(store, 3, 3, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 3, 0) + (tempz - Pz) * VY( 0, 3, 1) + tmp;
                                }
                                
                                //------------- FOR CASE I+J+K+L > 2 -----------------------------------
                                // includes NABCDTYPE = 12,21,22 situation
                                //----------------------------------------------------------------------
                                if ((I+J+K+L)>2) {//NABCDTYPE: 21, 12, 22
                                    //PSPS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, (Px*AB+Qx*CD)*ABCD - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                    
                                    QUICKDouble tmp = 0.5*ABCD * VY( 0, 0, 1);
                                    VY( 1, 1, 0) = (Px-RAx) * VY( 0, 1, 0) + (tempx - Px) * VY( 0, 1, 1) + tmp;
                                    VY( 2, 1, 0) = (Py-RAy) * VY( 0, 1, 0) + (tempy - Py) * VY( 0, 1, 1);
                                    VY( 3, 1, 0) = (Pz-RAz) * VY( 0, 1, 0) + (tempz - Pz) * VY( 0, 1, 1);
                                    
                                    VY( 1, 2, 0) = (Px-RAx) * VY( 0, 2, 0) + (tempx - Px) * VY( 0, 2, 1);
                                    VY( 2, 2, 0) = (Py-RAy) * VY( 0, 2, 0) + (tempy - Py) * VY( 0, 2, 1) + tmp;
                                    VY( 3, 2, 0) = (Pz-RAz) * VY( 0, 2, 0) + (tempz - Pz) * VY( 0, 2, 1);
                                    
                                    VY( 1, 3, 0) = (Px-RAx) * VY( 0, 3, 0) + (tempx - Px) * VY( 0, 3, 1);
                                    VY( 2, 3, 0) = (Py-RAy) * VY( 0, 3, 0) + (tempy - Py) * VY( 0, 3, 1);
                                    VY( 3, 3, 0) = (Pz-RAz) * VY( 0, 3, 0) + (tempz - Pz) * VY( 0, 3, 1) + tmp;
                                    
                                    if ((I+J)>1 && (K+L)>0) {
                                        //DSSS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5/AB, CD*ABCD);
                                        VY( 4, 0, 0) = (Px-RAx) * VY( 2, 0, 0) + (tempx - Px) * VY( 2, 0, 1);
                                        VY( 5, 0, 0) = (Py-RAy) * VY( 3, 0, 0) + (tempy - Py) * VY( 3, 0, 1);
                                        VY( 6, 0, 0) = (Px-RAx) * VY( 3, 0, 0) + (tempx - Px) * VY( 3, 0, 1);
                                        
                                        QUICKDouble tmp = 0.5 / AB *(VY( 0, 0, 0) - CD * ABCD * VY( 0, 0, 1));
                                        VY( 7, 0, 0) = (Px-RAx) * VY( 1, 0, 0) + (tempx - Px) * VY( 1, 0, 1) + tmp;
                                        VY( 8, 0, 0) = (Py-RAy) * VY( 2, 0, 0) + (tempy - Py) * VY( 2, 0, 1) + tmp;
                                        VY( 9, 0, 0) = (Pz-RAz) * VY( 3, 0, 0) + (tempz - Pz) * VY( 3, 0, 1) + tmp;
                                        
                                        
                                        //PSSS(2, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                        VY( 1, 0, 2) = (Px-RAx) * VY( 0, 0, 2) + (tempx - Px) * VY( 0, 0, 3);
                                        VY( 2, 0, 2) = (Py-RAy) * VY( 0, 0, 2) + (tempy - Py) * VY( 0, 0, 3);
                                        VY( 3, 0, 2) = (Pz-RAz) * VY( 0, 0, 2) + (tempz - Pz) * VY( 0, 0, 3);
                                        
                                        //DSSS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5/AB, CD*ABCD);
                                        VY( 4, 0, 1) = (Px-RAx) * VY( 2, 0, 1) + (tempx - Px) * VY( 2, 0, 2);
                                        VY( 5, 0, 1) = (Py-RAy) * VY( 3, 0, 1) + (tempy - Py) * VY( 3, 0, 2);
                                        VY( 6, 0, 1) = (Px-RAx) * VY( 3, 0, 1) + (tempx - Px) * VY( 3, 0, 2);
                                        
                                        tmp = 0.5 / AB *(VY( 0, 0, 1) - CD * ABCD * VY( 0, 0, 2));
                                        VY( 7, 0, 1) = (Px-RAx) * VY( 1, 0, 1) + (tempx - Px) * VY( 1, 0, 2) + tmp;
                                        VY( 8, 0, 1) = (Py-RAy) * VY( 2, 0, 1) + (tempy - Py) * VY( 2, 0, 2) + tmp;
                                        VY( 9, 0, 1) = (Pz-RAz) * VY( 3, 0, 1) + (tempz - Pz) * VY( 3, 0, 2) + tmp;
                                        
                                        LOC2(store, 4, 0, STOREDIM, STOREDIM) += VY( 4, 0, 0);
                                        LOC2(store, 5, 0, STOREDIM, STOREDIM) += VY( 5, 0, 0);
                                        LOC2(store, 6, 0, STOREDIM, STOREDIM) += VY( 6, 0, 0);
                                        LOC2(store, 7, 0, STOREDIM, STOREDIM) += VY( 7, 0, 0);
                                        LOC2(store, 8, 0, STOREDIM, STOREDIM) += VY( 8, 0, 0);
                                        LOC2(store, 9, 0, STOREDIM, STOREDIM) += VY( 9, 0, 0);
                                        
                                        tmp = 0.5 * ABCD * VY( 2, 0, 1);
                                        LOC2(store, 4, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 4, 0, 0) + (tempx - Qx) * VY( 4, 0, 1) + tmp;
                                        LOC2(store, 5, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 5, 0, 0) + (tempz - Qz) * VY( 5, 0, 1) + tmp;
                                        LOC2(store, 8, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 8, 0, 0) + (tempy - Qy) * VY( 8, 0, 1) + 2*tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 1, 0, 1);
                                        LOC2(store, 7, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 7, 0, 0) + (tempx - Qx) * VY( 7, 0, 1) + 2*tmp;
                                        LOC2(store, 4, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 4, 0, 0) + (tempy - Qy) * VY( 4, 0, 1) + tmp;
                                        LOC2(store, 6, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 6, 0, 0) + (tempz - Qz) * VY( 6, 0, 1) + tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 3, 0, 1);
                                        LOC2(store, 6, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 6, 0, 0) + (tempx - Qx) * VY( 6, 0, 1) + tmp;
                                        LOC2(store, 5, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 5, 0, 0) + (tempy - Qy) * VY( 5, 0, 1) + tmp;
                                        LOC2(store, 9, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 9, 0, 0) + (tempz - Qz) * VY( 9, 0, 1) + 2*tmp;
                                        
                                        LOC2(store, 5, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 5, 0, 0) + (tempx - Qx) * VY( 5, 0, 1);
                                        LOC2(store, 8, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 8, 0, 0) + (tempx - Qx) * VY( 8, 0, 1);
                                        LOC2(store, 9, 1, STOREDIM, STOREDIM) += (Qx-RCx) * VY( 9, 0, 0) + (tempx - Qx) * VY( 9, 0, 1);
                                        
                                        LOC2(store, 6, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 6, 0, 0) + (tempy - Qy) * VY( 6, 0, 1);
                                        LOC2(store, 7, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 7, 0, 0) + (tempy - Qy) * VY( 7, 0, 1);
                                        LOC2(store, 9, 2, STOREDIM, STOREDIM) += (Qy-RCy) * VY( 9, 0, 0) + (tempy - Qy) * VY( 9, 0, 1);
                                        
                                        LOC2(store, 4, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 4, 0, 0) + (tempz - Qz) * VY( 4, 0, 1);
                                        LOC2(store, 7, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 7, 0, 0) + (tempz - Qz) * VY( 7, 0, 1);
                                        LOC2(store, 8, 3, STOREDIM, STOREDIM) += (Qz-RCz) * VY( 8, 0, 0) + (tempz - Qz) * VY( 8, 0, 1);
                                           
                                    }
                                    
                                    if ((K+L)>1 && (I+J)>0) {
                                    
                                        //SSDS(0, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                        VY( 0, 4, 0) = (Qx-RCx) * VY( 0, 2, 0) + (tempx - Qx) * VY( 0, 2, 1);
                                        VY( 0, 5, 0) = (Qy-RCy) * VY( 0, 3, 0) + (tempy - Qy) * VY( 0, 3, 1);
                                        VY( 0, 6, 0) = (Qx-RCx) * VY( 0, 3, 0) + (tempx - Qx) * VY( 0, 3, 1);
                                        
                                        QUICKDouble tmp =  0.5 / CD *(VY( 0, 0, 0) - AB*ABCD * VY( 0, 0, 1));
                                        VY( 0, 7, 0) = (Qx-RCx) * VY( 0, 1, 0) + (tempx - Qx) * VY( 0, 1, 1) + tmp;
                                        VY( 0, 8, 0) = (Qy-RCy) * VY( 0, 2, 0) + (tempy - Qy) * VY( 0, 2, 1) + tmp;
                                        VY( 0, 9, 0) = (Qz-RCz) * VY( 0, 3, 0) + (tempz - Qz) * VY( 0, 3, 1) + tmp;                                
                                        
                                        //SSPS(2, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                        VY( 0, 1, 2) = (Qx-RCx) * VY( 0, 0, 2) + (tempx - Qx) * VY( 0, 0, 3);
                                        VY( 0, 2, 2) = (Qy-RCy) * VY( 0, 0, 2) + (tempy - Qy) * VY( 0, 0, 3);
                                        VY( 0, 3, 2) = (Qz-RCz) * VY( 0, 0, 2) + (tempz - Qz) * VY( 0, 0, 3);
                                        
                                        //SSDS(1, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                        VY( 0, 4, 1) = (Qx-RCx) * VY( 0, 2, 1) + (tempx - Qx) * VY( 0, 2, 2);
                                        VY( 0, 5, 1) = (Qy-RCy) * VY( 0, 3, 1) + (tempy - Qy) * VY( 0, 3, 2);
                                        VY( 0, 6, 1) = (Qx-RCx) * VY( 0, 3, 1) + (tempx - Qx) * VY( 0, 3, 2);
                                        
                                        tmp = 0.5 / CD *(VY( 0, 0, 1) - AB*ABCD * VY( 0, 0, 2)) ;
                                        VY( 0, 7, 1) = (Qx-RCx) * VY( 0, 1, 1) + (tempx - Qx) * VY( 0, 1, 2) + tmp;
                                        VY( 0, 8, 1) = (Qy-RCy) * VY( 0, 2, 1) + (tempy - Qy) * VY( 0, 2, 2) + tmp;
                                        VY( 0, 9, 1) = (Qz-RCz) * VY( 0, 3, 1) + (tempz - Qz) * VY( 0, 3, 2) + tmp;
                                        
                                        LOC2(store, 0, 4, STOREDIM, STOREDIM) += VY( 0, 4, 0);
                                        LOC2(store, 0, 5, STOREDIM, STOREDIM) += VY( 0, 5, 0);
                                        LOC2(store, 0, 6, STOREDIM, STOREDIM) += VY( 0, 6, 0);
                                        LOC2(store, 0, 7, STOREDIM, STOREDIM) += VY( 0, 7, 0);
                                        LOC2(store, 0, 8, STOREDIM, STOREDIM) += VY( 0, 8, 0);
                                        LOC2(store, 0, 9, STOREDIM, STOREDIM) += VY( 0, 9, 0);
                                        
                                        tmp = 0.5 * ABCD * VY( 0, 1, 1);
                                        LOC2(store, 2, 4, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 4, 0) + (tempy - Py) * VY( 0, 4, 1) + tmp;
                                        LOC2(store, 3, 6, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 6, 0) + (tempz - Pz) * VY( 0, 6, 1) + tmp;
                                        LOC2(store, 1, 7, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 7, 0) + (tempx - Px) * VY( 0, 7, 1) + 2*tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 0, 2, 1);
                                        LOC2(store, 1, 4, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 4, 0) + (tempx - Px) * VY( 0, 4, 1) + tmp;
                                        LOC2(store, 2, 8, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 8, 0) + (tempy - Py) * VY( 0, 8, 1) + 2*tmp;
                                        LOC2(store, 3, 5, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 5, 0) + (tempz - Pz) * VY( 0, 5, 1) + tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 0, 3, 1);
                                        LOC2(store, 1, 6, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 6, 0) + (tempx - Px) * VY( 0, 6, 1) + tmp;
                                        LOC2(store, 2, 5, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 5, 0) + (tempy - Py) * VY( 0, 5, 1) + tmp;
                                        LOC2(store, 3, 9, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 9, 0) + (tempz - Pz) * VY( 0, 9, 1) + 2*tmp;
                                    
                                        LOC2(store, 1, 5, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 5, 0) + (tempx - Px) * VY( 0, 5, 1);
                                        LOC2(store, 1, 8, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 8, 0) + (tempx - Px) * VY( 0, 8, 1);
                                        LOC2(store, 1, 9, STOREDIM, STOREDIM) += (Px-RAx) * VY( 0, 9, 0) + (tempx - Px) * VY( 0, 9, 1);
                                        
                                        LOC2(store, 2, 6, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 6, 0) + (tempy - Py) * VY( 0, 6, 1);
                                        LOC2(store, 2, 7, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 7, 0) + (tempy - Py) * VY( 0, 7, 1);
                                        LOC2(store, 2, 9, STOREDIM, STOREDIM) += (Py-RAy) * VY( 0, 9, 0) + (tempy - Py) * VY( 0, 9, 1);
                                        
                                        LOC2(store, 3, 4, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 4, 0) + (tempz - Pz) * VY( 0, 4, 1);
                                        LOC2(store, 3, 7, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 7, 0) + (tempz - Pz) * VY( 0, 7, 1);
                                        LOC2(store, 3, 8, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 0, 8, 0) + (tempz - Pz) * VY( 0, 8, 1);
                                        
                                    }
                                    
                                    if(NABCDTYPE == 22) {
                                        
                                        //PSDS(0, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                        QUICKDouble tmp = 0.5 * ABCD * VY( 0, 1, 1);
                                        VY( 2, 4, 0) = (Py-RAy) * VY( 0, 4, 0) + (tempy - Py) * VY( 0, 4, 1) + tmp;
                                        VY( 3, 6, 0) = (Pz-RAz) * VY( 0, 6, 0) + (tempz - Pz) * VY( 0, 6, 1) + tmp;
                                        VY( 1, 7, 0) = (Px-RAx) * VY( 0, 7, 0) + (tempx - Px) * VY( 0, 7, 1) + 2*tmp;
                                        tmp = 0.5 * ABCD * VY( 0, 2, 1);
                                        VY( 1, 4, 0) = (Px-RAx) * VY( 0, 4, 0) + (tempx - Px) * VY( 0, 4, 1) + tmp;
                                        VY( 3, 5, 0) = (Pz-RAz) * VY( 0, 5, 0) + (tempz - Pz) * VY( 0, 5, 1) + tmp;
                                        VY( 2, 8, 0) = (Py-RAy) * VY( 0, 8, 0) + (tempy - Py) * VY( 0, 8, 1) + 2*tmp;
                                        tmp = 0.5 * ABCD * VY( 0, 3, 1);
                                        VY( 2, 5, 0) = (Py-RAy) * VY( 0, 5, 0) + (tempy - Py) * VY( 0, 5, 1) + tmp;
                                        VY( 1, 6, 0) = (Px-RAx) * VY( 0, 6, 0) + (tempx - Px) * VY( 0, 6, 1) + tmp;
                                        VY( 3, 9, 0) = (Pz-RAz) * VY( 0, 9, 0) + (tempz - Pz) * VY( 0, 9, 1) + 2*tmp;
                                        
                                        VY( 3, 4, 0) = (Pz-RAz) * VY( 0, 4, 0) + (tempz - Pz) * VY( 0, 4, 1) ;
                                        VY( 1, 5, 0) = (Px-RAx) * VY( 0, 5, 0) + (tempx - Px) * VY( 0, 5, 1);
                                        VY( 2, 6, 0) = (Py-RAy) * VY( 0, 6, 0) + (tempy - Py) * VY( 0, 6, 1);
                                        VY( 2, 7, 0) = (Py-RAy) * VY( 0, 7, 0) + (tempy - Py) * VY( 0, 7, 1);
                                        VY( 3, 7, 0) = (Pz-RAz) * VY( 0, 7, 0) + (tempz - Pz) * VY( 0, 7, 1);
                                        VY( 1, 8, 0) = (Px-RAx) * VY( 0, 8, 0) + (tempx - Px) * VY( 0, 8, 1);
                                        VY( 3, 8, 0) = (Pz-RAz) * VY( 0, 8, 0) + (tempz - Pz) * VY( 0, 8, 1);
                                        VY( 1, 9, 0) = (Px-RAx) * VY( 0, 9, 0) + (tempx - Px) * VY( 0, 9, 1);
                                        VY( 2, 9, 0) = (Py-RAy) * VY( 0, 9, 0) + (tempy - Py) * VY( 0, 9, 1);
                                        
                                        //PSSS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz);
                                        VY( 1, 0, 1) = (Px-RAx) * VY( 0, 0, 1) + (tempx - Px) * VY( 0, 0, 2);
                                        VY( 2, 0, 1) = (Py-RAy) * VY( 0, 0, 1) + (tempy - Py) * VY( 0, 0, 2);
                                        VY( 3, 0, 1) = (Pz-RAz) * VY( 0, 0, 1) + (tempz - Pz) * VY( 0, 0, 2);
                                        
                                        //SSPS(3, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz);
                                        VY( 0, 1, 3) = (Qx-RCx) * VY( 0, 0, 3) + (tempx - Qx) * VY( 0, 0, 4);
                                        VY( 0, 2, 3) = (Qy-RCy) * VY( 0, 0, 3) + (tempy - Qy) * VY( 0, 0, 4);
                                        VY( 0, 3, 3) = (Qz-RCz) * VY( 0, 0, 3) + (tempz - Qz) * VY( 0, 0, 4);
                                        
                                        //SSDS(2, YVerticalTemp, Qx-RCx, Qy-RCy, Qz-RCz, tempx - Qx, tempy - Qy, tempz - Qz, 0.5/CD, AB*ABCD);
                                        VY( 0, 4, 2) = (Qx-RCx) * VY( 0, 2, 2) + (tempx - Qx) * VY( 0, 2, 3);
                                        VY( 0, 5, 2) = (Qy-RCy) * VY( 0, 3, 2) + (tempy - Qy) * VY( 0, 3, 3);
                                        VY( 0, 6, 2) = (Qx-RCx) * VY( 0, 3, 2) + (tempx - Qx) * VY( 0, 3, 3);
                                        
                                        tmp = 0.5 / CD *(VY( 0, 0, 2) - AB*ABCD * VY( 0, 0, 3)) ;
                                        VY( 0, 7, 2) = (Qx-RCx) * VY( 0, 1, 2) + (tempx - Qx) * VY( 0, 1, 3) + tmp;
                                        VY( 0, 8, 2) = (Qy-RCy) * VY( 0, 2, 2) + (tempy - Qy) * VY( 0, 2, 3) + tmp;
                                        VY( 0, 9, 2) = (Qz-RCz) * VY( 0, 3, 2) + (tempz - Qz) * VY( 0, 3, 3) + tmp;
                                        
                                        //PSDS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                        tmp = 0.5 * ABCD * VY( 0, 1, 2);
                                        VY( 2, 4, 1) = (Py-RAy) * VY( 0, 4, 1) + (tempy - Py) * VY( 0, 4, 2) + tmp;
                                        VY( 3, 6, 1) = (Pz-RAz) * VY( 0, 6, 1) + (tempz - Pz) * VY( 0, 6, 2) + tmp;
                                        VY( 1, 7, 1) = (Px-RAx) * VY( 0, 7, 1) + (tempx - Px) * VY( 0, 7, 2) + 2 * tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 0, 2, 2);
                                        VY( 2, 8, 1) = (Py-RAy) * VY( 0, 8, 1) + (tempy - Py) * VY( 0, 8, 2) + 2 * tmp;
                                        VY( 1, 4, 1) = (Px-RAx) * VY( 0, 4, 1) + (tempx - Px) * VY( 0, 4, 2) + tmp;
                                        VY( 3, 5, 1) = (Pz-RAz) * VY( 0, 5, 1) + (tempz - Pz) * VY( 0, 5, 2) + tmp;
                                        
                                        tmp = 0.5 * ABCD * VY( 0, 3, 2);
                                        VY( 2, 5, 1) = (Py-RAy) * VY( 0, 5, 1) + (tempy - Py) * VY( 0, 5, 2) + tmp;
                                        VY( 1, 6, 1) = (Px-RAx) * VY( 0, 6, 1) + (tempx - Px) * VY( 0, 6, 2) + tmp;
                                        VY( 3, 9, 1) = (Pz-RAz) * VY( 0, 9, 1) + (tempz - Pz) * VY( 0, 9, 2) + 2 * tmp;
                                        VY( 3, 4, 1) = (Pz-RAz) * VY( 0, 4, 1) + (tempz - Pz) * VY( 0, 4, 2) ;
                                        
                                        VY( 1, 5, 1) = (Px-RAx) * VY( 0, 5, 1) + (tempx - Px) * VY( 0, 5, 2);
                                        VY( 2, 6, 1) = (Py-RAy) * VY( 0, 6, 1) + (tempy - Py) * VY( 0, 6, 2);
                                        VY( 2, 7, 1) = (Py-RAy) * VY( 0, 7, 1) + (tempy - Py) * VY( 0, 7, 2);
                                        VY( 3, 7, 1) = (Pz-RAz) * VY( 0, 7, 1) + (tempz - Pz) * VY( 0, 7, 2);
                                        VY( 1, 8, 1) = (Px-RAx) * VY( 0, 8, 1) + (tempx - Px) * VY( 0, 8, 2);
                                        VY( 3, 8, 1) = (Pz-RAz) * VY( 0, 8, 1) + (tempz - Pz) * VY( 0, 8, 2);
                                        
                                        VY( 1, 9, 1) = (Px-RAx) * VY( 0, 9, 1) + (tempx - Px) * VY( 0, 9, 2);
                                        VY( 2, 9, 1) = (Py-RAy) * VY( 0, 9, 1) + (tempy - Py) * VY( 0, 9, 2);
                                        
                                        //PSPS(1, YVerticalTemp, Px-RAx, Py-RAy, Pz-RAz, tempx - Px, tempy - Py, tempz - Pz, 0.5*ABCD);
                                        VY( 1, 1, 1) = (Px-RAx) * VY( 0, 1, 1) + (tempx - Px) * VY( 0, 1, 2) + 0.5*ABCD * VY( 0, 0, 2);
                                        VY( 2, 1, 1) = (Py-RAy) * VY( 0, 1, 1) + (tempy - Py) * VY( 0, 1, 2);
                                        VY( 3, 1, 1) = (Pz-RAz) * VY( 0, 1, 1) + (tempz - Pz) * VY( 0, 1, 2);
                                        
                                        VY( 1, 2, 1) = (Px-RAx) * VY( 0, 2, 1) + (tempx - Px) * VY( 0, 2, 2);
                                        VY( 2, 2, 1) = (Py-RAy) * VY( 0, 2, 1) + (tempy - Py) * VY( 0, 2, 2) + 0.5*ABCD * VY( 0, 0, 2);
                                        VY( 3, 2, 1) = (Pz-RAz) * VY( 0, 2, 1) + (tempz - Pz) * VY( 0, 2, 2);
                                        
                                        VY( 1, 3, 1) = (Px-RAx) * VY( 0, 3, 1) + (tempx - Px) * VY( 0, 3, 2);
                                        VY( 2, 3, 1) = (Py-RAy) * VY( 0, 3, 1) + (tempy - Py) * VY( 0, 3, 2);
                                        VY( 3, 3, 1) = (Pz-RAz) * VY( 0, 3, 1) + (tempz - Pz) * VY( 0, 3, 2) + 0.5*ABCD * VY( 0, 0, 2);
                                        
                                        LOC2(store, 4, 4, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 4, 0) + (tempx - Px) * VY( 2, 4, 1) + 0.5 * ABCD * VY( 2, 2, 1);
                                        LOC2(store, 4, 5, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 5, 0) + (tempx - Px) * VY( 2, 5, 1);
                                        LOC2(store, 4, 6, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 6, 0) + (tempx - Px) * VY( 2, 6, 1) + 0.5 * ABCD * VY( 2, 3, 1);
                                        LOC2(store, 4, 7, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 7, 0) + (tempx - Px) * VY( 2, 7, 1) + ABCD * VY( 2, 1, 1);
                                        LOC2(store, 4, 8, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 8, 0) + (tempx - Px) * VY( 2, 8, 1);
                                        LOC2(store, 4, 9, STOREDIM, STOREDIM) += (Px-RAx) * VY( 2, 9, 0) + (tempx - Px) * VY( 2, 9, 1);
                                        
                                        LOC2(store, 5, 4, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 4, 0) + (tempy - Py) * VY( 3, 4, 1) + 0.5 * ABCD * VY( 3, 1, 1);
                                        LOC2(store, 5, 5, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 5, 0) + (tempy - Py) * VY( 3, 5, 1) + 0.5 * ABCD * VY( 3, 3, 1);
                                        LOC2(store, 5, 6, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 6, 0) + (tempy - Py) * VY( 3, 6, 1);
                                        LOC2(store, 5, 7, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 7, 0) + (tempy - Py) * VY( 3, 7, 1);
                                        LOC2(store, 5, 8, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 8, 0) + (tempy - Py) * VY( 3, 8, 1) + ABCD * VY( 3, 2, 1);
                                        LOC2(store, 5, 9, STOREDIM, STOREDIM) += (Py-RAy) * VY( 3, 9, 0) + (tempy - Py) * VY( 3, 9, 1);
                                        
                                        LOC2(store, 6, 4, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 4, 0) + (tempx - Px) * VY( 3, 4, 1) + 0.5 * ABCD * VY( 3, 2, 1);
                                        LOC2(store, 6, 5, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 5, 0) + (tempx - Px) * VY( 3, 5, 1);
                                        LOC2(store, 6, 6, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 6, 0) + (tempx - Px) * VY( 3, 6, 1) + 0.5 * ABCD * VY( 3, 3, 1);
                                        LOC2(store, 6, 7, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 7, 0) + (tempx - Px) * VY( 3, 7, 1) + ABCD * VY( 3, 1, 1);
                                        LOC2(store, 6, 8, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 8, 0) + (tempx - Px) * VY( 3, 8, 1);
                                        LOC2(store, 6, 9, STOREDIM, STOREDIM) += (Px-RAx) * VY( 3, 9, 0) + (tempx - Px) * VY( 3, 9, 1);
                                        
                                        tmp = 0.5/AB;
                                        VY( 0, 4,1) = tmp * (VY( 0, 4,0)-CD * ABCD*VY( 0, 4, 1));
                                        VY( 0, 5,1) = tmp * (VY( 0, 5,0)-CD * ABCD*VY( 0, 5, 1));
                                        VY( 0, 6,1) = tmp * (VY( 0, 6,0)-CD * ABCD*VY( 0, 6, 1));
                                        VY( 0, 7,1) = tmp * (VY( 0, 7,0)-CD * ABCD*VY( 0, 7, 1));
                                        VY( 0, 8,1) = tmp * (VY( 0, 8,0)-CD * ABCD*VY( 0, 8, 1));
                                        VY( 0, 9,1) = tmp * (VY( 0, 9,0)-CD * ABCD*VY( 0, 9, 1));
                                        
                                        LOC2(store, 7, 4, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 4, 0) + (tempx - Px) * VY( 1, 4, 1) +  VY( 0, 4, 1) + 0.5 * ABCD * VY( 1, 2, 1);
                                        LOC2(store, 7, 5, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 5, 0) + (tempx - Px) * VY( 1, 5, 1) +  VY( 0, 5, 1) ;
                                        LOC2(store, 7, 6, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 6, 0) + (tempx - Px) * VY( 1, 6, 1) +  VY( 0, 6, 1) + 0.5 * ABCD * VY( 1, 3, 1);
                                        LOC2(store, 7, 7, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 7, 0) + (tempx - Px) * VY( 1, 7, 1) +  VY( 0, 7, 1) + ABCD * VY( 1, 1, 1);
                                        LOC2(store, 7, 8, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 8, 0) + (tempx - Px) * VY( 1, 8, 1) +  VY( 0, 8, 1) ;
                                        LOC2(store, 7, 9, STOREDIM, STOREDIM) += (Px-RAx) * VY( 1, 9, 0) + (tempx - Px) * VY( 1, 9, 1) +  VY( 0, 9, 1) ;
                                        
                                        LOC2(store, 8, 4, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 4, 0) + (tempy - Py) * VY( 2, 4, 1) +  VY( 0, 4, 1) + 0.5 * ABCD * VY( 2, 1, 1);
                                        LOC2(store, 8, 5, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 5, 0) + (tempy - Py) * VY( 2, 5, 1) +  VY( 0, 5, 1)  + 0.5 * ABCD * VY( 2, 3, 1);
                                        LOC2(store, 8, 6, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 6, 0) + (tempy - Py) * VY( 2, 6, 1) +  VY( 0, 6, 1) ;
                                        LOC2(store, 8, 7, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 7, 0) + (tempy - Py) * VY( 2, 7, 1) +  VY( 0, 7, 1) ;
                                        LOC2(store, 8, 8, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 8, 0) + (tempy - Py) * VY( 2, 8, 1) +  VY( 0, 8, 1) + ABCD * VY( 2, 2, 1);
                                        LOC2(store, 8, 9, STOREDIM, STOREDIM) += (Py-RAy) * VY( 2, 9, 0) + (tempy - Py) * VY( 2, 9, 1) +  VY( 0, 9, 1) ;
                                        
                                        
                                        LOC2(store, 9, 4, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 4, 0) + (tempz - Pz) * VY( 3, 4, 1) +  VY( 0, 4, 1);
                                        LOC2(store, 9, 5, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 5, 0) + (tempz - Pz) * VY( 3, 5, 1) +  VY( 0, 5, 1) + 0.5 * ABCD * VY( 3, 2, 1);
                                        LOC2(store, 9, 6, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 6, 0) + (tempz - Pz) * VY( 3, 6, 1) +  VY( 0, 6, 1) + 0.5 * ABCD * VY( 3, 1, 1);
                                        LOC2(store, 9, 7, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 7, 0) + (tempz - Pz) * VY( 3, 7, 1) +  VY( 0, 7, 1) ;
                                        LOC2(store, 9, 8, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 8, 0) + (tempz - Pz) * VY( 3, 8, 1) +  VY( 0, 8, 1) ;
                                        LOC2(store, 9, 9, STOREDIM, STOREDIM) += (Pz-RAz) * VY( 3, 9, 0) + (tempz - Pz) * VY( 3, 9, 1) +  VY( 0, 9, 1) + ABCD * VY( 3, 3, 1);
                                    }
                                    
                                }
                            }
                        }
                    }   
                }
            }
        }
    }

    
    // IJKLTYPE is the I, J, K,L type
    int IJKLTYPE = (int) (1000 * I + 100 *J + 10 * K + L);
    
    RBx = LOC2(devSim.xyz, 0 , devSim.katom[JJ]-1, 3, devSim.natom);
    RBy = LOC2(devSim.xyz, 1 , devSim.katom[JJ]-1, 3, devSim.natom);
    RBz = LOC2(devSim.xyz, 2 , devSim.katom[JJ]-1, 3, devSim.natom);
    
    
    RDx = LOC2(devSim.xyz, 0 , devSim.katom[LL]-1, 3, devSim.natom);
    RDy = LOC2(devSim.xyz, 1 , devSim.katom[LL]-1, 3, devSim.natom);
    RDz = LOC2(devSim.xyz, 2 , devSim.katom[LL]-1, 3, devSim.natom);
        
    int III1 = LOC2(devSim.Qsbasis, II, I, nshell, 4);
    int III2 = LOC2(devSim.Qfbasis, II, I, nshell, 4);
    int JJJ1 = LOC2(devSim.Qsbasis, JJ, J, nshell, 4);
    int JJJ2 = LOC2(devSim.Qfbasis, JJ, J, nshell, 4);
    int KKK1 = LOC2(devSim.Qsbasis, KK, K, nshell, 4);
    int KKK2 = LOC2(devSim.Qfbasis, KK, K, nshell, 4);
    int LLL1 = LOC2(devSim.Qsbasis, LL, L, nshell, 4);
    int LLL2 = LOC2(devSim.Qfbasis, LL, L, nshell, 4);
    
    
    // maxIJKL is the max of I,J,K,L
    int maxIJKL = (int)MAX(MAX(I,J),MAX(K,L));
    
    if (((maxIJKL == 2)&&(J != 0 || L!=0)) || (maxIJKL >= 3)) {
        IJKLTYPE = 999;
    }
    
    if ((II < JJ) && (II < KK) && (KK < LL)) {
        for (int III = III1; III <= III2; III++) {
            for (int JJJ = JJJ1; JJJ <= JJJ2; JJJ++) {
                for (int KKK = KKK1; KKK <= KKK2 ; KKK++) {
                    for (int LLL = LLL1; LLL <= LLL2; LLL++) {
                        QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);
                        QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELK = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);

                        // Find the (ij|kl) integrals where j>i, k>i, l>k, and k and j are equal.
                        QUICKULL val1 = (QUICKULL) (fabs(2.0*DENSELK*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELK*Y < (QUICKDouble)0.0)
                        val1 = 0ull - val1;
                        
                        QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEJI*Y < (QUICKDouble)0.0)
                        val2 = 0ull - val2;
                        
                        QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSELJ*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELJ*Y < (QUICKDouble)0.0)
                        val3 = 0ull - val3;
                        
                        QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEKJ*Y < (QUICKDouble)0.0)
                        val4 = 0ull - val4;
                        
                        QUICKULL val5 = (QUICKULL) (fabs(0.5*DENSELI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELI*Y < (QUICKDouble)0.0)
                        val5 = 0ull - val5;
                        
                        QUICKULL val6 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEKI*Y < (QUICKDouble)0.0)
                        val6 = 0ull - val6;
                        
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                        QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                        QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, LLL-1, devSim.nbasis, devSim.nbasis), 0ull-val6);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val6);               
                    }
                }
            }
        }
    }else {
        for (int III = III1; III <= III2; III++) {
            for (int JJJ = MAX(III,JJJ1); JJJ <= JJJ2; JJJ++) {
                for (int KKK = MAX(III,KKK1); KKK <= KKK2; KKK++) {
                    for (int LLL = MAX(KKK,LLL1); LLL <= LLL2; LLL++) {
                        if (III < KKK) {
                        QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);

                            if ((III < JJJ)&&(KKK < LLL)) {
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELK = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                // Find the (ij|kl) integrals where j>i, k>i, l>k, and k and j are equal.
                                QUICKULL val1 = (QUICKULL) (fabs(2.0*DENSELK*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELK*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;

                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSELJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELJ*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;
                                
                                QUICKULL val5 = (QUICKULL) (fabs(0.5*DENSELI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELI*Y < (QUICKDouble)0.0)
                                val5 = 0ull - val5;
                                
                                QUICKULL val6 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val6 = 0ull - val6;
                                
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, LLL-1, devSim.nbasis, devSim.nbasis), 0ull-val6);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val6);                              
                                
                            }else if ((III == JJJ)&&(KKK == LLL)) {
                                
                                // Find  all the (ii|jj) integrals.
                                QUICKDouble DENSEJI = LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJJ = LOC2(devSim.dense, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEII = LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);

                                QUICKULL val1 = (QUICKULL) (fabs(DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEII*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);

                            }else if ((JJJ == KKK)&&(JJJ==LLL)) {

                                // Find all the (ij|jj) integrals.
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJJ = (QUICKDouble) LOC2(devSim.dense, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis);

                                QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis), val2);
                            
                            }else if ((KKK == LLL)&&(III<JJJ)&&(JJJ!=KKK)) {
                                
                                //Find all the (ij|kk) integrals where j>i, k>j.
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKK = (QUICKDouble) LOC2(devSim.dense, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
 
                                QUICKULL val1 = (QUICKULL) (fabs(DENSEKK*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKK*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;

                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val4);

                            }else if ((III==JJJ)&&(KKK<LLL)) {
                                
                                //Find all the (ii|jk) integrals where j>i, k>j.
                                QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                
                                QUICKULL val1 = (QUICKULL) (fabs(DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEII*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;

                                QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                            } 
                            
                        }else {
                        
                            if (JJJ <= LLL) {
                                QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);

                                if((III==JJJ)&&(III==KKK)&&(III==LLL)){
                                    // do all the (ii|ii) integrals
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                }else if ((III == JJJ) && (III == KKK) && (III < LLL)){
                                    
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                                                      

                                    QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val2);

                                }else if ((III == KKK) && (JJJ == LLL) && (III < JJJ)){
                                    
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEJJ = (QUICKDouble) LOC2(devSim.dense, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKULL val1 = (QUICKULL) (fabs(1.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                    val3 = 0ull - val3;                               
                                    
                                    
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val2);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                }else if ((III == KKK) && (III <  JJJ) && (JJJ < LLL)){
                                    
                                    QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                   
                                    QUICKULL val1 = (QUICKULL) (fabs(1.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEKI*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(1.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKULL val3 = (QUICKULL) (fabs(1.0*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                    val3 = 0ull - val3;                               
                                    
                                    QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val4 = 0ull - val4;
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), val2);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
								}
							}
                        }
                    }
                }
            }
        }
    }
	return;
}


#ifndef TEST
__device__
#endif
  void FmT(int MaxM, QUICKDouble X, QUICKDouble* YVerticalTemp)
{
    const QUICKDouble PIE4 = (QUICKDouble) PI/4.0 ;
    const QUICKDouble XINV = (QUICKDouble) 1.0 /X;
    const QUICKDouble E = (QUICKDouble) exp(-X);
    QUICKDouble WW1;
    
    if (X > 5.0 ) {
        if (X>15.0 ) {
            if (X>33.0 ) {
                WW1 = sqrt(PIE4 * XINV);
            }else {
                WW1 = (( 1.9623264149430E-01 *XINV-4.9695241464490E-01 )*XINV - \
                       6.0156581186481E-05 )*E + sqrt(PIE4*XINV);
            }
        }else if (X>10.0 ) {
            WW1 = (((-1.8784686463512E-01 *XINV+2.2991849164985E-01 )*XINV - \
                    4.9893752514047E-01 )*XINV-2.1916512131607E-05 )*E + sqrt(PIE4*XINV);
        }else {
            WW1 = (((((( 4.6897511375022E-01  *XINV-6.9955602298985E-01 )*XINV + \
                       5.3689283271887E-01 )*XINV-3.2883030418398E-01 )*XINV + \
                     2.4645596956002E-01 )*XINV-4.9984072848436E-01 )*XINV - \
                   3.1501078774085E-06 )*E + sqrt(PIE4*XINV);
        }
    }else if (X >1.0 ) {
        if (X>3.0 ) {
            QUICKDouble Y = (QUICKDouble) X - 4.0 ;
            QUICKDouble F1 = ((((((((((-2.62453564772299E-11 *Y+3.24031041623823E-10  )*Y- \
                                      3.614965656163E-09 )*Y+3.760256799971E-08 )*Y- \
                                    3.553558319675E-07 )*Y+3.022556449731E-06 )*Y- \
                                  2.290098979647E-05 )*Y+1.526537461148E-04 )*Y- \
                                8.81947375894379E-04 )*Y+4.33207949514611E-03 )*Y- \
                              1.75257821619926E-02 )*Y+5.28406320615584E-02 ;
            WW1 = (X+X)*F1+E;
        }else {
            QUICKDouble Y = (QUICKDouble) X - 2.0 ;
            QUICKDouble F1 = ((((((((((-1.61702782425558E-10 *Y+1.96215250865776E-09  )*Y- \
                                      2.14234468198419E-08  )*Y+2.17216556336318E-07  )*Y- \
                                    1.98850171329371E-06  )*Y+1.62429321438911E-05  )*Y- \
                                  1.16740298039895E-04  )*Y+7.24888732052332E-04  )*Y- \
                                3.79490003707156E-03  )*Y+1.61723488664661E-02  )*Y- \
                              5.29428148329736E-02  )*Y+1.15702180856167E-01 ;
            WW1 = (X+X)*F1+E;
        }
        
    }else if (X > 3.0E-7 ) {
        QUICKDouble F1 =(((((((( -8.36313918003957E-08 *X+1.21222603512827E-06  )*X- \
                               1.15662609053481E-05  )*X+9.25197374512647E-05  )*X- \
                             6.40994113129432E-04  )*X+3.78787044215009E-03  )*X- \
                           1.85185172458485E-02  )*X+7.14285713298222E-02  )*X- \
                         1.99999999997023E-01  )*X+3.33333333333318E-01 ;
        WW1 = (X+X)*F1+E;
    }else {
        WW1 = (1.0 -X)/(QUICKDouble)(2.0 * MaxM+1);
    }
    
    if (X > 3.0E-7 ) {
        LOC3(YVerticalTemp, 0, 0, 0, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = 1; m<= MaxM; m++) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (((2*m-1)*LOC3(YVerticalTemp, 0, 0, m-1, VDIM1, VDIM2, VDIM3))- E)*0.5*XINV;
        }
    }else {
        LOC3(YVerticalTemp, 0, 0, MaxM, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = MaxM-1; m >=0; m--) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (2.0 * X * LOC3(YVerticalTemp, 0, 0, m+1, VDIM1, VDIM2, VDIM3) + E) / (QUICKDouble)(m*2+1);
        }
    }
	return;
}


#ifndef TEST
__device__
#endif
void PSSS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz)
{
    VY( 1, 0, mtemp) = Ptempx * VY( 0, 0, mtemp) + WPtempx * VY( 0, 0, mtemp+1);
    VY( 2, 0, mtemp) = Ptempy * VY( 0, 0, mtemp) + WPtempy * VY( 0, 0, mtemp+1);
    VY( 3, 0, mtemp) = Ptempz * VY( 0, 0, mtemp) + WPtempz * VY( 0, 0, mtemp+1);
	return;
}

#ifndef TEST
__device__
#endif
void SSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz)
{
    
    VY( 0, 1, mtemp) = Qtempx * VY( 0, 0, mtemp) + WQtempx * VY( 0, 0, mtemp+1);
    VY( 0, 2, mtemp) = Qtempy * VY( 0, 0, mtemp) + WQtempy * VY( 0, 0, mtemp+1);
    VY( 0, 3, mtemp) = Qtempz * VY( 0, 0, mtemp) + WQtempz * VY( 0, 0, mtemp+1);
    
	return;
}

#ifndef TEST
__device__
#endif
void PSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp)
{
    
    VY( 1, 1, mtemp) = Ptempx * VY( 0, 1, mtemp) + WPtempx * VY( 0, 1, mtemp+1) + ABCDtemp * VY( 0, 0, mtemp+1);
    VY( 2, 1, mtemp) = Ptempy * VY( 0, 1, mtemp) + WPtempy * VY( 0, 1, mtemp+1);
    VY( 3, 1, mtemp) = Ptempz * VY( 0, 1, mtemp) + WPtempz * VY( 0, 1, mtemp+1);
    
    VY( 1, 2, mtemp) = Ptempx * VY( 0, 2, mtemp) + WPtempx * VY( 0, 2, mtemp+1);
    VY( 2, 2, mtemp) = Ptempy * VY( 0, 2, mtemp) + WPtempy * VY( 0, 2, mtemp+1) + ABCDtemp * VY( 0, 0, mtemp+1);
    VY( 3, 2, mtemp) = Ptempz * VY( 0, 2, mtemp) + WPtempz * VY( 0, 2, mtemp+1);
    
    VY( 1, 3, mtemp) = Ptempx * VY( 0, 3, mtemp) + WPtempx * VY( 0, 3, mtemp+1);
    VY( 2, 3, mtemp) = Ptempy * VY( 0, 3, mtemp) + WPtempy * VY( 0, 3, mtemp+1);
    VY( 3, 3, mtemp) = Ptempz * VY( 0, 3, mtemp) + WPtempz * VY( 0, 3, mtemp+1) + ABCDtemp * VY( 0, 0, mtemp+1);
    
	return;
}

#ifndef TEST
__device__
#endif
void DSSS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABtemp, QUICKDouble CDcom)
{
    VY( 4, 0, mtemp) = Ptempx * VY( 2, 0, mtemp) + WPtempx * VY( 2, 0, mtemp+1);
    VY( 5, 0, mtemp) = Ptempy * VY( 3, 0, mtemp) + WPtempy * VY( 3, 0, mtemp+1);
    VY( 6, 0, mtemp) = Ptempx * VY( 3, 0, mtemp) + WPtempx * VY( 3, 0, mtemp+1);
    
    VY( 7, 0, mtemp) = Ptempx * VY( 1, 0, mtemp) + WPtempx * VY( 1, 0, mtemp+1)+ ABtemp*(VY( 0, 0, mtemp) - CDcom * VY( 0, 0, mtemp+1));
    VY( 8, 0, mtemp) = Ptempy * VY( 2, 0, mtemp) + WPtempy * VY( 2, 0, mtemp+1)+ ABtemp*(VY( 0, 0, mtemp) - CDcom * VY( 0, 0, mtemp+1));
    VY( 9, 0, mtemp) = Ptempz * VY( 3, 0, mtemp) + WPtempz * VY( 3, 0, mtemp+1)+ ABtemp*(VY( 0, 0, mtemp) - CDcom * VY( 0, 0, mtemp+1));
    
	return;
}

#ifndef TEST
__device__
#endif
void SSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz, QUICKDouble CDtemp, QUICKDouble ABcom)
{
    VY( 0, 4, mtemp) = Qtempx * VY( 0, 2, mtemp) + WQtempx * VY( 0, 2, mtemp+1);
    VY( 0, 5, mtemp) = Qtempy * VY( 0, 3, mtemp) + WQtempy * VY( 0, 3, mtemp+1);
    VY( 0, 6, mtemp) = Qtempx * VY( 0, 3, mtemp) + WQtempx * VY( 0, 3, mtemp+1);
    
    VY( 0, 7, mtemp) = Qtempx * VY( 0, 1, mtemp) + WQtempx * VY( 0, 1, mtemp+1)+ CDtemp*(VY( 0, 0, mtemp) - ABcom * VY( 0, 0, mtemp+1));
    VY( 0, 8, mtemp) = Qtempy * VY( 0, 2, mtemp) + WQtempy * VY( 0, 2, mtemp+1)+ CDtemp*(VY( 0, 0, mtemp) - ABcom * VY( 0, 0, mtemp+1));
    VY( 0, 9, mtemp) = Qtempz * VY( 0, 3, mtemp) + WQtempz * VY( 0, 3, mtemp+1)+ CDtemp*(VY( 0, 0, mtemp) - ABcom * VY( 0, 0, mtemp+1));
    
	return;
}



#ifndef TEST
__device__
#endif
void DSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz, QUICKDouble ABCDtemp)
{
    
    VY( 4, 1, mtemp) = Qtempx * VY( 4, 0, mtemp) + WQtempx * VY( 4, 0, mtemp + 1) + ABCDtemp * VY( 2, 0, mtemp + 1);
    VY( 4, 2, mtemp) = Qtempy * VY( 4, 0, mtemp) + WQtempy * VY( 4, 0, mtemp + 1) + ABCDtemp * VY( 1, 0, mtemp + 1);
    VY( 4, 3, mtemp) = Qtempz * VY( 4, 0, mtemp) + WQtempz * VY( 4, 0, mtemp + 1);
    
    VY( 5, 1, mtemp) = Qtempx * VY( 5, 0, mtemp) + WQtempx * VY( 5, 0, mtemp + 1);
    VY( 5, 2, mtemp) = Qtempy * VY( 5, 0, mtemp) + WQtempy * VY( 5, 0, mtemp + 1) + ABCDtemp * VY( 3, 0, mtemp + 1);
    VY( 5, 3, mtemp) = Qtempz * VY( 5, 0, mtemp) + WQtempz * VY( 5, 0, mtemp + 1) + ABCDtemp * VY( 2, 0, mtemp + 1);
    
    VY( 6, 1, mtemp) = Qtempx * VY( 6, 0, mtemp) + WQtempx * VY( 6, 0, mtemp + 1) + ABCDtemp * VY( 3, 0, mtemp + 1);
    VY( 6, 2, mtemp) = Qtempy * VY( 6, 0, mtemp) + WQtempy * VY( 6, 0, mtemp + 1);
    VY( 6, 3, mtemp) = Qtempz * VY( 6, 0, mtemp) + WQtempz * VY( 6, 0, mtemp + 1) + ABCDtemp * VY( 1, 0, mtemp + 1);
    
    VY( 7, 1, mtemp) = Qtempx * VY( 7, 0, mtemp) + WQtempx * VY( 7, 0, mtemp + 1) + ABCDtemp * VY( 1, 0, mtemp + 1) * 2;
    VY( 7, 2, mtemp) = Qtempy * VY( 7, 0, mtemp) + WQtempy * VY( 7, 0, mtemp + 1);
    VY( 7, 3, mtemp) = Qtempz * VY( 7, 0, mtemp) + WQtempz * VY( 7, 0, mtemp + 1);
    
    VY( 8, 1, mtemp) = Qtempx * VY( 8, 0, mtemp) + WQtempx * VY( 8, 0, mtemp + 1);
    VY( 8, 2, mtemp) = Qtempy * VY( 8, 0, mtemp) + WQtempy * VY( 8, 0, mtemp + 1) + ABCDtemp * VY( 2, 0, mtemp + 1) * 2;
    VY( 8, 3, mtemp) = Qtempz * VY( 8, 0, mtemp) + WQtempz * VY( 8, 0, mtemp + 1);
    
    VY( 9, 1, mtemp) = Qtempx * VY( 9, 0, mtemp) + WQtempx * VY( 9, 0, mtemp + 1);
    VY( 9, 2, mtemp) = Qtempy * VY( 9, 0, mtemp) + WQtempy * VY( 9, 0, mtemp + 1);
    VY( 9, 3, mtemp) = Qtempz * VY( 9, 0, mtemp) + WQtempz * VY( 9, 0, mtemp + 1) + ABCDtemp * VY( 3, 0, mtemp + 1) * 2;            
    
	return;
}

#ifndef TEST
__device__
#endif
void PSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp)
{
    
    
    VY( 1, 4, mtemp) = Ptempx * VY( 0, 4, mtemp) + WPtempx * VY( 0, 4, mtemp + 1) + ABCDtemp * VY( 0, 2, mtemp + 1);
    VY( 2, 4, mtemp) = Ptempy * VY( 0, 4, mtemp) + WPtempy * VY( 0, 4, mtemp + 1) + ABCDtemp * VY( 0, 1, mtemp + 1);
    VY( 3, 4, mtemp) = Ptempz * VY( 0, 4, mtemp) + WPtempz * VY( 0, 4, mtemp + 1);
    
    VY( 1, 5, mtemp) = Ptempx * VY( 0, 5, mtemp) + WPtempx * VY( 0, 5, mtemp + 1);
    VY( 2, 5, mtemp) = Ptempy * VY( 0, 5, mtemp) + WPtempy * VY( 0, 5, mtemp + 1) + ABCDtemp * VY( 0, 3, mtemp + 1);
    VY( 3, 5, mtemp) = Ptempz * VY( 0, 5, mtemp) + WPtempz * VY( 0, 5, mtemp + 1) + ABCDtemp * VY( 0, 2, mtemp + 1);
    
    VY( 1, 6, mtemp) = Ptempx * VY( 0, 6, mtemp) + WPtempx * VY( 0, 6, mtemp + 1) + ABCDtemp * VY( 0, 3, mtemp + 1);
    VY( 2, 6, mtemp) = Ptempy * VY( 0, 6, mtemp) + WPtempy * VY( 0, 6, mtemp + 1);
    VY( 3, 6, mtemp) = Ptempz * VY( 0, 6, mtemp) + WPtempz * VY( 0, 6, mtemp + 1) + ABCDtemp * VY( 0, 1, mtemp + 1);
    
    VY( 1, 7, mtemp) = Ptempx * VY( 0, 7, mtemp) + WPtempx * VY( 0, 7, mtemp + 1) + ABCDtemp * VY( 0, 1, mtemp + 1) * 2;
    VY( 2, 7, mtemp) = Ptempy * VY( 0, 7, mtemp) + WPtempy * VY( 0, 7, mtemp + 1);
    VY( 3, 7, mtemp) = Ptempz * VY( 0, 7, mtemp) + WPtempz * VY( 0, 7, mtemp + 1);
    
    VY( 1, 8, mtemp) = Ptempx * VY( 0, 8, mtemp) + WPtempx * VY( 0, 8, mtemp + 1);
    VY( 2, 8, mtemp) = Ptempy * VY( 0, 8, mtemp) + WPtempy * VY( 0, 8, mtemp + 1) + ABCDtemp * VY( 0, 2, mtemp + 1) * 2;
    VY( 3, 8, mtemp) = Ptempz * VY( 0, 8, mtemp) + WPtempz * VY( 0, 8, mtemp + 1);
    
    VY( 1, 9, mtemp) = Ptempx * VY( 0, 9, mtemp) + WPtempx * VY( 0, 9, mtemp + 1);
    VY( 2, 9, mtemp) = Ptempy * VY( 0, 9, mtemp) + WPtempy * VY( 0, 9, mtemp + 1);
    VY( 3, 9, mtemp) = Ptempz * VY( 0, 9, mtemp) + WPtempz * VY( 0, 9, mtemp + 1) + ABCDtemp * VY( 0, 3, mtemp + 1) * 2;    
    
	return;
}

#ifndef TEST
__device__
#endif
void DSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp, QUICKDouble ABtemp, QUICKDouble CDcom)
{
    
    VY( 4, 4, mtemp) = Ptempx * VY( 2, 4, mtemp) + WPtempx * VY( 2, 4, mtemp+1) + ABCDtemp * VY( 2, 2, mtemp+1);
    VY( 4, 5, mtemp) = Ptempx * VY( 2, 5, mtemp) + WPtempx * VY( 2, 5, mtemp+1);
    VY( 4, 6, mtemp) = Ptempx * VY( 2, 6, mtemp) + WPtempx * VY( 2, 6, mtemp+1) + ABCDtemp * VY( 2, 3, mtemp+1);
    VY( 4, 7, mtemp) = Ptempx * VY( 2, 7, mtemp) + WPtempx * VY( 2, 7, mtemp+1) + 2 * ABCDtemp * VY( 2, 1, mtemp+1);
    VY( 4, 8, mtemp) = Ptempx * VY( 2, 8, mtemp) + WPtempx * VY( 2, 8, mtemp+1);
    VY( 4, 9, mtemp) = Ptempx * VY( 2, 9, mtemp) + WPtempx * VY( 2, 9, mtemp+1);
    
    VY( 5, 4, mtemp) = Ptempy * VY( 3, 4, mtemp) + WPtempy * VY( 3, 4, mtemp+1) + ABCDtemp * VY( 3, 1, mtemp+1);
    VY( 5, 5, mtemp) = Ptempy * VY( 3, 5, mtemp) + WPtempy * VY( 3, 5, mtemp+1) + ABCDtemp * VY( 3, 3, mtemp+1);
    VY( 5, 6, mtemp) = Ptempy * VY( 3, 6, mtemp) + WPtempy * VY( 3, 6, mtemp+1);
    VY( 5, 7, mtemp) = Ptempy * VY( 3, 7, mtemp) + WPtempy * VY( 3, 7, mtemp+1);
    VY( 5, 8, mtemp) = Ptempy * VY( 3, 8, mtemp) + WPtempy * VY( 3, 8, mtemp+1) + 2 * ABCDtemp * VY( 3, 2, mtemp+1);
    VY( 5, 9, mtemp) = Ptempy * VY( 3, 9, mtemp) + WPtempy * VY( 3, 9, mtemp+1);
    
    VY( 6, 4, mtemp) = Ptempx * VY( 3, 4, mtemp) + WPtempx * VY( 3, 4, mtemp+1) + ABCDtemp * VY( 3, 2, mtemp+1);
    VY( 6, 5, mtemp) = Ptempx * VY( 3, 5, mtemp) + WPtempx * VY( 3, 5, mtemp+1);
    VY( 6, 6, mtemp) = Ptempx * VY( 3, 6, mtemp) + WPtempx * VY( 3, 6, mtemp+1) + ABCDtemp * VY( 3, 3, mtemp+1);
    VY( 6, 7, mtemp) = Ptempx * VY( 3, 7, mtemp) + WPtempx * VY( 3, 7, mtemp+1) + 2 * ABCDtemp * VY( 3, 1, mtemp+1);
    VY( 6, 8, mtemp) = Ptempx * VY( 3, 8, mtemp) + WPtempx * VY( 3, 8, mtemp+1);
    VY( 6, 9, mtemp) = Ptempx * VY( 3, 9, mtemp) + WPtempx * VY( 3, 9, mtemp+1);
    
    VY( 7, 4, mtemp) = Ptempx * VY( 1, 4, mtemp) + WPtempx * VY( 1, 4, mtemp+1) +  ABtemp * (VY( 0, 4,mtemp)-CDcom*VY( 0, 4,mtemp+1)) + ABCDtemp * VY( 1, 2, mtemp+1);
    VY( 7, 5, mtemp) = Ptempx * VY( 1, 5, mtemp) + WPtempx * VY( 1, 5, mtemp+1) +  ABtemp * (VY( 0, 5,mtemp)-CDcom*VY( 0, 5,mtemp+1));
    VY( 7, 6, mtemp) = Ptempx * VY( 1, 6, mtemp) + WPtempx * VY( 1, 6, mtemp+1) +  ABtemp * (VY( 0, 6,mtemp)-CDcom*VY( 0, 6,mtemp+1)) + ABCDtemp * VY( 1, 3, mtemp+1);
    VY( 7, 7, mtemp) = Ptempx * VY( 1, 7, mtemp) + WPtempx * VY( 1, 7, mtemp+1) +  ABtemp * (VY( 0, 7,mtemp)-CDcom*VY( 0, 7,mtemp+1)) + 2 * ABCDtemp * VY( 1, 1, mtemp+1);
    VY( 7, 8, mtemp) = Ptempx * VY( 1, 8, mtemp) + WPtempx * VY( 1, 8, mtemp+1) +  ABtemp * (VY( 0, 8,mtemp)-CDcom*VY( 0, 8,mtemp+1));
    VY( 7, 9, mtemp) = Ptempx * VY( 1, 9, mtemp) + WPtempx * VY( 1, 9, mtemp+1) +  ABtemp * (VY( 0, 9,mtemp)-CDcom*VY( 0, 9,mtemp+1));
    
    
    VY( 8, 4, mtemp) = Ptempy * VY( 2, 4, mtemp) + WPtempy * VY( 2, 4, mtemp+1) +  ABtemp * (VY( 0, 4,mtemp)-CDcom*VY( 0, 4,mtemp+1)) + ABCDtemp * VY( 2, 1, mtemp+1);
    VY( 8, 5, mtemp) = Ptempy * VY( 2, 5, mtemp) + WPtempy * VY( 2, 5, mtemp+1) +  ABtemp * (VY( 0, 5,mtemp)-CDcom*VY( 0, 5,mtemp+1)) + ABCDtemp * VY( 2, 3, mtemp+1);
    VY( 8, 6, mtemp) = Ptempy * VY( 2, 6, mtemp) + WPtempy * VY( 2, 6, mtemp+1) +  ABtemp * (VY( 0, 6,mtemp)-CDcom*VY( 0, 6,mtemp+1));
    VY( 8, 7, mtemp) = Ptempy * VY( 2, 7, mtemp) + WPtempy * VY( 2, 7, mtemp+1) +  ABtemp * (VY( 0, 7,mtemp)-CDcom*VY( 0, 7,mtemp+1));
    VY( 8, 8, mtemp) = Ptempy * VY( 2, 8, mtemp) + WPtempy * VY( 2, 8, mtemp+1) +  ABtemp * (VY( 0, 8,mtemp)-CDcom*VY( 0, 8,mtemp+1)) + 2 * ABCDtemp * VY( 2, 2, mtemp+1);
    VY( 8, 9, mtemp) = Ptempy * VY( 2, 9, mtemp) + WPtempy * VY( 2, 9, mtemp+1) +  ABtemp * (VY( 0, 9,mtemp)-CDcom*VY( 0, 9,mtemp+1));
    
    VY( 9, 4, mtemp) = Ptempz * VY( 3, 4, mtemp) + WPtempz * VY( 3, 4, mtemp+1) +  ABtemp * (VY( 0, 4,mtemp)-CDcom*VY( 0, 4,mtemp+1));
    VY( 9, 5, mtemp) = Ptempz * VY( 3, 5, mtemp) + WPtempz * VY( 3, 5, mtemp+1) +  ABtemp * (VY( 0, 5,mtemp)-CDcom*VY( 0, 5,mtemp+1)) + ABCDtemp * VY( 3, 2, mtemp+1);
    VY( 9, 6, mtemp) = Ptempz * VY( 3, 6, mtemp) + WPtempz * VY( 3, 6, mtemp+1) +  ABtemp * (VY( 0, 6,mtemp)-CDcom*VY( 0, 6,mtemp+1)) + ABCDtemp * VY( 3, 1, mtemp+1);
    VY( 9, 7, mtemp) = Ptempz * VY( 3, 7, mtemp) + WPtempz * VY( 3, 7, mtemp+1) +  ABtemp * (VY( 0, 7,mtemp)-CDcom*VY( 0, 7,mtemp+1));
    VY( 9, 8, mtemp) = Ptempz * VY( 3, 8, mtemp) + WPtempz * VY( 3, 8, mtemp+1) +  ABtemp * (VY( 0, 8,mtemp)-CDcom*VY( 0, 8,mtemp+1));
    VY( 9, 9, mtemp) = Ptempz * VY( 3, 9, mtemp) + WPtempz * VY( 3, 9, mtemp+1) +  ABtemp * (VY( 0, 9,mtemp)-CDcom*VY( 0, 9,mtemp+1)) + 2 * ABCDtemp * VY( 3, 3, mtemp+1);
    
	return;
}


#ifndef TEST
__device__
#endif
QUICKDouble hrrwhole(int III, int JJJ, int KKK, int LLL, int IJKLTYPE, QUICKDouble* store, \
                     QUICKDouble RAx,QUICKDouble RAy,QUICKDouble RAz, \
                     QUICKDouble RBx,QUICKDouble RBy,QUICKDouble RBz, \
                     QUICKDouble RCx,QUICKDouble RCy,QUICKDouble RCz, \
                     QUICKDouble RDx,QUICKDouble RDy,QUICKDouble RDz)
{
    QUICKDouble Y;
    
    int NAx = LOC2(devSim.KLMN,0,III-1,3,devSim.nbasis);
    int NAy = LOC2(devSim.KLMN,1,III-1,3,devSim.nbasis);
    int NAz = LOC2(devSim.KLMN,2,III-1,3,devSim.nbasis);
    
    int NBx = LOC2(devSim.KLMN,0,JJJ-1,3,devSim.nbasis);
    int NBy = LOC2(devSim.KLMN,1,JJJ-1,3,devSim.nbasis);
    int NBz = LOC2(devSim.KLMN,2,JJJ-1,3,devSim.nbasis);
    
    int NCx = LOC2(devSim.KLMN,0,KKK-1,3,devSim.nbasis);
    int NCy = LOC2(devSim.KLMN,1,KKK-1,3,devSim.nbasis);
    int NCz = LOC2(devSim.KLMN,2,KKK-1,3,devSim.nbasis);
    
    int NDx = LOC2(devSim.KLMN,0,LLL-1,3,devSim.nbasis);
    int NDy = LOC2(devSim.KLMN,1,LLL-1,3,devSim.nbasis);
    int NDz = LOC2(devSim.KLMN,2,LLL-1,3,devSim.nbasis);
    
    
    int MA = LOC3(devTrans, NAx, NAy, NAz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MB = LOC3(devTrans, NBx, NBy, NBz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MC = LOC3(devTrans, NCx, NCy, NCz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MD = LOC3(devTrans, NDx, NDy, NDz, TRANSDIM, TRANSDIM, TRANSDIM);

    switch (IJKLTYPE) {
        case 0:
        case 10:
        case 1000:
        case 1010:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            break;
        }
        case 2000:
        case 20:
        case 2010:
        case 1020:
        case 2020:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM) * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
            break;
        }
        case 100:
        {
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 110:
        {
        
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 101:
        {
            QUICKDouble Y1,Y2;
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 111:
        {
            QUICKDouble Y1,Y2;
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1100:
        {
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1110:
        {   
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1101:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1111:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);

            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            
            break;
        }
        case 1:
        {
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 11:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1001:
        {   
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
        }
        case 1011:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        default:
        {   
            int numAngularL, numAngularR;
            int angularL[20], angularR[20];
            QUICKDouble coefAngularL[20], coefAngularR[20];
            Y = (QUICKDouble) 0;
//            lefthrr
//            lefthrr
            numAngularL = 1;
            // delete the above line.
            for (int i = 0; i<numAngularL; i++) {
                for (int j = 0; j<numAngularR; j++) {
                    Y += coefAngularL[i] * coefAngularR[i] * LOC2(store, angularL[i]-1, angularR[i]-1 , STOREDIM, STOREDIM);
                }
            }
            
            Y = Y * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
            break;
        }
    }
    return Y;
}  