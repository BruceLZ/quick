#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *
 */

#include "gpu.h"
#include <hip/hip_runtime.h>

static 
#ifndef TEST
__constant__
#endif
 gpu_simulation_type devSim;

static
#ifndef TEST
__constant__
#endif
int devTrans[TRANSDIM*TRANSDIM*TRANSDIM];

static
#ifndef TEST
__constant__
#endif
int devMcal[MCALDIM*3];

static
#ifndef TEST
__constant__
#endif
int Sumindex[10]={0,0,1,4,10,20,35,56,84,120};

void upload_sim_to_constant(_gpu_type gpu){
    hipError_t status;
#ifdef TEST
    memcpy(&devSim, &gpu->gpu_sim, sizeof(gpu_simulation_type));
#else    
    status = hipMemcpyToSymbol(HIP_SYMBOL("devSim"), &gpu->gpu_sim, sizeof(gpu_simulation_type), 0, hipMemcpyHostToDevice);
    PRINTERROR(status, " hipMemcpyToSymbol, sim copy to constants failed")
#endif
}

void upload_para_to_const(){

    int trans[TRANSDIM*TRANSDIM*TRANSDIM];
    // Data to trans
    {
        LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   1;
        LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   4;
        LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  10;
        LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  20;
        LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  35;
        LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  56;
        LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) =  84;
        LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 120;
        LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   3;
        LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   6;
        LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  17;
        LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  32;
        LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  48;
        LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  67;
        LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
        LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   9;
        LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  16;
        LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  23;
        LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  42;
        LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  73;
        LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
        LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  19;
        LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  31;
        LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  43;
        LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  79;
        LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
        LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  34;
        LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  49;
        LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  74;
        LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
        LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  55;
        LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  68;
        LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 107;
        LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  83;
        LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
        LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
        LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   2;
        LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   7;
        LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  15;
        LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  28;
        LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  50;
        LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  69;
        LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
        LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   5;
        LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  11;
        LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  26;
        LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  41;
        LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  59;
        LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  87;
        LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  13;
        LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  25;
        LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  36;
        LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  60;
        LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  88;
        LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  30;
        LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  40;
        LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  61;
        LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  94;
        LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  52;
        LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  58;
        LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  89;
        LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  71;
        LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  86;
        LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
        LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   8;
        LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  14;
        LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  22;
        LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  44;
        LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  75;
        LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
        LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  12;
        LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  24;
        LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  37;
        LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  62;
        LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  90;
        LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  21;
        LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  38;
        LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  66;
        LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  99;
        LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  46;
        LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  64;
        LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  98;
        LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  77;
        LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  92;
        LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
        LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  18;
        LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  27;
        LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  45;
        LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  80;
        LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
        LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  29;
        LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  39;
        LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  63;
        LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  95;
        LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  47;
        LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  65;
        LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  97;
        LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  81;
        LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  96;
        LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
        LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  33;
        LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  51;
        LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  76;
        LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
        LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  53;
        LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  57;
        LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  91;
        LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  78;
        LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  93;
        LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;
        LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  54;
        LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  70;
        LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
        LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  72;
        LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  85;
        LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
        LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  82;
        LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
        LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
        LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    }
    
    int Mcal[3*MCALDIM];
    {
        LOC2(Mcal, 0,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   1, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   2, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   3, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   4, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   5, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   6, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,   7, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,   8, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,   9, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  11, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  12, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  13, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  14, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  15, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  16, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  17, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  18, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  19, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  20, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  21, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  22, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  23, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  24, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  25, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  26, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  27, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  28, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  29, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  30, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  31, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  32, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  33, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  34, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  35, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  36, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  37, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  38, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  39, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  40, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  41, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  42, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  43, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  44, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  45, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  46, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  47, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  48, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  49, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  50, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  51, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  52, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  53, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  54, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  55, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  56, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  57, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  58, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  59, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  60, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  61, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  62, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  63, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  64, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  66, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  67, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  68, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  69, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  70, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  71, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  72, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  73, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  74, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  75, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  76, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  77, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  78, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  79, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  80, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  81, 3, MCALDIM) =   6;
        LOC2(Mcal, 0,  82, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  83, 3, MCALDIM) =   0;
        LOC2(Mcal, 0,  84, 3, MCALDIM) =   5;
        LOC2(Mcal, 0,  85, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  86, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  87, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  88, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  89, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  90, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  91, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  92, 3, MCALDIM) =   4;
        LOC2(Mcal, 0,  93, 3, MCALDIM) =   1;
        LOC2(Mcal, 0,  94, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  95, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  96, 3, MCALDIM) =   3;
        LOC2(Mcal, 0,  97, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  98, 3, MCALDIM) =   2;
        LOC2(Mcal, 0,  99, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 100, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 101, 3, MCALDIM) =   1;
        LOC2(Mcal, 0, 102, 3, MCALDIM) =   6;
        LOC2(Mcal, 0, 103, 3, MCALDIM) =   1;
        LOC2(Mcal, 0, 104, 3, MCALDIM) =   6;
        LOC2(Mcal, 0, 105, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 106, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 107, 3, MCALDIM) =   2;
        LOC2(Mcal, 0, 108, 3, MCALDIM) =   5;
        LOC2(Mcal, 0, 109, 3, MCALDIM) =   2;
        LOC2(Mcal, 0, 110, 3, MCALDIM) =   5;
        LOC2(Mcal, 0, 111, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 112, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 113, 3, MCALDIM) =   3;
        LOC2(Mcal, 0, 114, 3, MCALDIM) =   4;
        LOC2(Mcal, 0, 115, 3, MCALDIM) =   3;
        LOC2(Mcal, 0, 116, 3, MCALDIM) =   4;
        LOC2(Mcal, 0, 117, 3, MCALDIM) =   7;
        LOC2(Mcal, 0, 118, 3, MCALDIM) =   0;
        LOC2(Mcal, 0, 119, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   1, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   2, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   3, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   4, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   5, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,   6, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   7, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,   8, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,   9, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  11, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  12, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  13, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  14, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  15, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  16, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  17, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  18, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  19, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  20, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  21, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  22, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  23, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  24, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  25, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  26, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  27, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  28, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  29, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  30, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  31, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  32, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  33, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  34, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  35, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  36, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  37, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  38, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  39, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  40, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  41, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  42, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  43, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  44, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  45, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  46, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  47, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  48, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  49, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  50, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  51, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  52, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  53, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  54, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  55, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  56, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  57, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  58, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  59, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  60, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  61, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  62, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  63, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  64, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  66, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  67, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  68, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  69, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  70, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  71, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  72, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  73, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  74, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  75, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  76, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  77, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  78, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  79, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  80, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  81, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  82, 3, MCALDIM) =   6;
        LOC2(Mcal, 1,  83, 3, MCALDIM) =   0;
        LOC2(Mcal, 1,  84, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  85, 3, MCALDIM) =   5;
        LOC2(Mcal, 1,  86, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  87, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  88, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  89, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  90, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  91, 3, MCALDIM) =   4;
        LOC2(Mcal, 1,  92, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  93, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  94, 3, MCALDIM) =   1;
        LOC2(Mcal, 1,  95, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  96, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  97, 3, MCALDIM) =   3;
        LOC2(Mcal, 1,  98, 3, MCALDIM) =   2;
        LOC2(Mcal, 1,  99, 3, MCALDIM) =   1;
        LOC2(Mcal, 1, 100, 3, MCALDIM) =   6;
        LOC2(Mcal, 1, 101, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 102, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 103, 3, MCALDIM) =   6;
        LOC2(Mcal, 1, 104, 3, MCALDIM) =   1;
        LOC2(Mcal, 1, 105, 3, MCALDIM) =   2;
        LOC2(Mcal, 1, 106, 3, MCALDIM) =   5;
        LOC2(Mcal, 1, 107, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 108, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 109, 3, MCALDIM) =   5;
        LOC2(Mcal, 1, 110, 3, MCALDIM) =   2;
        LOC2(Mcal, 1, 111, 3, MCALDIM) =   3;
        LOC2(Mcal, 1, 112, 3, MCALDIM) =   4;
        LOC2(Mcal, 1, 113, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 114, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 115, 3, MCALDIM) =   4;
        LOC2(Mcal, 1, 116, 3, MCALDIM) =   3;
        LOC2(Mcal, 1, 117, 3, MCALDIM) =   0;
        LOC2(Mcal, 1, 118, 3, MCALDIM) =   7;
        LOC2(Mcal, 1, 119, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   0, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   1, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   2, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   3, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   4, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   5, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   6, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,   7, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   8, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,   9, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  10, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  11, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  12, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  13, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  14, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  15, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  16, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  17, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  18, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  19, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  20, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  21, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  22, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  23, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  24, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  25, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  26, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  27, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  28, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  29, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  30, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  31, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  32, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  33, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  34, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  35, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  36, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  37, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  38, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  39, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  40, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  41, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  42, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  43, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  44, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  45, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  46, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  47, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  48, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  49, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  50, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  51, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  52, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  53, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  54, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  55, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  56, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  57, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  58, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  59, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  60, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  61, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  62, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  63, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  64, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  65, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  66, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  67, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  68, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  69, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  70, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  71, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  72, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  73, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  74, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  75, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  76, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  77, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  78, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  79, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  80, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  81, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  82, 3, MCALDIM) =   0;
        LOC2(Mcal, 2,  83, 3, MCALDIM) =   6;
        LOC2(Mcal, 2,  84, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  85, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  86, 3, MCALDIM) =   5;
        LOC2(Mcal, 2,  87, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  88, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  89, 3, MCALDIM) =   4;
        LOC2(Mcal, 2,  90, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  91, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  92, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  93, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  94, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  95, 3, MCALDIM) =   1;
        LOC2(Mcal, 2,  96, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  97, 3, MCALDIM) =   2;
        LOC2(Mcal, 2,  98, 3, MCALDIM) =   3;
        LOC2(Mcal, 2,  99, 3, MCALDIM) =   6;
        LOC2(Mcal, 2, 100, 3, MCALDIM) =   1;
        LOC2(Mcal, 2, 101, 3, MCALDIM) =   6;
        LOC2(Mcal, 2, 102, 3, MCALDIM) =   1;
        LOC2(Mcal, 2, 103, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 104, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 105, 3, MCALDIM) =   5;
        LOC2(Mcal, 2, 106, 3, MCALDIM) =   2;
        LOC2(Mcal, 2, 107, 3, MCALDIM) =   5;
        LOC2(Mcal, 2, 108, 3, MCALDIM) =   2;
        LOC2(Mcal, 2, 109, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 110, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 111, 3, MCALDIM) =   4;
        LOC2(Mcal, 2, 112, 3, MCALDIM) =   3;
        LOC2(Mcal, 2, 113, 3, MCALDIM) =   4;
        LOC2(Mcal, 2, 114, 3, MCALDIM) =   3;
        LOC2(Mcal, 2, 115, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 116, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 117, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 118, 3, MCALDIM) =   0;
        LOC2(Mcal, 2, 119, 3, MCALDIM) =   7;
    }
    
    // upload to trans device location
    hipError_t status;
#ifdef TEST
    memcpy(devTrans, trans, sizeof(QUICKDouble)*TRANSDIM*TRANSDIM*TRANSDIM);
    memcpy(devMcal, Mcal, sizeof(QUICKDouble)*3*MCALDIM);
#else    
    status = hipMemcpyToSymbol(HIP_SYMBOL(devTrans), trans, sizeof(int)*TRANSDIM*TRANSDIM*TRANSDIM);
    PRINTERROR(status, " hipMemcpyToSymbol, Trans copy to constants failed")
    
    status = hipMemcpyToSymbol(HIP_SYMBOL(devMcal), Mcal, sizeof(int)*3*MCALDIM);
    PRINTERROR(status, " hipMemcpyToSymbol, Mcal copy to constants failed")
#endif
}

#ifdef DEBUG
static float totTime;
#endif

void get2e(_gpu_type gpu)
{
//    dim3 blocks(64,64);
//    gpu->threadsPerBlock = 1;
#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif
	get2e_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    totTime+=time;
    printf("this cycle:%f ms total time:%f ms\n", time, totTime);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif
    
}

__device__ int total(int a)
{
    int e = a*(a+1)/2;
    int f = e*e + e*(2*a+1)/3;
    f = f/2;
    return f;
}

__global__ void 
get2e_kernel()
{
    int aa;
    aa = 0;

    unsigned int offside = blockIdx.x*blockDim.x+threadIdx.x;
    int ii,jj,kk,ll;
    int totalThreads = blockDim.x*gridDim.x;

    int jshell = devSim.jshell;
    unsigned long int totalInt = total(jshell);
    int myInt = (int)totalInt / totalThreads;
    if ((totalInt - myInt*totalThreads)> offside) myInt++;

    unsigned long int currentInt = offside;
    for (int i = 1; i <= myInt; i++) {

        int ii = (int)(sqrt(sqrt((double)(totalInt-currentInt-1)*8)));

        if (total(ii) <= (totalInt-currentInt-1)) ii++;

        int d = totalInt-currentInt;
        d = total(ii)-d;

        int a2 = ii*(ii+1)/2;

        int jj = d / a2;
        d = d % a2;
        ii = jshell-ii+1;
        jj = ii+jj;

        int kk = (int)sqrt((double)(d*2));
        if (d>=(kk*(kk+1)/2)) kk++;
        int ll = kk*(kk+1)/2-d;
        kk = jshell-kk+1;
        ll = jshell-ll+1;

        gpu_shell(ii,jj,kk,ll);

        currentInt += totalThreads;
    }

}


#ifndef TEST
__device__
#endif
void gpu_shell(unsigned int II, unsigned int JJ, unsigned int KK, unsigned int LL)
{
    int nshell = devSim.nshell;
    int QstartI, QstartJ, QstartK, QstartL;
    int QfinalI, QfinalJ, QfinalK, QfinalL;
    
    
    QstartI = devSim.Qstart[II-1];
    QstartJ = devSim.Qstart[JJ-1];
    QstartK = devSim.Qstart[KK-1];
    QstartL = devSim.Qstart[LL-1];
    
    QfinalI = devSim.Qfinal[II-1];
    QfinalJ = devSim.Qfinal[JJ-1];
    QfinalK = devSim.Qfinal[KK-1];
    QfinalL = devSim.Qfinal[LL-1];        
    
    QUICKDouble DNMax = MAX(MAX(4.0*LOC2(devSim.cutMatrix, II-1, JJ-1, nshell, nshell), 4.0*LOC2(devSim.cutMatrix, KK-1, LL-1, nshell, nshell)),
                            MAX(MAX(LOC2(devSim.cutMatrix, II-1, LL-1, nshell, nshell),     LOC2(devSim.cutMatrix, II-1, KK-1, nshell, nshell)),
                                MAX(LOC2(devSim.cutMatrix, JJ-1, KK-1, nshell, nshell),     LOC2(devSim.cutMatrix, JJ-1, LL-1, nshell, nshell))));
    
    if ((LOC2(devSim.YCutoff, KK-1, LL-1, nshell, nshell) * LOC2(devSim.YCutoff, II-1, JJ-1, nshell, nshell))> devSim.integralCutoff && \
        (LOC2(devSim.YCutoff, KK-1, LL-1, nshell, nshell) * LOC2(devSim.YCutoff, II-1, JJ-1, nshell, nshell) * DNMax) > devSim.integralCutoff) {
        
        int indexNum = 0;
        int4 index[16];
        for (int i = QstartI; i<= QfinalI; i++) {
            for (int j = QstartJ; j<= QfinalJ; j++) {
                for (int k = QstartK; k <= QfinalK; k++) {
                    for (int l = QstartL; l<= QfinalL; l++) {
                        index[indexNum].x = i;
                        index[indexNum].y = j;
                        index[indexNum].z = k;
                        index[indexNum].w = l;
                        indexNum++;
                    }
                }
            }
        }
        
        for (int i = 0; i<indexNum; i++) {
            iclass(index[i].x, index[i].y, index[i].z, index[i].w, II, JJ, KK, LL, DNMax);
        }
    }
    
  	return;
}



__device__ QUICKDouble quick_dsqr(QUICKDouble a)
{
    return a*a;
}



#ifndef TEST
__device__
#endif
void iclass(int I, int J, int K, int L, unsigned int II, unsigned int JJ, unsigned int KK, unsigned int LL, QUICKDouble DNMax)
{

      QUICKDouble sharedGcexpoK[6], sharedGcexpoL[6];

      QUICKDouble RAx, RAy, RAz;
      QUICKDouble RBx, RBy, RBz;
      QUICKDouble RCx, RCy, RCz;
      QUICKDouble RDx, RDy, RDz;
//     int KsumtypeI, KsumtypeJ, KsumtypeK, KsumtypeL;
     int kPrimI, kPrimJ, kPrimL, kPrimK;
     int kStartI, kStartJ, kStartK, kStartL;

        /* 
         kAtom A, B, C ,D is the coresponding atom for shell ii, jj, kk, ll
         and be careful with the index difference between Fortran and C++, 
         Fortran starts array index with 1 and C++ starts 0.
         */
       int  katomA = devSim.katom[II-1];
       int  katomB = devSim.katom[JJ-1];
       int  katomC = devSim.katom[KK-1];
       int  katomD = devSim.katom[LL-1];
        /*
         NII1 is the starting angular momenta for shell i and NII2 is the ending
         angular momenta.So it is with other varibles
         */
        int NABCDTYPE =(int) (devSim.Qfinal[II-1]+devSim.Qfinal[JJ-1])*10u+devSim.Qfinal[KK-1]+devSim.Qfinal[LL-1]; // NABCDTYPE is used for hrr
        int NABCD= (int) devSim.Qfinal[II-1]+devSim.Qfinal[JJ-1]+devSim.Qfinal[KK-1]+devSim.Qfinal[LL-1];
        
        /*RA, RB, RC, and RD are the coordinates for atom katomA, katomB, katomC and katomD, 
         which means they are corrosponding coorinates for shell II, JJ, KK, and LL.
         */
        RAx = LOC2(devSim.xyz, 0 , katomA-1, 3, devSim.natom);
        RAy = LOC2(devSim.xyz, 1 , katomA-1, 3, devSim.natom);
        RAz = LOC2(devSim.xyz, 2 , katomA-1, 3, devSim.natom);
        
        RBx = LOC2(devSim.xyz, 0 , katomB-1, 3, devSim.natom);
        RBy = LOC2(devSim.xyz, 1 , katomB-1, 3, devSim.natom);
        RBz = LOC2(devSim.xyz, 2 , katomB-1, 3, devSim.natom);
        
        RCx = LOC2(devSim.xyz, 0 , katomC-1, 3, devSim.natom);
        RCy = LOC2(devSim.xyz, 1 , katomC-1, 3, devSim.natom);
        RCz = LOC2(devSim.xyz, 2 , katomC-1, 3, devSim.natom);
        
        RDx = LOC2(devSim.xyz, 0 , katomD-1, 3, devSim.natom);
        RDy = LOC2(devSim.xyz, 1 , katomD-1, 3, devSim.natom);
        RDz = LOC2(devSim.xyz, 2 , katomD-1, 3, devSim.natom);
        
//        KsumtypeI = devSim.Ksumtype[II-1];
//        KsumtypeJ = devSim.Ksumtype[JJ-1];
//        KsumtypeK = devSim.Ksumtype[KK-1];
//        KsumtypeL = devSim.Ksumtype[LL-1];
        
        kPrimI = devSim.kprim[II-1];
        kPrimJ = devSim.kprim[JJ-1];
        kPrimK = devSim.kprim[KK-1];
        kPrimL = devSim.kprim[LL-1];
        
        kStartI = devSim.kstart[II-1];
        kStartJ = devSim.kstart[JJ-1];
        kStartK = devSim.kstart[KK-1];
        kStartL = devSim.kstart[LL-1];
        
        for (int i = 0; i<6; i++) {
//            sharedGcexpoI[i]  = LOC2(devSim.gcexpo, i , devSim.Ksumtype[II-1]-1, 6, devSim.nbasis);
//            sharedGcexpoJ[i]  = LOC2(devSim.gcexpo, i , devSim.Ksumtype[JJ-1]-1, 6, devSim.nbasis);
            sharedGcexpoK[i]  = LOC2(devSim.gcexpo, i , devSim.Ksumtype[KK-1]-1, 6, devSim.nbasis);
            sharedGcexpoL[i]  = LOC2(devSim.gcexpo, i , devSim.Ksumtype[LL-1]-1, 6, devSim.nbasis);
        }
    
    

    
    
    QUICKDouble store[STOREDIM*STOREDIM];
	memset(store, 0, STOREDIM*STOREDIM*sizeof(QUICKDouble));
    
    for (int JJJ = 0; JJJ < kPrimJ; JJJ++) {
        QUICKDouble BB = LOC2(devSim.gcexpo, JJJ , devSim.Ksumtype[JJ-1]-1, 6, devSim.nbasis);
        for (int III = 0; III < kPrimI; III++) {
			/* In the following comments, we have I, J, K, L denote the primitive gaussian function we use, and
             for example, expo(III, ksumtype(II)) stands for the expo for the IIIth primitive guassian function for II shell, 
             we use I to express the corresponding index.
             AA = expo(I)
             BB = expo(J)
             AB = expo(I)+expo(J)
                            1
             ABtemp = -------------------
                      2(expo(I) + expo(J))
             */
            QUICKDouble AA = LOC2(devSim.gcexpo, III , devSim.Ksumtype[II-1]-1, 6, devSim.nbasis);
            QUICKDouble AB = 1/ (AA + BB);
            
			/*
                              --->                --->
             ->     expo(I) * xyz (I) + expo(J) * xyz(J)
             P  = ---------------------------------------
                              expo(I) + expo(J)
             
                                    -->             -->
             ----->        expo(I)*xyz(I) + expo(J)*xyz(J)                                 -->            -->
             AAtemp = ----------------------------------- * (expo(I) + expo(J)) = expo(I)*xyz(I)+expo(J)*xyz(J)
                                  expo(I) + expo(J)
             
             ----->   ->  ->
             Ptemp  = P - A
             */            
            QUICKDouble cutoffPrim = DNMax * LOC2(devSim.cutPrim, kStartI+III-1, kStartJ+JJJ-1, devSim.jbasis, devSim.jbasis);
            
            QUICKDouble X1 = LOC4(devSim.Xcoeff, kStartI+III-1, kStartJ+JJJ-1, I, J, devSim.jbasis, devSim.jbasis, 4, 4);
                
            for (int LLL = 0 ; LLL < kPrimL; LLL++) {
                for (int KKK = 0; KKK < kPrimK; KKK++) {
                    QUICKDouble DD = sharedGcexpoL[LLL];
                    QUICKDouble cutoffPrim2 = cutoffPrim * LOC2(devSim.cutPrim, kStartK+KKK-1, kStartL+LLL-1, devSim.jbasis, devSim.jbasis);
                    if (cutoffPrim2> devSim.primLimit) {
                        /*
                         CC = expo(L)
                         DD = expo(K)
                         CD = expo(L)+expo(K)
                                            1
                         CDtemp = ----------------------
                                    2(expo(I) + expo(J))
                         ABCD = AB + CD = expo(I)+expo(J)+expo(K)+expo(L)
                                                 AB * CD      (expo(I)+expo(J))*(expo(K)+expo(L))
                         Rou(Greek Letter) =   ----------- = ------------------------------------
                                                 AB + CD         expo(I)+expo(J)+expo(K)+expo(L)

                                      expo(I)+expo(J)                        expo(K)+expo(L)
                         ABcom = --------------------------------  CDcom = --------------------------------
                                  expo(I)+expo(J)+expo(K)+expo(L)           expo(I)+expo(J)+expo(K)+expo(L)
                         
                         ABCDtemp = 1/2(expo(I)+expo(J)+expo(K)+expo(L))                    
                         */
                        QUICKDouble CC = sharedGcexpoK[KKK];
                        QUICKDouble CD = (QUICKDouble) 1/ (CC + DD);
                        QUICKDouble ABCD = (QUICKDouble) 1 / (AA+BB+CC+DD);

                        /*
                         Q' is the weighting center of K and L
                                                   --->           --->
                         ->  ------>       expo(K)*xyz(K)+expo(L)*xyz(L)
                         Q = P'(K,L)  = ------------------------------
                                                 expo(K) + expo(L)
                         
                         W' is the weight center for I, J, K, L
                         
                                        --->             --->             --->            --->
                         ->     expo(I)*xyz(I) + expo(J)*xyz(J) + expo(K)*xyz(K) +expo(L)*xyz(L)
                         W = -------------------------------------------------------------------
                                             expo(I) + expo(J) + expo(K) + expo(L)                                            
                               ->  ->  2
                         RPQ =| P - Q | 
                         
                        ---->   ->  ->
                         Qtemp = Q - K
                         ----->   ->  ->
                         WQtemp = W - Q
                         ----->   ->  ->
                         WPtemp = W - P

                         ->  -> 2
                         T = ROU * | P - Q|
                         */
                        QUICKDouble T = (AA+BB) * (CC+DD) * ABCD * 
                                        (quick_dsqr((RAx * AA + RBx * BB) * AB - (RCx * CC + RDx * DD) * CD)+
                                         quick_dsqr((RAy * AA + RBy * BB) * AB - (RCy * CC + RDy * DD) * CD)+
                                         quick_dsqr((RAz * AA + RBz * BB) * AB - (RCz * CC + RDz * DD) * CD));
                        
                        QUICKDouble YVerticalTemp[VDIM1*VDIM2*VDIM3];
                        FmT(NABCD, T, YVerticalTemp, sqrt(ABCD));
                        vertical(NABCDTYPE, YVerticalTemp, 
                                 (RAx * AA + RBx * BB) * AB - RAx,              // Ptempx
                                 (RAy * AA + RBy * BB) * AB - RAy,              // Ptempy
                                 (RAz * AA + RBz * BB) * AB - RAz,              // Ptempz
                                 (RAx * AA + RBx * BB + RCx * CC + RDx * DD) * ABCD - (RAx * AA + RBx * BB) * AB,   // WPtempx
                                 (RAy * AA + RBy * BB + RCy * CC + RDy * DD) * ABCD - (RAy * AA + RBy * BB) * AB,   // WPtempy
                                 (RAz * AA + RBz * BB + RCz * CC + RDz * DD) * ABCD - (RAz * AA + RBz * BB) * AB,   // WPtempz
                                 (RCx * CC + RDx * DD) * CD - RCx,              // Qtempx
                                 (RCy * CC + RDy * DD) * CD - RCy,              // Qtempy
                                 (RCz * CC + RDz * DD) * CD - RCz,              // Qtempz
                                 (RAx * AA + RBx * BB + RCx * CC + RDx * DD) * ABCD - (RCx * CC + RDx * DD) * CD,   // WQtempx
                                 (RAy * AA + RBy * BB + RCy * CC + RDy * DD) * ABCD - (RCy * CC + RDy * DD) * CD,   // WQtempy
                                 (RAz * AA + RBz * BB + RCz * CC + RDz * DD) * ABCD - (RCz * CC + RDz * DD) * CD,   // WQtempz
                                 0.5 * ABCD,  0.5 * AB , 0.5 * CD, (AA+BB)*ABCD, (CC+DD)*ABCD); 
                        
                        /*
                         X2 is the multiplication of four indices normalized coeffecient
                         */
                        QUICKDouble X2 = X1 * LOC4(devSim.Xcoeff, kStartK+KKK-1, kStartL+LLL-1, K, L, devSim.jbasis, devSim.jbasis, 4, 4);                    
                        for (int i = Sumindex[K+1]+1; i<= Sumindex[K+L+2]; i++) {
                            for (int j = Sumindex[I+1]+1; j<= Sumindex[I+J+2]; j++) {
                                LOC2(store, j-1, i-1, STOREDIM, STOREDIM) += X2 * LOC3(YVerticalTemp, j-1,i-1,0, VDIM1, VDIM2, VDIM3);
                            }
                        }
                    }   
                }
            }
        }
    }
    
    
    int III1 = devSim.Ksumtype[II-1]+LOC2(devSim.Qsbasis, II-1, I, nshell, 4);
    int III2 = devSim.Ksumtype[II-1]+LOC2(devSim.Qfbasis, II-1, I, nshell, 4);
    int JJJ1 = devSim.Ksumtype[JJ-1]+LOC2(devSim.Qsbasis, JJ-1, J, nshell, 4);
    int JJJ2 = devSim.Ksumtype[JJ-1]+LOC2(devSim.Qfbasis, JJ-1, J, nshell, 4);
    int KKK1 = devSim.Ksumtype[KK-1]+LOC2(devSim.Qsbasis, KK-1, K, nshell, 4);
    int KKK2 = devSim.Ksumtype[KK-1]+LOC2(devSim.Qfbasis, KK-1, K, nshell, 4);
    int LLL1 = devSim.Ksumtype[LL-1]+LOC2(devSim.Qsbasis, LL-1, L, nshell, 4);
    int LLL2 = devSim.Ksumtype[LL-1]+LOC2(devSim.Qfbasis, LL-1, L, nshell, 4);
    
    // IJKLTYPE is the I, J, K,L type
    int IJKLTYPE = (int) (1000 * I + 100 *J + 10 * K + L);
    
    // maxIJKL is the max of I,J,K,L
    int maxIJKL = (int)MAX(MAX(I,J),MAX(K,L));
    
    
    if (((maxIJKL == 2)&&(J != 0 || L!=0)) || (maxIJKL >= 3)) {
        IJKLTYPE = 999;
    }
    
    if ((II < JJ) && (II < KK) && (KK < LL)) {
        for (int III = III1; III <= III2; III++) {
            for (int JJJ = JJJ1; JJJ <= JJJ2; JJJ++) {
                for (int KKK = KKK1; KKK <= KKK2 ; KKK++) {
                    for (int LLL = LLL1; LLL <= LLL2; LLL++) {
                        QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);
                        QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSELK = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                        QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);

                        // Find the (ij|kl) integrals where j>i, k>i, l>k, and k and j are equal.
                        QUICKULL val1 = (QUICKULL) (fabs(2.0*DENSELK*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELK*Y < (QUICKDouble)0.0)
                        val1 = 0ull - val1;
                        
                        QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEJI*Y < (QUICKDouble)0.0)
                        val2 = 0ull - val2;
                        
                        QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSELJ*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELJ*Y < (QUICKDouble)0.0)
                        val3 = 0ull - val3;
                        
                        QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEKJ*Y < (QUICKDouble)0.0)
                        val4 = 0ull - val4;
                        
                        QUICKULL val5 = (QUICKULL) (fabs(0.5*DENSELI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSELI*Y < (QUICKDouble)0.0)
                        val5 = 0ull - val5;
                        
                        QUICKULL val6 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                        if ( DENSEKI*Y < (QUICKDouble)0.0)
                        val6 = 0ull - val6;
                        
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                        QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                        QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                        QUICKADD(LOC2(devSim.oULL, JJJ-1, LLL-1, devSim.nbasis, devSim.nbasis), 0ull-val6);
                        QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val6);               
                    }
                }
            }
        }
    }else {
        for (int III = III1; III <= III2; III++) {
            for (int JJJ = MAX(III,JJJ1); JJJ <= JJJ2; JJJ++) {
                for (int KKK = MAX(III,KKK1); KKK <= KKK2; KKK++) {
                    for (int LLL = MAX(KKK,LLL1); LLL <= LLL2; LLL++) {
                        if (III < KKK) {
                        QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);

                            if ((III < JJJ)&&(KKK < LLL)) {
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSELK = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                // Find the (ij|kl) integrals where j>i, k>i, l>k, and k and j are equal.
                                QUICKULL val1 = (QUICKULL) (fabs(2.0*DENSELK*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELK*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;

                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSELJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELJ*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;
                                
                                QUICKULL val5 = (QUICKULL) (fabs(0.5*DENSELI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSELI*Y < (QUICKDouble)0.0)
                                val5 = 0ull - val5;
                                
                                QUICKULL val6 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val6 = 0ull - val6;
                                
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, LLL-1, devSim.nbasis, devSim.nbasis), 0ull-val6);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val5);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val6);                              
                                
                            }else if ((III == JJJ)&&(KKK == LLL)) {
                                
                                // Find  all the (ii|jj) integrals.
                                QUICKDouble DENSEJI = LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJJ = LOC2(devSim.dense, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEII = LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);

                                QUICKULL val1 = (QUICKULL) (fabs(DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEII*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);

                            }else if ((JJJ == KKK)&&(JJJ==LLL)) {

                                // Find all the (ij|jj) integrals.
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJJ = (QUICKDouble) LOC2(devSim.dense, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis);

                                QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis), val2);
                            
                            }else if ((KKK == LLL)&&(III<JJJ)&&(JJJ!=KKK)) {
                                
                                //Find all the (ij|kk) integrals where j>i, k>j.
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKK = (QUICKDouble) LOC2(devSim.dense, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
 
                                QUICKULL val1 = (QUICKULL) (fabs(DENSEKK*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKK*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;

                                QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, KKK-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                                QUICKADD(LOC2(devSim.oULL, JJJ-1, KKK-1, devSim.nbasis, devSim.nbasis), 0ull-val4);

                            }else if ((III==JJJ)&&(KKK<LLL)) {
                                
                                //Find all the (ii|jk) integrals where j>i, k>j.
                                QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, KKK-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis);
                                
                                QUICKULL val1 = (QUICKULL) (fabs(DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEII*Y < (QUICKDouble)0.0)
                                val1 = 0ull - val1;                               
                                
                                QUICKULL val2 = (QUICKULL) (fabs(2.0*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                val2 = 0ull - val2;
                                
                                QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEKI*Y < (QUICKDouble)0.0)
                                val3 = 0ull - val3;
                                
                                QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                if ( DENSEJI*Y < (QUICKDouble)0.0)
                                val4 = 0ull - val4;

                                QUICKADD(LOC2(devSim.oULL, LLL-1, KKK-1, devSim.nbasis, devSim.nbasis), val1);
                                QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val2);
                                QUICKADD(LOC2(devSim.oULL, KKK-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
                            } 
                            
                        }else {
                        
                            if (JJJ <= LLL) {
                                QUICKDouble Y = (QUICKDouble) hrrwhole(III, JJJ, KKK, LLL, IJKLTYPE, store, \
                                                               RAx, RAy, RAz, RBx, RBy, RBz, \
                                                               RCx, RCy, RCz, RDx, RDy, RDz);

                                if((III==JJJ)&&(III==KKK)&&(III==LLL)){
                                    // do all the (ii|ii) integrals
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                }else if ((III == JJJ) && (III == KKK) && (III < LLL)){
                                    
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                                                      

                                    QUICKULL val1 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), val2);

                                }else if ((III == KKK) && (JJJ == LLL) && (III < JJJ)){
                                    
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEJJ = (QUICKDouble) LOC2(devSim.dense, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKULL val1 = (QUICKULL) (fabs(1.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKULL val3 = (QUICKULL) (fabs(0.5*DENSEJJ*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJJ*Y < (QUICKDouble)0.0)
                                    val3 = 0ull - val3;                               
                                    
                                    
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val2);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                }else if ((III == KKK) && (III <  JJJ) && (JJJ < LLL)){
                                    
                                    QUICKDouble DENSEKI = (QUICKDouble) LOC2(devSim.dense, LLL-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEKJ = (QUICKDouble) LOC2(devSim.dense, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEII = (QUICKDouble) LOC2(devSim.dense, III-1, III-1, devSim.nbasis, devSim.nbasis);
                                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(devSim.dense, JJJ-1, III-1, devSim.nbasis, devSim.nbasis);
                                   
                                    QUICKULL val1 = (QUICKULL) (fabs(1.5*DENSEKI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEKI*Y < (QUICKDouble)0.0)
                                    val1 = 0ull - val1;                               
                                    
                                    QUICKULL val2 = (QUICKULL) (fabs(1.5*DENSEJI*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEJI*Y < (QUICKDouble)0.0)
                                    val2 = 0ull - val2;                               
                                    
                                    QUICKULL val3 = (QUICKULL) (fabs(1.0*DENSEKJ*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEKJ*Y < (QUICKDouble)0.0)
                                    val3 = 0ull - val3;                               
                                    
                                    QUICKULL val4 = (QUICKULL) (fabs(0.5*DENSEII*Y*OSCALE) + (QUICKDouble)0.5);
                                    if ( DENSEII*Y < (QUICKDouble)0.0)
                                    val4 = 0ull - val4;
                                    QUICKADD(LOC2(devSim.oULL, JJJ-1, III-1, devSim.nbasis, devSim.nbasis), val1);
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, III-1, devSim.nbasis, devSim.nbasis), val2);
                                    QUICKADD(LOC2(devSim.oULL, III-1, III-1, devSim.nbasis, devSim.nbasis), 0ull-val3);
                                    QUICKADD(LOC2(devSim.oULL, LLL-1, JJJ-1, devSim.nbasis, devSim.nbasis), 0ull-val4);
								}
							}
                        }
                    }
                }
            }
        }
    }
	return;
}


#ifndef TEST
__device__
#endif
  void FmT(int MaxM, QUICKDouble X, QUICKDouble* YVerticalTemp, QUICKDouble sqrtABCD)
{
    const QUICKDouble PIE4 = (QUICKDouble) PI/4.0 ;
    const QUICKDouble XINV = (QUICKDouble) 1.0 /X;
    const QUICKDouble E = (QUICKDouble) exp(-X);
    QUICKDouble WW1;
    
    if (X > 5.0 ) {
        if (X>15.0 ) {
            if (X>33.0 ) {
                WW1 = sqrt(PIE4 * XINV);
            }else {
                WW1 = (( 1.9623264149430E-01 *XINV-4.9695241464490E-01 )*XINV - \
                       6.0156581186481E-05 )*E + sqrt(PIE4*XINV);
            }
        }else if (X>10.0 ) {
            WW1 = (((-1.8784686463512E-01 *XINV+2.2991849164985E-01 )*XINV - \
                    4.9893752514047E-01 )*XINV-2.1916512131607E-05 )*E + sqrt(PIE4*XINV);
        }else {
            WW1 = (((((( 4.6897511375022E-01  *XINV-6.9955602298985E-01 )*XINV + \
                       5.3689283271887E-01 )*XINV-3.2883030418398E-01 )*XINV + \
                     2.4645596956002E-01 )*XINV-4.9984072848436E-01 )*XINV - \
                   3.1501078774085E-06 )*E + sqrt(PIE4*XINV);
        }
    }else if (X >1.0 ) {
        if (X>3.0 ) {
            QUICKDouble Y = (QUICKDouble) X - 4.0 ;
            QUICKDouble F1 = ((((((((((-2.62453564772299E-11 *Y+3.24031041623823E-10  )*Y- \
                                      3.614965656163E-09 )*Y+3.760256799971E-08 )*Y- \
                                    3.553558319675E-07 )*Y+3.022556449731E-06 )*Y- \
                                  2.290098979647E-05 )*Y+1.526537461148E-04 )*Y- \
                                8.81947375894379E-04 )*Y+4.33207949514611E-03 )*Y- \
                              1.75257821619926E-02 )*Y+5.28406320615584E-02 ;
            WW1 = (X+X)*F1+E;
        }else {
            QUICKDouble Y = (QUICKDouble) X - 2.0 ;
            QUICKDouble F1 = ((((((((((-1.61702782425558E-10 *Y+1.96215250865776E-09  )*Y- \
                                      2.14234468198419E-08  )*Y+2.17216556336318E-07  )*Y- \
                                    1.98850171329371E-06  )*Y+1.62429321438911E-05  )*Y- \
                                  1.16740298039895E-04  )*Y+7.24888732052332E-04  )*Y- \
                                3.79490003707156E-03  )*Y+1.61723488664661E-02  )*Y- \
                              5.29428148329736E-02  )*Y+1.15702180856167E-01 ;
            WW1 = (X+X)*F1+E;
        }
        
    }else if (X > 3.0E-7 ) {
        QUICKDouble F1 =(((((((( -8.36313918003957E-08 *X+1.21222603512827E-06  )*X- \
                               1.15662609053481E-05  )*X+9.25197374512647E-05  )*X- \
                             6.40994113129432E-04  )*X+3.78787044215009E-03  )*X- \
                           1.85185172458485E-02  )*X+7.14285713298222E-02  )*X- \
                         1.99999999997023E-01  )*X+3.33333333333318E-01 ;
        WW1 = (X+X)*F1+E;
    }else {
        WW1 = (1.0 -X)/(QUICKDouble)(2.0 * MaxM+1);
    }
    
    if (X > 3.0E-7 ) {
        LOC3(YVerticalTemp, 0, 0, 0, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = 1; m<= MaxM; m++) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (((2*m-1)*LOC3(YVerticalTemp, 0, 0, m-1, VDIM1, VDIM2, VDIM3))- E)*0.5*XINV;
            LOC3(YVerticalTemp, 0, 0, m-1, VDIM1, VDIM2, VDIM3) = LOC3(YVerticalTemp, 0, 0, m-1, VDIM1, VDIM2, VDIM3)*sqrtABCD;
        }
        LOC3(YVerticalTemp, 0, 0, MaxM, VDIM1, VDIM2, VDIM3) = LOC3(YVerticalTemp, 0, 0, MaxM, VDIM1, VDIM2, VDIM3)*sqrtABCD;
    }else {
        LOC3(YVerticalTemp, 0, 0, MaxM, VDIM1, VDIM2, VDIM3) = WW1;
        for (int m = MaxM-1; m >=0; m--) {
            LOC3(YVerticalTemp, 0, 0, m, VDIM1, VDIM2, VDIM3) = (2.0 * X * LOC3(YVerticalTemp, 0, 0, m+1, VDIM1, VDIM2, VDIM3) + E) / (QUICKDouble)(m*2+1);
            LOC3(YVerticalTemp, 0, 0, m+1, VDIM1, VDIM2, VDIM3) = LOC3(YVerticalTemp, 0, 0, m+1, VDIM1, VDIM2, VDIM3)*sqrtABCD;
        }
        LOC3(YVerticalTemp, 0, 0, 0, VDIM1, VDIM2, VDIM3) = LOC3(YVerticalTemp, 0, 0, 0, VDIM1, VDIM2, VDIM3)*sqrtABCD;
    }
	return;
}

#ifndef TEST
__device__
#endif
void vertical(int NABCDTYPE, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy, QUICKDouble Ptempz,\
              QUICKDouble WPtempx,QUICKDouble WPtempy,QUICKDouble WPtempz,\
              QUICKDouble Qtempx, QUICKDouble Qtempy, QUICKDouble Qtempz,\
              QUICKDouble WQtempx,QUICKDouble WQtempy,QUICKDouble WQtempz, \
              QUICKDouble ABCDtemp,QUICKDouble ABtemp,QUICKDouble CDtemp, QUICKDouble ABcom, QUICKDouble CDcom)
{
    switch (NABCDTYPE) {
        // SSSS oribital
        case 0:
        break;
        // PSSS orbital
        case 10:
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        break;
        // SSPS orbital
        case 1:
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        break;
        // PSPS orbital
        case 11:
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSPS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        PSPS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        break;
        case 20:
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        PSSS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        DSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABtemp, CDcom);
        break;
        case 2:
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSPS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSDS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        break;
        case 21:
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        SSPS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        PSPS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        
        PSSS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        DSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABtemp, CDcom);
        
        PSSS(2, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        DSSS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABtemp, CDcom);
        DSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, ABCDtemp);
        break;
        case 12:
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        SSPS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        PSPS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        
        SSDS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        
        SSPS(2, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSDS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        PSDS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        break;
        case 22:
        SSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        
        PSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        SSPS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        PSPS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        
        SSDS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        SSPS(2, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSDS(1, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        PSDS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        
        PSSS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        DSSS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABtemp, CDcom);
        PSSS(2, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz);
        DSSS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABtemp, CDcom);
        DSPS(0, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, ABCDtemp);
        
        SSPS(3, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz);
        SSDS(2, YVerticalTemp, Qtempx, Qtempy, Qtempz, WQtempx, WQtempy, WQtempz, CDtemp, ABcom);
        PSDS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        
        PSPS(1, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp);
        DSDS(0, YVerticalTemp, Ptempx, Ptempy, Ptempz, WPtempx, WPtempy, WPtempz, ABCDtemp, ABtemp, CDcom);
        break;
        
        default:
        break;
    }
	return;
}

#ifndef TEST
__device__
#endif
void PSSS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz)
{
    LOC3(YVerticalTemp, 1, 0,  mtemp, VDIM1, VDIM2, VDIM3) = Ptempx  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WPtempx * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
    LOC3(YVerticalTemp, 2, 0,  mtemp, VDIM1, VDIM2, VDIM3) = Ptempy  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WPtempy * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
    LOC3(YVerticalTemp, 3, 0,  mtemp, VDIM1, VDIM2, VDIM3) = Ptempz  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WPtempz * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
	return;
}

#ifndef TEST
__device__
#endif
void SSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz)
{
    
    LOC3(YVerticalTemp, 0, 1,  mtemp, VDIM1, VDIM2, VDIM3) = Qtempx  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WQtempx * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
    LOC3(YVerticalTemp, 0, 2,  mtemp, VDIM1, VDIM2, VDIM3) = Qtempy  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WQtempy * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
    LOC3(YVerticalTemp, 0, 3,  mtemp, VDIM1, VDIM2, VDIM3) = Qtempz  * LOC3( YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
    + WQtempz * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
	return;
}

#ifndef TEST
__device__
#endif
void PSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp)
{
    
    for (int j = 0; j<3; j++) {
        LOC3(YVerticalTemp, 1 ,j+1, mtemp, VDIM1,VDIM2,VDIM3) = Ptempx * LOC3( YVerticalTemp, 0, j+1, mtemp, VDIM1, VDIM2, VDIM3) \
        +  WPtempx* LOC3( YVerticalTemp, 0, j+1, mtemp+1, VDIM1, VDIM2, VDIM3);
        LOC3(YVerticalTemp, 2 ,j+1, mtemp, VDIM1,VDIM2,VDIM3) = Ptempy * LOC3( YVerticalTemp, 0, j+1, mtemp, VDIM1, VDIM2, VDIM3) \
        +  WPtempy* LOC3( YVerticalTemp, 0, j+1, mtemp+1, VDIM1, VDIM2, VDIM3);
        LOC3(YVerticalTemp, 3 ,j+1, mtemp, VDIM1,VDIM2,VDIM3) = Ptempz * LOC3( YVerticalTemp, 0, j+1, mtemp, VDIM1, VDIM2, VDIM3) \
        +  WPtempz* LOC3( YVerticalTemp, 0, j+1, mtemp+1, VDIM1, VDIM2, VDIM3);
    }
    
    for (int i = 0; i<3; i++) {
        LOC3(YVerticalTemp, i+1, i+1, mtemp, VDIM1, VDIM2, VDIM3) += ABCDtemp * LOC3( YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3);
    }
    
	return;
}

#ifndef TEST
__device__
#endif
void DSSS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABtemp, QUICKDouble CDcom)
{
    int Bx, By, Bz;
    
    for (int i = 4; i<10; i++) {
        Bx = LOC2(devMcal, 0, i, 3, MCALDIM);
        By = LOC2(devMcal, 1, i, 3, MCALDIM);
        Bz = LOC2(devMcal, 2, i, 3, MCALDIM);
        
        if (Bx != 0) {
            Bx = Bx - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempx * LOC3( YVerticalTemp, ii-1, 0, mtemp, VDIM1, VDIM2, VDIM3) \
            + WPtempx * LOC3( YVerticalTemp, ii-1, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (Bx > 0) {
                LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) += ABtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - CDcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }else if (By != 0){
            By = By - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempy * LOC3( YVerticalTemp, ii-1, 0, mtemp, VDIM1, VDIM2, VDIM3) \
            + WPtempy * LOC3( YVerticalTemp, ii-1, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (By > 0) {
                LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) += ABtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - CDcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }else if (Bz != 0) {
            Bz = Bz - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempz * LOC3( YVerticalTemp, ii-1, 0, mtemp, VDIM1, VDIM2, VDIM3) \
            + WPtempz * LOC3( YVerticalTemp, ii-1, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (Bz > 0) {
                LOC3(YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) += ABtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - CDcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }
    }
	return;
}

#ifndef TEST
__device__
#endif
void SSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz, QUICKDouble CDtemp, QUICKDouble ABcom)
{
    int Bx, By, Bz;
    for (int i = 4; i<10; i++) {
        Bx = LOC2(devMcal, 0, i, 3, MCALDIM);
        By = LOC2(devMcal, 1, i, 3, MCALDIM);
        Bz = LOC2(devMcal, 2, i, 3, MCALDIM);
        
        if (Bx != 0) {
            Bx = Bx - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempx * LOC3( YVerticalTemp, 0, ii-1, mtemp, VDIM1, VDIM2, VDIM3) \
            + WQtempx * LOC3( YVerticalTemp, 0, ii-1, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (Bx > 0) {
                LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) += CDtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - ABcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }else if (By != 0){
            By = By - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempy * LOC3( YVerticalTemp, 0, ii-1, mtemp, VDIM1, VDIM2, VDIM3) \
            + WQtempy * LOC3( YVerticalTemp, 0, ii-1, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (By > 0) {
                LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) += CDtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - ABcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }else if (Bz != 0) {
            Bz = Bz - 1;
            int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
            LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempz * LOC3( YVerticalTemp, 0, ii-1, mtemp, VDIM1, VDIM2, VDIM3) \
            + WQtempz * LOC3( YVerticalTemp, 0, ii-1, mtemp+1,VDIM1, VDIM2, VDIM3);
            if (Bz > 0) {
                LOC3(YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) += CDtemp * (LOC3(YVerticalTemp, 0, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                                                                                   - ABcom * LOC3(YVerticalTemp, 0, 0, mtemp+1, VDIM1, VDIM2, VDIM3));
            }
        }
    }
	return;
}



#ifndef TEST
__device__
#endif
void DSPS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Qtempx, QUICKDouble Qtempy,QUICKDouble Qtempz, \
          QUICKDouble WQtempx, QUICKDouble WQtempy, QUICKDouble WQtempz, QUICKDouble ABCDtemp)
{

    int Bx, By, Bz;
    for (int i = 4; i<10; i++) {
        for (int jtemp = 2; jtemp <= 4; jtemp++) {
            Bx = LOC2(devMcal, 0, i, 3, MCALDIM);
            By = LOC2(devMcal, 1, i, 3, MCALDIM);
            Bz = LOC2(devMcal, 2, i, 3, MCALDIM);
            
            if (LOC2(devMcal, 0, jtemp-1, 3, MCALDIM) != 0 ) {
                LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempx * LOC3( YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                + WQtempx * LOC3( YVerticalTemp, i, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (Bx != 0) {
                    Bx = Bx -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, ii-1, 0, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 0, i, 3, MCALDIM);
                }       
            }else if (LOC2(devMcal, 1, jtemp-1, 3, MCALDIM) != 0) {
                LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempy * LOC3( YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                + WQtempy * LOC3( YVerticalTemp, i, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (By != 0) {
                    By = By -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, ii-1, 0, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 1, i, 3, MCALDIM);
                }
            }else if (LOC2(devMcal, 2, jtemp-1, 3, MCALDIM) != 0) {
                LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) =  Qtempz * LOC3( YVerticalTemp, i, 0, mtemp, VDIM1, VDIM2, VDIM3) \
                + WQtempz * LOC3( YVerticalTemp, i, 0, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (Bz != 0) {
                    Bz = Bz -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, i, jtemp-1, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, ii-1, 0, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 2, i, 3, MCALDIM);
                }
            }
        }
    }
    
	return;
}

#ifndef TEST
__device__
#endif
void PSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
          QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp)
{
    int Bx, By, Bz;
    for (int i = 4; i<10; i++) {
        for (int jtemp = 2; jtemp <= 4; jtemp++) {
            Bx = LOC2(devMcal, 0, i, 3, MCALDIM);
            By = LOC2(devMcal, 1, i, 3, MCALDIM);
            Bz = LOC2(devMcal, 2, i, 3, MCALDIM);
            
            if (LOC2(devMcal, 0, jtemp-1, 3, MCALDIM) != 0 ) {
                LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempx * LOC3( YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) \
                + WPtempx * LOC3( YVerticalTemp, 0, i, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (Bx != 0) {
                    Bx = Bx -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, 0, ii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 0, i, 3, MCALDIM);
                }       
            }else if (LOC2(devMcal, 1, jtemp-1, 3, MCALDIM) != 0) {
                LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempy * LOC3( YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) \
                + WPtempy * LOC3( YVerticalTemp, 0, i, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (By != 0) {
                    By = By -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, 0, ii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 1, i, 3, MCALDIM);
                }
            }else if (LOC2(devMcal, 2, jtemp-1, 3, MCALDIM) != 0) {
                LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) =  Ptempz * LOC3( YVerticalTemp, 0, i, mtemp, VDIM1, VDIM2, VDIM3) \
                + WPtempz * LOC3( YVerticalTemp, 0, i, mtemp+1,VDIM1, VDIM2, VDIM3);
                if (Bz != 0) {
                    Bz = Bz -1;
                    int ii = (int) LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                    LOC3(YVerticalTemp, jtemp-1, i, mtemp, VDIM1, VDIM2, VDIM3) += \
                    ABCDtemp * LOC3(YVerticalTemp, 0, ii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 2, i, 3, MCALDIM);
                }
            }
        }
    }
	return;
}

#ifndef TEST
__device__
#endif
void DSDS(int mtemp, QUICKDouble* YVerticalTemp, QUICKDouble Ptempx, QUICKDouble Ptempy,QUICKDouble Ptempz, \
    QUICKDouble WPtempx, QUICKDouble WPtempy, QUICKDouble WPtempz, QUICKDouble ABCDtemp, QUICKDouble ABtemp, QUICKDouble CDcom)
{
    int Ax, Ay, Az;
    int Bx, By, Bz;
    
    for (int i = 4; i<10; i++) {
        for (int jtemp = 4; jtemp < 10; jtemp++) {
            Bx = LOC2(devMcal, 0, i, 3, MCALDIM);
            By = LOC2(devMcal, 1, i, 3, MCALDIM);
            Bz = LOC2(devMcal, 2, i, 3, MCALDIM);
            
            Ax = LOC2(devMcal, 0, jtemp, 3, MCALDIM);
            Ay = LOC2(devMcal, 1, jtemp, 3, MCALDIM);
            Az = LOC2(devMcal, 2, jtemp, 3, MCALDIM);
            
            if (Bx != 0) {
                Bx = Bx - 1;
                int ii = LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) = Ptempx * LOC3( YVerticalTemp, ii-1, jtemp, mtemp, VDIM1, VDIM2, VDIM3) + \
                                                                          +WPtempx * LOC3( YVerticalTemp, ii-1, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3);
            
                if (LOC2(devMcal, 0, i, 3, MCALDIM) >= 2) {
                    LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                          ABtemp * (LOC3(YVerticalTemp, 0, jtemp, mtemp, VDIM1, VDIM2, VDIM3)
                         - CDcom *  LOC3(YVerticalTemp, 0, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3));
                }
                if (Ax != 0) {
                    Ax = Ax -1;
                    int iii = (int) LOC3(devTrans,  Ax, Ay, Az, TRANSDIM, TRANSDIM, TRANSDIM);
                        LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                        ABCDtemp * LOC3(YVerticalTemp, ii-1, iii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 0, jtemp, 3, MCALDIM);
                }
            }else if (By != 0) {
                By = By - 1;
                int ii = LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) = Ptempy * LOC3( YVerticalTemp, ii-1, jtemp, mtemp, VDIM1, VDIM2, VDIM3) + \
                                                                          +WPtempy * LOC3( YVerticalTemp, ii-1, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3);
            
                if (LOC2(devMcal, 1, i, 3, MCALDIM) >= 2) {
                    LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                          ABtemp * (LOC3(YVerticalTemp, 0, jtemp, mtemp, VDIM1, VDIM2, VDIM3)
                         - CDcom *  LOC3(YVerticalTemp, 0, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3));
                }
                if (Ay != 0) {
                    Ay = Ay -1;
                    int iii = (int) LOC3(devTrans,  Ax, Ay, Az, TRANSDIM, TRANSDIM, TRANSDIM);
                        LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                        ABCDtemp * LOC3(YVerticalTemp, ii-1, iii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 1, jtemp, 3, MCALDIM);
                }
            }else if (Bz != 0) {
                Bz = Bz - 1;
                int ii = LOC3(devTrans, Bx, By, Bz, TRANSDIM, TRANSDIM, TRANSDIM);
                LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) = Ptempz * LOC3( YVerticalTemp, ii-1, jtemp, mtemp, VDIM1, VDIM2, VDIM3) + \
                                                                          +WPtempz * LOC3( YVerticalTemp, ii-1, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3);
            
                if (LOC2(devMcal, 2, i, 3, MCALDIM) >= 2) {
                    LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                          ABtemp * (LOC3(YVerticalTemp, 0, jtemp, mtemp, VDIM1, VDIM2, VDIM3)
                         - CDcom *  LOC3(YVerticalTemp, 0, jtemp, mtemp+1, VDIM1, VDIM2, VDIM3));
                }
                if (Az != 0) {
                    Az = Az -1;
                    int iii = (int) LOC3(devTrans,  Ax, Ay, Az, TRANSDIM, TRANSDIM, TRANSDIM);
                        LOC3(YVerticalTemp, i, jtemp, mtemp, VDIM1, VDIM2, VDIM3) += \
                        ABCDtemp * LOC3(YVerticalTemp, ii-1, iii-1, mtemp+1, VDIM1, VDIM2, VDIM3) * LOC2(devMcal, 2, jtemp, 3, MCALDIM);
                }
            }
            
        }
    }
	return;
}
 

#ifndef TEST
__device__
#endif
QUICKDouble hrrwhole(int III, int JJJ, int KKK, int LLL, int IJKLTYPE, QUICKDouble* store, \
                     QUICKDouble RAx,QUICKDouble RAy,QUICKDouble RAz, \
                     QUICKDouble RBx,QUICKDouble RBy,QUICKDouble RBz, \
                     QUICKDouble RCx,QUICKDouble RCy,QUICKDouble RCz, \
                     QUICKDouble RDx,QUICKDouble RDy,QUICKDouble RDz)
{
    QUICKDouble Y;
    
    int NAx = LOC2(devSim.KLMN,0,III-1,3,devSim.nbasis);
    int NAy = LOC2(devSim.KLMN,1,III-1,3,devSim.nbasis);
    int NAz = LOC2(devSim.KLMN,2,III-1,3,devSim.nbasis);
    
    int NBx = LOC2(devSim.KLMN,0,JJJ-1,3,devSim.nbasis);
    int NBy = LOC2(devSim.KLMN,1,JJJ-1,3,devSim.nbasis);
    int NBz = LOC2(devSim.KLMN,2,JJJ-1,3,devSim.nbasis);
    
    int NCx = LOC2(devSim.KLMN,0,KKK-1,3,devSim.nbasis);
    int NCy = LOC2(devSim.KLMN,1,KKK-1,3,devSim.nbasis);
    int NCz = LOC2(devSim.KLMN,2,KKK-1,3,devSim.nbasis);
    
    int NDx = LOC2(devSim.KLMN,0,LLL-1,3,devSim.nbasis);
    int NDy = LOC2(devSim.KLMN,1,LLL-1,3,devSim.nbasis);
    int NDz = LOC2(devSim.KLMN,2,LLL-1,3,devSim.nbasis);
    
    
    int MA = LOC3(devTrans, NAx, NAy, NAz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MB = LOC3(devTrans, NBx, NBy, NBz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MC = LOC3(devTrans, NCx, NCy, NCz, TRANSDIM, TRANSDIM, TRANSDIM);
    int MD = LOC3(devTrans, NDx, NDy, NDz, TRANSDIM, TRANSDIM, TRANSDIM);

    switch (IJKLTYPE) {
        case 0:
        case 10:
        case 1000:
        case 1010:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            break;
        }
        case 2000:
        case 20:
        case 2010:
        case 1020:
        case 2020:
        {
            Y = (QUICKDouble) LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM) * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
            break;
        }
        case 100:
        {
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, 0, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 110:
        {
        
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MB-1, MC-1, STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 101:
        {
            QUICKDouble Y1,Y2;
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 111:
        {
            QUICKDouble Y1,Y2;
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MB-1,  MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,    0, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,    0,  MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1100:
        {
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, 0 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1110:
        {   
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            
            if (NBx != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAx-RBx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBy != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAy-RBy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NBz != 0) {
                Y = (QUICKDouble) LOC2(store, MAB-1, MC-1 , STOREDIM, STOREDIM) + (RAz-RBz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1101:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1,  0, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1,  0, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            break;
        }
        case 1111:
        {
            QUICKDouble Y1,Y2;
            int MAB = (int) LOC3(devTrans, NAx+NBx, NAy+NBy, NAz+NBz, TRANSDIM, TRANSDIM, TRANSDIM);
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);

            if (NDx != 0) {
                QUICKDouble c = (QUICKDouble) (RCx - RDx);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                QUICKDouble c = (QUICKDouble) (RCy - RDy);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                QUICKDouble c = (QUICKDouble) (RCz - RDz);
                Y1 = (QUICKDouble) LOC2(store, MAB-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store, MAB-1, MC-1, STOREDIM, STOREDIM);
                Y2 = (QUICKDouble) LOC2(store,  MA-1, MCD-1 , STOREDIM, STOREDIM) + c * LOC2(store,  MA-1, MC-1, STOREDIM, STOREDIM);
            }
            
            if (NBx != 0) {
                Y = Y1 + (RAx-RBx)*Y2;
            }else if (NBy != 0) {
                Y = Y1 + (RAy-RBy)*Y2;
            }else if (NBz != 0) {
                Y = Y1 + (RAz-RBz)*Y2;
            }
            
            break;
        }
        case 1:
        {
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, 0, STOREDIM, STOREDIM);
            }
            break;
        }
        case 11:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, 0, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, 0, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        case 1001:
        {   
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, 0, STOREDIM, STOREDIM);
            }
        }
        case 1011:
        {
            int MCD = (int) LOC3(devTrans, NCx+NDx, NCy+NDy, NCz+NDz, TRANSDIM, TRANSDIM, TRANSDIM);
            if (NDx != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCx-RDx)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDy != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCy-RDy)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }else if (NDz != 0) {
                Y = (QUICKDouble) LOC2(store, MA-1, MCD-1, STOREDIM, STOREDIM) + (RCz-RDz)*LOC2(store, MA-1, MC-1, STOREDIM, STOREDIM);
            }
            break;
        }
        default:
        {   
            int numAngularL, numAngularR;
            int angularL[20], angularR[20];
            QUICKDouble coefAngularL[20], coefAngularR[20];
            Y = (QUICKDouble) 0;
//            lefthrr
//            lefthrr
            numAngularL = 1;
            // delete the above line.
            for (int i = 0; i<numAngularL; i++) {
                for (int j = 0; j<numAngularR; j++) {
                    Y += coefAngularL[i] * coefAngularR[i] * LOC2(store, angularL[i]-1, angularR[i]-1 , STOREDIM, STOREDIM);
                }
            }
            
            Y = Y * devSim.cons[III-1] * devSim.cons[JJJ-1] * devSim.cons[KKK-1] * devSim.cons[LLL-1];
            break;
        }
    }
    return Y;
}  