#include "hip/hip_runtime.h"
/*
 *  gpu_startup.cu
 *  new_quick
 *
 *  Created by Yipu Miao on 4/20/11.
 *  Copyright 2011 University of Florida. All rights reserved.
 *
 */

#include <stdio.h>
#include <string>
#include "gpu.h"


//-----------------------------------------------
// Set up specified device and be ready to ignite
//-----------------------------------------------
extern "C" void gpu_set_device_(int* gpu_dev_id)
{
    gpu->gpu_dev_id = *gpu_dev_id;
}

//-----------------------------------------------
// create gpu class
//-----------------------------------------------
extern "C" void gpu_startup_(void)
{
	PRINTDEBUG("BEGIN TO WARM UP")
#ifdef DEBUG
    debugFile = fopen("DEBUG", "w+");
#endif
    gpu = new gpu_type;
	PRINTDEBUG("CREATE NEW GPU")
}


//-----------------------------------------------
// Initialize the device
//-----------------------------------------------
extern "C" void gpu_init_(void)
{

    PRINTDEBUG("BEGIN TO INIT")

    int device = -1;
    int gpuCount = 0;
    hipError_t status;
    hipDeviceProp_t deviceProp;
    status = hipGetDeviceCount(&gpuCount);
    PRINTERROR(status,"hipGetDeviceCount gpu_init failed!");
    if (gpuCount == 0)
    {
        printf("NO CUDA-Enabled GPU FOUND.\n");
        hipDeviceReset();
        exit(-1);
    }
    
    if (gpu->gpu_dev_id == -1){
        device = 0;
        // if gpu count is greater than 1(multi-gpu) select one with bigger free memory, or available. 
        if (gpuCount > 1) {
            size_t maxMem = 0;
            for (int i = gpuCount-1; i>=0; i--) {
                /*status = hipSetDevice(i);
                 size_t free_mem = 0;
                 size_t tot_mem  = 0;
                 
                 status = hipMemGetInfo(&free_mem, &tot_mem); // If error returns, that is to say this device is unavailable.
                 // Else, use one with larger memory.
                 if (free_mem >= maxMem) {
                 maxMem = free_mem;
                 device = i;
                 }
                 hipDeviceReset();*/
                hipGetDeviceProperties(&deviceProp, i);
                
                if (((deviceProp.major >= 2) || ((deviceProp.major == 1) && (deviceProp.minor == 3))) &&
                    (deviceProp.totalGlobalMem >= maxMem))
                {
                    maxMem                          = deviceProp.totalGlobalMem;
                    device                          = i;
                }
                
            }
    	    gpu->gpu_dev_id = device;
        }       
        
    }else{
        if (gpu->gpu_dev_id >= gpuCount)
        {
            printf("GPU ID IS ILLEGAL, PLEASE SELECT FROM 0 TO %i.\n", gpuCount-1);
            hipDeviceReset();
            exit(-1);
        }
        
    	hipGetDeviceProperties(&deviceProp, gpu->gpu_dev_id);
    	if ( (deviceProp.major >=2) || ((deviceProp.major == 1) && (deviceProp.minor == 3)))
        	device = gpu->gpu_dev_id;
    	else {
        	printf("SELECT GPU HAS CUDA SUPPORTING VERSION UNDER 1.3. EXITING. \n");
        	hipDeviceReset();
        	exit(-1);
    	}
        device = gpu->gpu_dev_id;
    }
    
    if (device == -1) {
        printf("NO CUDA 1.3 (OR ABOVE) SUPPORTED GPU IS FOUND\n");
        gpu_shutdown_();
        exit(-1);
    }
   
    status = hipSetDevice(device);
    PRINTERROR(status, "hipSetDevice gpu_init failed!");
    hipDeviceSynchronize();
    
    gpu->blocks = deviceProp.multiProcessorCount;
    if (deviceProp.major ==1) {
        switch (deviceProp.minor) {
            case 0:
            case 1:
            case 2:
            case 5:
                printf("GPU SM VERSION SHOULD BE HIGHER THAN 1.3\n");
                gpu_shutdown_();
                exit(-1);
                break;
            default:
                gpu -> sm_version           =   SM_13;
                gpu -> threadsPerBlock      =   SM_13_THREADS_PER_BLOCK;
                gpu -> twoEThreadsPerBlock  =   SM_13_2E_THREADS_PER_BLOCK;
                gpu -> XCThreadsPerBlock    =   SM_13_XC_THREADS_PER_BLOCK;
                break;
        }
    }else {
        gpu -> sm_version               = SM_2X;
        gpu -> threadsPerBlock          = SM_2X_THREADS_PER_BLOCK;
        gpu -> twoEThreadsPerBlock      = SM_2X_2E_THREADS_PER_BLOCK;
        gpu -> XCThreadsPerBlock        = SM_2X_XC_THREADS_PER_BLOCK;
    }

    PRINTDEBUG("FINISH INIT")

    return;
}

extern "C" void gpu_get_device_info_(int* gpu_dev_count, int* gpu_dev_id,int* gpu_dev_mem,
                                     int* gpu_num_proc,double* gpu_core_freq,char* gpu_dev_name,int* name_len, int* majorv, int* minorv)
{
    hipError_t cuda_error;
    hipDeviceProp_t prop;
    size_t device_mem;
    
    *gpu_dev_id = gpu->gpu_dev_id;  // currently one single GPU is supported
    cuda_error = hipGetDeviceCount(gpu_dev_count);
    PRINTERROR(cuda_error,"hipGetDeviceCount gpu_get_device_info failed!");
    if (*gpu_dev_count == 0) 
    {
        printf("NO CUDA DEVICE FOUNDED \n");
        hipDeviceReset();
        exit(-1);
    }
    hipGetDeviceProperties(&prop,*gpu_dev_id);
    device_mem = (prop.totalGlobalMem/(1024*1024));
    *gpu_dev_mem = (int) device_mem;
    *gpu_num_proc = (int) (prop.multiProcessorCount);
    *gpu_core_freq = (double) (prop.clockRate * 1e-6f);
    strcpy(gpu_dev_name,prop.name);
    *name_len = strlen(gpu_dev_name);
    *majorv = prop.major;
    *minorv = prop.minor;
    
}

//-----------------------------------------------
// shutdonw gpu and terminate gpu calculation part
//-----------------------------------------------
extern "C" void gpu_shutdown_(void)
{
	PRINTDEBUG("BEGIN TO SHUTDOWN")
#ifdef DEBUG
    fclose(debugFile);
#endif
    delete gpu;
    hipDeviceReset();
	PRINTDEBUG("SHUTDOWN NORMALLY")
    return;
}
;
//-----------------------------------------------
//  Setup up basic infomation of the system
//-----------------------------------------------
extern "C" void gpu_setup_(int* natom, int* nbasis, int* nElec, int* imult, int* molchg, int* iAtomType)
{

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO SETUP")

    gpu -> natom                    =   *natom;
    gpu -> nbasis                   =   *nbasis;
    gpu -> nElec                    =   *nElec;
    gpu -> imult                    =   *imult;
    gpu -> molchg                   =   *molchg;
    gpu -> iAtomType                =   *iAtomType;
    gpu -> gpu_calculated           =   new gpu_calculated_type;
    gpu -> gpu_basis                =   new gpu_basis_type;
    gpu -> gpu_cutoff               =   new gpu_cutoff_type;
    gpu -> gpu_calculated -> natom  =   *natom;
    gpu -> gpu_basis -> natom       =   *natom;
    gpu -> gpu_calculated -> nbasis =   *nbasis;
    gpu -> gpu_basis -> nbasis      =   *nbasis;
    
    gpu -> gpu_sim.natom            =   *natom;
    gpu -> gpu_sim.nbasis           =   *nbasis;
    gpu -> gpu_sim.nElec            =   *nElec;
    gpu -> gpu_sim.imult            =   *imult;
    gpu -> gpu_sim.molchg           =   *molchg;
    gpu -> gpu_sim.iAtomType        =   *iAtomType;

	upload_para_to_const();

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD PARA TO CONST",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("FINISH SETUP")    
}

extern "C" void gpu_upload_method_(int* quick_method)
{
    if (*quick_method == 0) {
        gpu -> gpu_sim.method = HF;
    }else if (*quick_method == 1) {
        gpu -> gpu_sim.method = B3LYP;
    }else if (*quick_method == 2) {
        gpu -> gpu_sim.method = DFT;
    }
}

//-----------------------------------------------
//  upload coordinates
//-----------------------------------------------
extern "C" void gpu_upload_xyz_(QUICKDouble* atom_xyz)
{
#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO UPLOAD COORDINATES")
//    gpu -> gpu_basis -> xyz = new cuda_buffer_type<QUICKDouble>(atom_xyz, 3, gpu->natom);
//	gpu -> gpu_basis -> xyz ->Upload();
    gpu -> gpu_calculated -> distance = new cuda_buffer_type<QUICKDouble>(gpu->natom, gpu->natom);

    gpu -> xyz = new cuda_buffer_type<QUICKDouble>(atom_xyz, 3, gpu->natom);
    
    for (int i = 0; i < gpu->natom; i++) {
        for (int j = 0; j < gpu->natom; j++) {
            QUICKDouble distance = 0;
            for (int k = 0; k<3; k++) {
                distance += pow(LOC2(gpu->xyz->_hostData, k, i, 3, gpu->natom)
                                -LOC2(gpu->xyz->_hostData, k, j, 3, gpu->natom),2);
            }
            LOC2(gpu->gpu_calculated->distance->_hostData, i, j, gpu->natom, gpu->natom) = sqrt(distance);
        }
    }
    
    gpu -> xyz -> Upload();
    gpu -> gpu_calculated -> distance -> Upload();

    gpu -> gpu_sim.xyz =  gpu -> xyz -> _devData;
    gpu -> gpu_sim.distance = gpu -> gpu_calculated -> distance -> _devData;

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD XYZ",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("COMPLETE UPLOADING COORDINATES")

}


//-----------------------------------------------
//  upload molecule infomation
//-----------------------------------------------
extern "C" void gpu_upload_atom_and_chg_(int* atom, QUICKDouble* atom_chg)
{

    PRINTDEBUG("BEGIN TO UPLOAD ATOM AND CHARGE")
    
    gpu -> iattype = new cuda_buffer_type<int>(atom, gpu->natom);
    gpu -> chg     = new cuda_buffer_type<QUICKDouble>(atom_chg, gpu->natom);
    gpu -> iattype -> Upload();
    gpu -> chg     -> Upload();
    
    
    gpu -> gpu_sim.chg              = gpu -> chg -> _devData;
    gpu -> gpu_sim.iattype          = gpu -> iattype -> _devData;
    
    PRINTDEBUG("COMPLETE UPLOADING ATOM AND CHARGE")
}


//-----------------------------------------------
//  upload cutoff criteria, will update every 
//  interation
//-----------------------------------------------
extern "C" void gpu_upload_cutoff_(QUICKDouble* cutMatrix, QUICKDouble* integralCutoff,QUICKDouble* primLimit, QUICKDouble* DMCutoff)
{

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO UPLOAD CUTOFF")
    
    gpu -> gpu_cutoff -> integralCutoff = *integralCutoff;
    gpu -> gpu_cutoff -> primLimit      = *primLimit;
    gpu -> gpu_cutoff -> DMCutoff       = *DMCutoff;
    
    gpu -> gpu_cutoff -> cutMatrix  = new cuda_buffer_type<QUICKDouble>(cutMatrix, gpu->nshell, gpu->nshell);
    
    gpu -> gpu_cutoff -> cutMatrix  -> Upload();

    gpu -> gpu_cutoff -> cutMatrix  -> DeleteCPU();

    gpu -> gpu_sim.cutMatrix        = gpu -> gpu_cutoff -> cutMatrix -> _devData;
    gpu -> gpu_sim.integralCutoff   = gpu -> gpu_cutoff -> integralCutoff;
    gpu -> gpu_sim.primLimit        = gpu -> gpu_cutoff -> primLimit;
    gpu -> gpu_sim.DMCutoff         = gpu -> gpu_cutoff -> DMCutoff;

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD CUTOFF",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("COMPLETE UPLOADING CUTOFF")
}


//-----------------------------------------------
//  upload cutoff matrix, only update at first
//  interation
//-----------------------------------------------
extern "C" void gpu_upload_cutoff_matrix_(QUICKDouble* YCutoff,QUICKDouble* cutPrim)
{

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO UPLOAD CUTOFF")
    
    gpu -> gpu_cutoff -> natom      = gpu -> natom;
    gpu -> gpu_cutoff -> YCutoff    = new cuda_buffer_type<QUICKDouble>(YCutoff, gpu->nshell, gpu->nshell);
    gpu -> gpu_cutoff -> cutPrim    = new cuda_buffer_type<QUICKDouble>(cutPrim, gpu->jbasis, gpu->jbasis);
    
    gpu -> gpu_cutoff -> YCutoff    -> Upload();
    gpu -> gpu_cutoff -> cutPrim    -> Upload();
    
    gpu -> gpu_cutoff -> sqrQshell  = (gpu -> gpu_basis -> Qshell) * (gpu -> gpu_basis -> Qshell);
    gpu -> gpu_cutoff -> sorted_YCutoffIJ           = new cuda_buffer_type<int2>(gpu->gpu_cutoff->sqrQshell);
    
    
    int a = 0;
    bool flag = true;
    int2 temp; 
    
    for (int q = 0; q <= 2; q++) {
        for (int p = 0; p <= 2; p++) {
            
            // First to order ERI type
            // Second to order primitive Gaussian function number
            // Third to order Schwartz cutoff upbound
            
            int b=0;
            for (int i = 0; i < gpu->gpu_basis->Qshell; i++) {
                for (int j = 0; j<gpu->gpu_basis->Qshell; j++) {
                    if (gpu->gpu_basis->sorted_Qnumber->_hostData[i] == q && gpu->gpu_basis->sorted_Qnumber->_hostData[j] == p) {
                        if (LOC2(YCutoff, gpu->gpu_basis->sorted_Q->_hostData[i], gpu->gpu_basis->sorted_Q->_hostData[j], gpu->nshell, gpu->nshell) > 1E-9 && 
                            gpu->gpu_basis->sorted_Q->_hostData[i] <= gpu->gpu_basis->sorted_Q->_hostData[j]) {
                            gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[a].x = i;
                            gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[a].y = j;
                            a++;
                            b++;
                        }
                    }
                }
            }    

            PRINTDEBUG("FINISH STEP 1")  
            printf("a=%i b=%i\n", a, b); 
            for (int i = 0; i < b - 1; i ++)
            {
                flag = true;
                for (int j = 0; j < b - i - 1; j ++)
                {
                    if ((LOC2(YCutoff, gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].x], \
                              gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].y], gpu->nshell, gpu->nshell) < \
                         LOC2(YCutoff, gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1+a-b].x], \
                              gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1+a-b].y], gpu->nshell, gpu->nshell)))
                        //&&
                           //gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1].x] == q &&  \
                             //gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1].y]== p &&  \
                             //gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j].x] == q && \
                             //gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j].y] == p )
                    {
                        temp = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b];
                        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b] = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j + 1+a-b];
                        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j + 1+a-b] = temp;
                        flag = false;
                    }
                } 
                
                if (flag == true)
                    break;
            }
            
            PRINTDEBUG("FINISH STEP 2")
            flag = true;
             
            for (int i = 0; i < b - 1; i ++)
            {
                flag = true;
                for (int j = 0; j < b - i - 1; j ++)
                { 
                    if (gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].x]] *
                        gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].y]] <
                        gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1+a-b].x]] *
                        gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1+a-b].y]])
                    {
                        temp = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b];
                        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b] = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b + 1];
                        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b + 1] = temp;
                        flag = false;
                    }
                    else if (gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].x]] *
                              gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].y]] ==
                              gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b+1].x]] *
                              gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b+1].y]])
                    {
                        if (gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b].x]]<
                            gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+1+a-b].x]]) {
                            temp = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b];
                            gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j+a-b] = gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j + 1+a-b];
                            gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[j + 1+a-b] = temp;
                            flag = false;
                        }
                    }
                } 
                
                if (flag == true)
                    break;
            }
            
            flag = true;
            PRINTDEBUG("FINISH STEP 3") 
        }
    }
    
    printf("a = %i, total = %i, pect= %f\n", a, gpu->gpu_basis->Qshell * (gpu->gpu_basis->Qshell+1)/2, (float) 2*a/(gpu->gpu_basis->Qshell*(gpu->gpu_basis->Qshell)));
        
    gpu->gpu_cutoff->sqrQshell  = a;
   /* 
    printf("SS = %i\n",a);
    for (int i = 0; i<a; i++) {
        printf("%8i %4i %4i %18.13f Q=%4i %4i %4i %4i prim = %4i %4i\n", i, \
        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].x, \
        gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].y, \
        LOC2(YCutoff, gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].x], gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].y], gpu->nshell, gpu->nshell),\
        gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].x], \
        gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].y], \
        gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].x], \
        gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].y], \
        gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].x]],
        gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ ->_hostData[i].y]]);
    }
    */
    gpu -> gpu_cutoff -> sorted_YCutoffIJ  -> Upload();
    gpu -> gpu_sim.sqrQshell        = gpu -> gpu_cutoff -> sqrQshell;
    gpu -> gpu_sim.YCutoff          = gpu -> gpu_cutoff -> YCutoff -> _devData;
    gpu -> gpu_sim.cutPrim          = gpu -> gpu_cutoff -> cutPrim -> _devData;
    gpu -> gpu_sim.sorted_YCutoffIJ = gpu -> gpu_cutoff -> sorted_YCutoffIJ  -> _devData;
    
    
    gpu -> gpu_cutoff -> YCutoff -> DeleteCPU();
    gpu -> gpu_cutoff -> cutPrim -> DeleteCPU();
    gpu -> gpu_cutoff -> sorted_YCutoffIJ -> DeleteCPU();
 
#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD CUTOFF",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("COMPLETE UPLOADING CUTOFF")
}

//-----------------------------------------------
//  upload calculated information
//-----------------------------------------------
extern "C" void gpu_upload_calculated_(QUICKDouble* o, QUICKDouble* co, QUICKDouble* vec, QUICKDouble* dense)
{

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO UPLOAD O MATRIX")
    
    gpu -> gpu_calculated -> o        =   new cuda_buffer_type<QUICKDouble>(o,      gpu->nbasis, gpu->nbasis);
    gpu -> gpu_calculated -> o        ->  DeleteGPU();
    gpu -> gpu_calculated -> dense    =   new cuda_buffer_type<QUICKDouble>(dense,  gpu->nbasis, gpu->nbasis);
    gpu -> gpu_calculated -> oULL     =   new cuda_buffer_type<QUICKULL>(gpu->nbasis, gpu->nbasis);
    
    
    /*
        oULL is the unsigned long long int type of O matrix. The reason to do so is because 
        Atomic Operator for CUDA 2.0 is only available for integer. So for double precision type, 
        an comprimise way is to multiple a very large number (OSCALE), first and divided it
        after atomic operator.
     */
    for (int i = 0; i<gpu->nbasis; i++) {
        for (int j = 0; j<gpu->nbasis; j++) {
            QUICKULL valUII = (QUICKULL) (fabs ( LOC2( gpu->gpu_calculated->o->_hostData, i, j, gpu->nbasis, gpu->nbasis)*OSCALE + (QUICKDouble)0.5));

            if (LOC2( gpu->gpu_calculated->o->_hostData, i, j, gpu->nbasis, gpu->nbasis)<(QUICKDouble)0.0)
            {
                valUII = 0ull - valUII;
            }
            
            LOC2( gpu->gpu_calculated->oULL->_hostData, i, j, gpu->nbasis, gpu->nbasis) = valUII;
        }
    }
    
//    gpu -> gpu_calculated -> o        -> Upload();
    gpu -> gpu_calculated -> dense    -> Upload();
    gpu -> gpu_calculated -> oULL     -> Upload();
    
//    gpu -> gpu_sim.o                 =  gpu -> gpu_calculated -> o -> _devData;
    gpu -> gpu_sim.dense             =  gpu -> gpu_calculated -> dense -> _devData;
    gpu -> gpu_sim.oULL              =  gpu -> gpu_calculated -> oULL -> _devData;
    
    
#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD CALCULATE",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("COMPLETE UPLOADING O MATRIX")
}

//-----------------------------------------------
//  upload basis set information
//-----------------------------------------------
extern "C" void gpu_upload_basis_(int* nshell, int* nprim, int* jshell, int* jbasis, int* maxcontract, \
int* ncontract, int* itype,     QUICKDouble* aexp,      QUICKDouble* dcoeff,\
int* first_basis_function, int* last_basis_function, int* first_shell_basis_function, int* last_shell_basis_function, \
int* ncenter,   int* kstart,    int* katom,     int* ktype,     int* kprim,  int* kshell, int* Ksumtype, \
int* Qnumber,   int* Qstart,    int* Qfinal,    int* Qsbasis,   int* Qfbasis,\
QUICKDouble* gccoeff,           QUICKDouble* cons,      QUICKDouble* gcexpo, int* KLMN)
{

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

    PRINTDEBUG("BEGIN TO UPLOAD BASIS")
    
    gpu -> gpu_basis -> nshell          =   *nshell;
    gpu -> gpu_basis -> nprim           =   *nprim;
    gpu -> gpu_basis -> jshell          =   *jshell;
    gpu -> gpu_basis -> jbasis          =   *jbasis;
    gpu -> gpu_basis -> maxcontract     =   *maxcontract;
    
    gpu -> nshell                       =   *nshell;
    gpu -> nprim                        =   *nprim;
    gpu -> jshell                       =   *jshell;
    gpu -> jbasis                       =   *jbasis;

    gpu -> gpu_sim.nshell                   =   *nshell;
    gpu -> gpu_sim.nprim                    =   *nprim;
    gpu -> gpu_sim.jshell                   =   *jshell;
    gpu -> gpu_sim.jbasis                   =   *jbasis;
    gpu -> gpu_sim.maxcontract              =   *maxcontract;


    gpu -> gpu_basis -> ncontract                   =   new cuda_buffer_type<int>(ncontract, gpu->nbasis);//gpu->nbasis);
    gpu -> gpu_basis -> itype                       =   new cuda_buffer_type<int>(itype, 3,  gpu->nbasis);//3, gpu->nbasis);
    gpu -> gpu_basis -> aexp                        =   new cuda_buffer_type<QUICKDouble>(aexp, gpu->gpu_basis->maxcontract, gpu->nbasis);//gpu->gpu_basis->maxcontract, gpu->nbasis);
    gpu -> gpu_basis -> dcoeff                      =   new cuda_buffer_type<QUICKDouble>(dcoeff, gpu->gpu_basis->maxcontract, gpu->nbasis);//gpu->gpu_basis->maxcontract, gpu->nbasis);
/*
    gpu -> gpu_basis -> first_basis_function        =   new cuda_buffer_type<int>(first_basis_function, 1);//gpu->natom);
    gpu -> gpu_basis -> last_basis_function         =   new cuda_buffer_type<int>(last_basis_function,  1);//gpu->natom);

    gpu -> gpu_basis -> first_shell_basis_function  =   new cuda_buffer_type<int>(first_shell_basis_function, 1);//gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> last_shell_basis_function   =   new cuda_buffer_type<int>(last_shell_basis_function,  1);//gpu->gpu_basis->nshell);
 
    gpu -> gpu_basis -> ktype                       =   new cuda_buffer_type<int>(ktype,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> kshell                      =   new cuda_buffer_type<int>(kshell,   93);
*/
    gpu -> gpu_basis -> ncenter                     =   new cuda_buffer_type<int>(ncenter,  gpu->gpu_basis->nbasis);

    gpu -> gpu_basis -> kstart                      =   new cuda_buffer_type<int>(kstart,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> katom                       =   new cuda_buffer_type<int>(katom,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> kprim                       =   new cuda_buffer_type<int>(kprim,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Ksumtype                    =   new cuda_buffer_type<int>(Ksumtype, gpu->gpu_basis->nshell+1);

    gpu -> gpu_basis -> Qnumber                     =   new cuda_buffer_type<int>(Qnumber,  gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qstart                      =   new cuda_buffer_type<int>(Qstart,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qfinal                      =   new cuda_buffer_type<int>(Qfinal,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qsbasis                     =   new cuda_buffer_type<int>(Qsbasis,  gpu->gpu_basis->nshell, 4);
    gpu -> gpu_basis -> Qfbasis                     =   new cuda_buffer_type<int>(Qfbasis,  gpu->gpu_basis->nshell, 4);
    gpu -> gpu_basis -> gccoeff                     =   new cuda_buffer_type<QUICKDouble>(gccoeff, MAXPRIM, gpu->nbasis);

    gpu -> gpu_basis -> cons                        =   new cuda_buffer_type<QUICKDouble>(cons, gpu->nbasis);
    gpu -> gpu_basis -> gcexpo                      =   new cuda_buffer_type<QUICKDouble>(gcexpo, MAXPRIM, gpu->nbasis);
    gpu -> gpu_basis -> KLMN                        =   new cuda_buffer_type<int>(KLMN, 3, gpu->nbasis);
    
    gpu -> gpu_basis -> prim_start                  =   new cuda_buffer_type<int>(gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> prim_total = 0;
    
    for (int i = 0 ; i < gpu->gpu_basis->nshell; i++) {
        gpu -> gpu_basis -> prim_start -> _hostData[i] = gpu -> gpu_basis -> prim_total;
        gpu -> gpu_basis -> prim_total += gpu -> gpu_basis -> kprim -> _hostData[i];
    }
    
    for (int i = 0; i<gpu->gpu_basis->nshell; i++) {
        printf("for %i prim= %i, start= %i\n", i, gpu -> gpu_basis -> kprim -> _hostData[i], gpu -> gpu_basis -> prim_start -> _hostData[i]);
    }
    printf("total=%i\n", gpu -> gpu_basis -> prim_total);
    int prim_total = gpu -> gpu_basis -> prim_total;
    gpu -> gpu_sim.prim_total = gpu -> gpu_basis -> prim_total;
    
    gpu -> gpu_basis -> Xcoeff                      =   new cuda_buffer_type<QUICKDouble>(2*gpu->jbasis, 2*gpu->jbasis);
    gpu -> gpu_basis -> expoSum                     =   new cuda_buffer_type<QUICKDouble>(prim_total, prim_total);
    gpu -> gpu_basis -> weightedCenterX             =   new cuda_buffer_type<QUICKDouble>(prim_total, prim_total);
    gpu -> gpu_basis -> weightedCenterY             =   new cuda_buffer_type<QUICKDouble>(prim_total, prim_total);
    gpu -> gpu_basis -> weightedCenterZ             =   new cuda_buffer_type<QUICKDouble>(prim_total, prim_total);
    
    
    /*
        After uploading basis set information, we want to do some more things on CPU so that will accelarate GPU.
        The very first is to sort orbital type. In this case, we will calculate s orbitals then p, d, and etc.
        Here Qshell is the number of shell orbtials, for example, sp orbitals account for 2 shell orbitals, and s orbital accounts
        1 shell orbital.
     */
    gpu->gpu_basis->Qshell = 0;
    for (int i = 0; i<gpu->nshell; i++) {
        gpu->gpu_basis->Qshell += gpu->gpu_basis->Qfinal->_hostData[i] - gpu->gpu_basis->Qstart->_hostData[i] + 1;
    }
    
    for (int i = 0; i<gpu->gpu_basis->nshell; i++) {
        for (int j = 0; j<4; j++) {
            LOC2(gpu->gpu_basis->Qsbasis->_hostData, i, j, gpu->gpu_basis->nshell, 4) += gpu->gpu_basis->Ksumtype->_hostData[i];
            LOC2(gpu->gpu_basis->Qfbasis->_hostData, i, j, gpu->gpu_basis->nshell, 4) += gpu->gpu_basis->Ksumtype->_hostData[i];
        }
    }
    
    gpu -> gpu_sim.Qshell = gpu->gpu_basis->Qshell;
    
    gpu -> gpu_basis -> sorted_Q                    =   new cuda_buffer_type<int>( gpu->gpu_basis->Qshell);
    gpu -> gpu_basis -> sorted_Qnumber              =   new cuda_buffer_type<int>( gpu->gpu_basis->Qshell);
    
    /*
        Now because to sort, sorted_Q stands for the shell no, and sorted_Qnumber is the shell orbital type (or angular momentum).
        For instance:
        
        original: s sp s s s sp s s
        sorteed : s s  s s s s  s s p p
        
        move p orbital to the end of the sequence. so the Qshell stands for the length of sequence after sorting.
     */
    int a = 0;
    for (int i = 0; i<gpu->gpu_basis->nshell; i++) {
        for (int j = gpu->gpu_basis->Qstart->_hostData[i]; j<= gpu->gpu_basis->Qfinal->_hostData[i]; j++) {

            if (a == 0) {
                gpu->gpu_basis->sorted_Q->_hostData[0] = i;
                gpu->gpu_basis->sorted_Qnumber->_hostData[0] = j;
            }else {
                for (int k = 0; k<a; k++) {
                    if (j<gpu->gpu_basis->sorted_Qnumber->_hostData[k]) {
                    
                        int kk = k;
                        for (int l = a; l> kk; l--) {
                            gpu->gpu_basis->sorted_Q->_hostData[l] = gpu->gpu_basis->sorted_Q->_hostData[l-1];
                            gpu->gpu_basis->sorted_Qnumber->_hostData[l] = gpu->gpu_basis->sorted_Qnumber->_hostData[l-1];
                        }
                        
                        gpu->gpu_basis->sorted_Q->_hostData[kk] = i;
                        gpu->gpu_basis->sorted_Qnumber->_hostData[kk] = j;
                        break;
                    }
                    gpu->gpu_basis->sorted_Q->_hostData[a] = i;
                    gpu->gpu_basis->sorted_Qnumber->_hostData[a] = j;
                }
            }
            a++;
        }
    }
    
    
    
    /*
    for (int i = 0; i<gpu->gpu_basis->Qshell; i++) {
        for (int j = i; j<gpu->gpu_basis->Qshell; j++) {
            if (gpu->gpu_basis->sorted_Qnumber->_hostData[i] == gpu->gpu_basis->sorted_Qnumber->_hostData[j]) {
                if (gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[i]] < gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[j]]) {
                    int temp = gpu->gpu_basis->sorted_Q->_hostData[j];
                    gpu->gpu_basis->sorted_Q->_hostData[j] = gpu->gpu_basis->sorted_Q->_hostData[i];
                    gpu->gpu_basis->sorted_Q->_hostData[i] = temp;
                }
            }
        }
    }*/
    
    printf("Pre-Sorted orbitals:\n");
    printf("Qshell = %i\n", gpu->gpu_basis->Qshell);
    for (int i = 0; i<gpu->gpu_basis->Qshell; i++) {
        printf("i= %i, Q=%i, Qnumber= %i, nprim = %i \n", i, gpu->gpu_basis->sorted_Q->_hostData[i], gpu->gpu_basis->sorted_Qnumber->_hostData[i],
                                                             gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[i]]);
    }
    
    
    /*
        some pre-calculated variables includes
        
        expoSum(i,j) = expo(i)+expo(j)
        ------------->                 ->          ->
        weightedCenter(i,j) = (expo(i)*i + expo(j)*j)/(expo(i)+expo(j))
     */
    for (int i = 0; i<gpu->jshell; i++) {
        for (int j = 0; j<gpu->jshell; j++) {
            int kAtomI = gpu->gpu_basis->katom->_hostData[i];
            int kAtomJ = gpu->gpu_basis->katom->_hostData[j];
            int KsumtypeI = gpu->gpu_basis->Ksumtype->_hostData[i];
            int KsumtypeJ = gpu->gpu_basis->Ksumtype->_hostData[j];
            int kstartI = gpu->gpu_basis->kstart->_hostData[i];
            int kstartJ = gpu->gpu_basis->kstart->_hostData[j];
            
            QUICKDouble distance = 0;
            for (int k = 0; k<3; k++) {
                    distance += pow(LOC2(gpu->xyz->_hostData, k, kAtomI-1, 3, gpu->natom)
                                   -LOC2(gpu->xyz->_hostData, k, kAtomJ-1, 3, gpu->natom),2);
            }
            
            QUICKDouble DIJ = distance;
            
            for (int ii = 0; ii<gpu->gpu_basis->kprim->_hostData[i]; ii++) {
                for (int jj = 0; jj<gpu->gpu_basis->kprim->_hostData[j]; jj++) {
                    
                    QUICKDouble II = LOC2(gpu->gpu_basis->gcexpo->_hostData, ii , KsumtypeI-1, MAXPRIM, gpu->nbasis);
                    QUICKDouble JJ = LOC2(gpu->gpu_basis->gcexpo->_hostData, jj , KsumtypeJ-1, MAXPRIM, gpu->nbasis);
                    
                    int ii_start = gpu->gpu_basis->prim_start->_hostData[i];
                    int jj_start = gpu->gpu_basis->prim_start->_hostData[j];
                    
                    //expoSum(i,j) = expo(i)+expo(j)
                    LOC2(gpu->gpu_basis->expoSum->_hostData, ii_start+ii, jj_start+jj, prim_total, prim_total) = II + JJ;
                    
                    
                    //        ------------->                 ->          ->
                    //        weightedCenter(i,j) = (expo(i)*i + expo(j)*j)/(expo(i)+expo(j))
                    LOC2(gpu->gpu_basis->weightedCenterX->_hostData, ii_start+ii, jj_start+jj, prim_total, prim_total) = \
                        (LOC2(gpu->xyz->_hostData, 0, kAtomI-1, 3, gpu->natom) * II + LOC2(gpu->xyz->_hostData, 0, kAtomJ-1, 3, gpu->natom)*JJ)/(II+JJ);
                    LOC2(gpu->gpu_basis->weightedCenterY->_hostData, ii_start+ii, jj_start+jj, prim_total, prim_total) = \
                        (LOC2(gpu->xyz->_hostData, 1, kAtomI-1, 3, gpu->natom) * II + LOC2(gpu->xyz->_hostData, 1, kAtomJ-1, 3, gpu->natom)*JJ)/(II+JJ);
                    LOC2(gpu->gpu_basis->weightedCenterZ->_hostData, ii_start+ii, jj_start+jj, prim_total, prim_total) = \
                        (LOC2(gpu->xyz->_hostData, 2, kAtomI-1, 3, gpu->natom) * II + LOC2(gpu->xyz->_hostData, 2, kAtomJ-1, 3, gpu->natom)*JJ)/(II+JJ);
                    
                    
                    // Xcoeff = exp(-II*JJ/(II+JJ) * DIJ) / (II+JJ) * coeff(i) * coeff(j) * X0
                    QUICKDouble X = exp(-II*JJ/(II+JJ)*DIJ)/(II+JJ);
                    
                    for (int itemp = gpu->gpu_basis->Qstart->_hostData[i]; itemp <= gpu->gpu_basis->Qfinal->_hostData[i]; itemp++) {
                        for (int itemp2 = gpu->gpu_basis->Qstart->_hostData[j]; itemp2 <= gpu->gpu_basis->Qfinal->_hostData[j]; itemp2++) {
                            LOC4(gpu->gpu_basis->Xcoeff->_hostData, kstartI+ii-1, kstartJ+jj-1, \
                                 itemp-gpu->gpu_basis->Qstart->_hostData[i], itemp2-gpu->gpu_basis->Qstart->_hostData[j], gpu->jbasis, gpu->jbasis, 2, 2)
                            = X0 * X * LOC2(gpu->gpu_basis->gccoeff->_hostData, ii, KsumtypeI+itemp-1, MAXPRIM, gpu->nbasis) \
                                     * LOC2(gpu->gpu_basis->gccoeff->_hostData, jj, KsumtypeJ+itemp2-1, MAXPRIM, gpu->nbasis);
                        }
                    }
                }
            }
        }
    }
    
    gpu -> gpu_basis -> upload_all();
    
    gpu -> gpu_sim.expoSum                      =   gpu -> gpu_basis -> expoSum -> _devData;
    gpu -> gpu_sim.weightedCenterX              =   gpu -> gpu_basis -> weightedCenterX -> _devData;
    gpu -> gpu_sim.weightedCenterY              =   gpu -> gpu_basis -> weightedCenterY -> _devData;
    gpu -> gpu_sim.weightedCenterZ              =   gpu -> gpu_basis -> weightedCenterZ -> _devData;
    gpu -> gpu_sim.sorted_Q                     =   gpu -> gpu_basis -> sorted_Q -> _devData;
    gpu -> gpu_sim.sorted_Qnumber               =   gpu -> gpu_basis -> sorted_Qnumber -> _devData;
     
    gpu -> gpu_sim.Xcoeff                       =   gpu -> gpu_basis -> Xcoeff -> _devData;

    gpu -> gpu_sim.ncontract                    =   gpu -> gpu_basis -> ncontract -> _devData;
    gpu -> gpu_sim.dcoeff                       =   gpu -> gpu_basis -> dcoeff -> _devData;
    gpu -> gpu_sim.aexp                         =   gpu -> gpu_basis -> aexp -> _devData;
    gpu -> gpu_sim.ncenter                      =   gpu -> gpu_basis -> ncenter -> _devData;
    gpu -> gpu_sim.itype                        =   gpu -> gpu_basis -> itype -> _devData;
    gpu -> gpu_sim.prim_start                   =   gpu -> gpu_basis -> prim_start -> _devData;
/*
    gpu -> gpu_sim.first_basis_function         =   gpu -> gpu_basis -> first_basis_function -> _devData;
    gpu -> gpu_sim.last_basis_function          =   gpu -> gpu_basis -> last_basis_function -> _devData;
    gpu -> gpu_sim.first_shell_basis_function   =   gpu -> gpu_basis -> first_shell_basis_function -> _devData;
    gpu -> gpu_sim.last_shell_basis_function    =   gpu -> gpu_basis -> last_shell_basis_function -> _devData;
    gpu -> gpu_sim.ktype                        =   gpu -> gpu_basis -> ktype -> _devData;
    gpu -> gpu_sim.kshell                       =   gpu -> gpu_basis -> kshell -> _devData;
  */
    gpu -> gpu_sim.kstart                       =   gpu -> gpu_basis -> kstart -> _devData;    
    gpu -> gpu_sim.katom                        =   gpu -> gpu_basis -> katom -> _devData;
    gpu -> gpu_sim.kprim                        =   gpu -> gpu_basis -> kprim -> _devData;    
    gpu -> gpu_sim.Ksumtype                     =   gpu -> gpu_basis -> Ksumtype -> _devData;
    gpu -> gpu_sim.Qnumber                      =   gpu -> gpu_basis -> Qnumber -> _devData;
    gpu -> gpu_sim.Qstart                       =   gpu -> gpu_basis -> Qstart -> _devData;
    gpu -> gpu_sim.Qfinal                       =   gpu -> gpu_basis -> Qfinal -> _devData;    
    gpu -> gpu_sim.Qsbasis                      =   gpu -> gpu_basis -> Qsbasis -> _devData;
    gpu -> gpu_sim.Qfbasis                      =   gpu -> gpu_basis -> Qfbasis -> _devData;
    gpu -> gpu_sim.gccoeff                      =   gpu -> gpu_basis -> gccoeff -> _devData;
    gpu -> gpu_sim.cons                         =   gpu -> gpu_basis -> cons -> _devData;
    gpu -> gpu_sim.gcexpo                       =   gpu -> gpu_basis -> gcexpo -> _devData;
    gpu -> gpu_sim.KLMN                         =   gpu -> gpu_basis -> KLMN -> _devData;    


    gpu -> gpu_basis -> expoSum -> DeleteCPU();
    gpu -> gpu_basis -> weightedCenterX -> DeleteCPU();
    gpu -> gpu_basis -> weightedCenterY -> DeleteCPU();
    gpu -> gpu_basis -> weightedCenterZ -> DeleteCPU();
    gpu -> gpu_basis -> Xcoeff -> DeleteCPU();
    
    gpu -> gpu_basis -> ncontract -> DeleteCPU();
    gpu -> gpu_basis -> dcoeff -> DeleteCPU();
    gpu -> gpu_basis -> aexp -> DeleteCPU();
    gpu -> gpu_basis -> ncenter -> DeleteCPU();
    gpu -> gpu_basis -> itype -> DeleteCPU();
    
    gpu -> gpu_basis -> kstart -> DeleteCPU();
    gpu -> gpu_basis -> katom -> DeleteCPU();
    //kprim can not be deleted since it will be used later
    //gpu -> gpu_basis -> kprim -> DeleteCPU();
    gpu -> gpu_basis -> Ksumtype -> DeleteCPU();
    gpu -> gpu_basis -> prim_start -> DeleteCPU();
    
    gpu -> gpu_basis -> Qnumber -> DeleteCPU();
    gpu -> gpu_basis -> Qstart -> DeleteCPU();
    gpu -> gpu_basis -> Qfinal -> DeleteCPU();

    gpu -> gpu_basis -> Qsbasis -> DeleteCPU();
    gpu -> gpu_basis -> Qfbasis -> DeleteCPU();
    gpu -> gpu_basis -> gccoeff -> DeleteCPU();
    gpu -> gpu_basis -> cons -> DeleteCPU();
    gpu -> gpu_basis -> gcexpo -> DeleteCPU();
    gpu -> gpu_basis -> KLMN -> DeleteCPU();


#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("UPLOAD BASIS",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    PRINTDEBUG("COMPLETE UPLOADING BASIS")
}


//-----------------------------------------------
//  core part, compute 2-e integrals
//-----------------------------------------------
extern "C" void gpu_get2e_(QUICKDouble* o)
{
    PRINTDEBUG("BEGIN TO RUN GET2E")

    upload_sim_to_constant(gpu);

    PRINTDEBUG("BEGIN TO RUN KERNEL") 

    get2e(gpu);

    PRINTDEBUG("COMPLETE KERNEL")
    gpu -> gpu_calculated -> oULL -> Download();
    
    for (int i = 0; i< gpu->nbasis; i++) {
        for (int j = i; j< gpu->nbasis; j++) {
            QUICKULL valULL = LOC2(gpu->gpu_calculated->oULL->_hostData, j, i, gpu->nbasis, gpu->nbasis);
            QUICKDouble valDB;
            
            if (valULL >= 0x8000000000000000ull) {
                valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
            }
            else
            {
                valDB  = (QUICKDouble) valULL;
            }
            LOC2(gpu->gpu_calculated->o->_hostData,i,j,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
            LOC2(gpu->gpu_calculated->o->_hostData,j,i,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
        }
    }
   
#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif
 
    gpu -> gpu_calculated -> o    -> Download(o);

#ifdef DEBUG
    hipEventRecord(end, 0);    
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("DOWNLOAD O",time);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif 

    PRINTDEBUG("DELETE TEMP VARIABLES")
    
    delete gpu->gpu_calculated->o;
    delete gpu->gpu_calculated->dense;
    delete gpu->gpu_calculated->oULL;

    delete gpu->gpu_cutoff->cutMatrix;

    PRINTDEBUG("COMPLETE RUNNING GET2E")
}

extern "C" void gpu_getxc_(int* isg, QUICKDouble* sigrad2, QUICKDouble* Eelxc, QUICKDouble* aelec, QUICKDouble* belec, QUICKDouble *o)
{
    PRINTDEBUG("BEGIN TO RUN GETXC")

    
    gpu -> gpu_sim.isg = *isg;
    gpu -> gpu_basis -> sigrad2 = new cuda_buffer_type<QUICKDouble>(sigrad2, gpu->nbasis);
    gpu -> gpu_basis -> sigrad2 -> Upload();
    gpu -> gpu_sim.sigrad2      = gpu->gpu_basis->sigrad2->_devData;
    
    gpu -> DFT_calculated       = new cuda_buffer_type<DFT_calculated_type>(1, 1);
    
    
    QUICKULL valUII = (QUICKULL) (fabs ( *Eelxc * OSCALE + (QUICKDouble)0.5));

    if (*Eelxc<(QUICKDouble)0.0)
    {
                valUII = 0ull - valUII;
    }
            
    gpu -> DFT_calculated -> _hostData[0].Eelxc = valUII;
    
    valUII = (QUICKULL) (fabs ( *aelec * OSCALE + (QUICKDouble)0.5));

    if (*aelec<(QUICKDouble)0.0)
    {
                valUII = 0ull - valUII;
    }
    gpu -> DFT_calculated -> _hostData[0].aelec = valUII;
    
    valUII = (QUICKULL) (fabs ( *belec * OSCALE + (QUICKDouble)0.5));

    if (*belec<(QUICKDouble)0.0)
    {
                valUII = 0ull - valUII;
    }
    
    gpu -> DFT_calculated -> _hostData[0].belec = valUII;
    
    gpu -> DFT_calculated -> Upload();
    gpu -> gpu_sim.DFT_calculated= gpu -> DFT_calculated->_devData;
    
    upload_sim_to_constant_dft(gpu);
    PRINTDEBUG("BEGIN TO RUN KERNEL")
    
    getxc(gpu);
    gpu -> gpu_calculated -> oULL -> Download();
    gpu -> DFT_calculated -> Download();
    
    for (int i = 0; i< gpu->nbasis; i++) {
        for (int j = i; j< gpu->nbasis; j++) {
            QUICKULL valULL = LOC2(gpu->gpu_calculated->oULL->_hostData, j, i, gpu->nbasis, gpu->nbasis);
            QUICKDouble valDB;
            
            if (valULL >= 0x8000000000000000ull) {
                valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
            }
            else
            {
                valDB  = (QUICKDouble) valULL;
            }
            LOC2(gpu->gpu_calculated->o->_hostData,i,j,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
            LOC2(gpu->gpu_calculated->o->_hostData,j,i,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
        }
    }
    gpu -> gpu_calculated -> o    -> Download(o);

    
    QUICKULL valULL = gpu->DFT_calculated -> _hostData[0].Eelxc;
    QUICKDouble valDB;
    
    if (valULL >= 0x8000000000000000ull) {
        valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
    }
    else
    {
        valDB  = (QUICKDouble) valULL;
    }
    *Eelxc = (QUICKDouble)valDB*ONEOVEROSCALE;
    
    valULL = gpu->DFT_calculated -> _hostData[0].aelec;
    
    if (valULL >= 0x8000000000000000ull) {
        valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
    }
    else
    {
        valDB  = (QUICKDouble) valULL;
    }
    *aelec = (QUICKDouble)valDB*ONEOVEROSCALE;
    
    valULL = gpu->DFT_calculated -> _hostData[0].belec;
    
    if (valULL >= 0x8000000000000000ull) {
        valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
    }
    else
    {
        valDB  = (QUICKDouble) valULL;
    }
    *belec = (QUICKDouble)valDB*ONEOVEROSCALE;
    
    
    PRINTDEBUG("DELETE TEMP VARIABLES")
    
	delete gpu->gpu_calculated->o;
	delete gpu->gpu_calculated->dense;
	delete gpu->gpu_calculated->oULL;
}



char *trim(char *s) {
    char *ptr;
    if (!s)
        return NULL;   // handle NULL string
    if (!*s)
        return s;      // handle empty string
    for (ptr = s + strlen(s) - 1; (ptr >= s) && isspace(*ptr); --ptr);
    ptr[1] = '\0';
    return s;
}


extern "C" void gpu_aoint_(QUICKDouble* leastIntegralCutoff, QUICKDouble* maxIntegralCutoff, int* intNum, char* intFileName)
{
    PRINTDEBUG("BEGIN TO RUN AOINT")
    
    ERI_entry a;
    FILE *intFile;
    intFile = fopen(trim(intFileName), "wb");
    if (! intFile) {
        printf("UNABLE TO OPEN INT FILE\n");
    }
 	
    int iBatchCount = 0;
    int const availableMem = 400000000;
    int const availableERI = availableMem/sizeof(ERI_entry);
    int nBatchStart[1000], nBatchEnd[1000], nBatchSize[1000];
    int maxIntCount = 0;
    int currentCount = 0;

    
    nBatchStart[0] = 0;
    
    /* 
        fill up the GPU memory and if it is full, run another batch
     */
    
    
    for (int i = 0; i < gpu -> gpu_cutoff -> sqrQshell; i++) {
       
        int intCount = 0;
        
        intCount = (gpu -> gpu_cutoff -> sqrQshell ) * 5;
        
        if (currentCount + intCount < availableERI) {
            currentCount = currentCount + intCount;
        }else{
            nBatchStart[iBatchCount + 1] = i + 1;
            nBatchEnd  [iBatchCount]     = i ;
            nBatchSize [iBatchCount]     = currentCount;
            iBatchCount++;
            currentCount = intCount;
        }
        
    }
    
    
    nBatchEnd[iBatchCount] = gpu -> gpu_cutoff -> sqrQshell - 1  ;
    nBatchSize[iBatchCount]= currentCount;
    iBatchCount++;
    
    for (int i = 0; i < iBatchCount; i++) {
        if (maxIntCount < nBatchSize[i]) {
            maxIntCount = nBatchSize[i];
        }
    }
    
    
    printf("batch count = %i\n", iBatchCount);
    printf("max int count = %i\n", maxIntCount * sizeof(ERI_entry));
    for (int i = 0; i<iBatchCount; i++) {
        printf(" %i from %i to %i %i\n", i, nBatchStart[i], nBatchEnd[i], nBatchSize[i] * sizeof(ERI_entry));
    }
    
    int nBatchERICount = maxIntCount; 
    
    gpu -> aoint_buffer                 = new cuda_buffer_type<ERI_entry>( nBatchERICount, true );
    gpu -> gpu_sim.aoint_buffer         = gpu -> aoint_buffer -> _devData;
    gpu -> gpu_sim.leastIntegralCutoff  = *leastIntegralCutoff;
    gpu -> gpu_sim.maxIntegralCutoff    = *maxIntegralCutoff;
    gpu -> gpu_sim.iBatchSize           = nBatchERICount;
    gpu -> intCount                     = new cuda_buffer_type<QUICKULL>(1);
    gpu -> intCount -> _hostData[0]     = 0;
    gpu -> intCount -> Upload();
    gpu -> gpu_sim.intCount = gpu->intCount->_devData;
    
    upload_sim_to_constant(gpu);
    
#ifdef DEBUG
    float time_downloadERI, time_kernel, time_io;
    time_downloadERI = 0;
    time_io = 0;
    time_kernel = 0;
    hipEvent_t start_tot,end_tot;
    hipEventCreate(&start_tot);
    hipEventCreate(&end_tot);
    hipEventRecord(start_tot, 0);
#endif

    
    for (int iBatch = 0; iBatch < iBatchCount; iBatch++) {
        
        printf("batch %i start %i end %i\n", iBatch, nBatchStart[iBatch], nBatchEnd[iBatch]);

#ifdef DEBUG
        hipEvent_t start,end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);
#endif
        
        gpu -> intCount -> _hostData[0] = 0;
        gpu -> intCount -> Upload();

        // calculate ERI, kernel part
        getAOInt(gpu, nBatchStart[iBatch], nBatchEnd[iBatch]);

#ifdef DEBUG
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float time;
        hipEventElapsedTime(&time, start, end);
        PRINTUSINGTIME("KERNEL",time);
        time_kernel += time;
        hipEventDestroy(start);
        hipEventDestroy(end);
#endif

        
#ifdef DEBUG
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);
#endif
        
        gpu -> intCount -> Download();
        // download ERI from GPU, this is time-consuming part, that need to be reduced
        hipMemcpy(gpu->aoint_buffer->_hostData, gpu->aoint_buffer->_devData, gpu->intCount->_hostData[0]*sizeof(ERI_entry), hipMemcpyDeviceToHost);
        
#ifdef DEBUG
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&time, start, end);
        PRINTUSINGTIME("DOWNLOAD ERI",time);
        time_downloadERI += time;
        hipEventDestroy(start);
        hipEventDestroy(end);
#endif
        
        
#ifdef DEBUG
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);
#endif
        
        
        printf("intCount = %i\n", gpu->intCount->_hostData[0]);
        // write to disk. there is no way to avoid this part currently.
        for (int i = 0; i < gpu->intCount->_hostData[0]  ; i++) {
            
            a = gpu -> aoint_buffer -> _hostData[i];
            
            if (abs(a.value) > *maxIntegralCutoff) {
                fwrite(&a, sizeof(ERI_entry), 1, intFile);
                *intNum = *intNum + 1;
            }
            
        }
        
        
#ifdef DEBUG
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&time, start, end);
        PRINTUSINGTIME("IO",time);
        time_io += time;
        hipEventDestroy(start);
        hipEventDestroy(end);
#endif
        
        
    }
    
    
    delete gpu->aoint_buffer;

#ifdef DEBUG
    
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif
    
    fclose(intFile);

#ifdef DEBUG
    float time;
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    PRINTUSINGTIME("IO FLUSHING",time);
    time_io += time;
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    
    printf(" TOTAL INT = %i \n", *intNum);
    PRINTDEBUG("END TO RUN AOINT KERNEL")
    
#ifdef DEBUG
    hipEventRecord(end_tot, 0);
    hipEventSynchronize(end_tot);
    float time_tot = 0;
    hipEventElapsedTime(&time_tot, start_tot, end_tot);
    PRINTUSINGTIME("KERNEL",time_kernel);
    PRINTUSINGTIME("DOWNLOAD ERI", time_downloadERI);
    PRINTUSINGTIME("IO", time_io);
    PRINTUSINGTIME("TOTAL",time_tot);
    hipEventDestroy(start_tot);
    hipEventDestroy(end_tot);
#endif

}
