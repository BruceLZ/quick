#include "hip/hip_runtime.h"
/*
 *  gpu_startup.cu
 *  new_quick
 *
 *  Created by Yipu Miao on 4/20/11.
 *  Copyright 2011 University of Florida. All rights reserved.
 *
 */

#include <stdio.h>

#include "gpu.h"

//-----------------------------------------------
// Set up specified device and be ready to ignite
//-----------------------------------------------
extern "C" void gpu_set_device_(int* gpu_dev_id)
{
    gpu->gpu_dev_id = *gpu_dev_id;
}

//-----------------------------------------------
// create gpu class
//-----------------------------------------------
extern "C" void gpu_startup_(void)
{
	PRINTDEBUG("BEGIN TO WARM UP")
#ifdef DEBUG
    debugFile = fopen("DEBUG", "w+");
#endif
    gpu = new gpu_type;
	PRINTDEBUG("CREATE NEW GPU")
}


//-----------------------------------------------
// Initialize the device
//-----------------------------------------------
extern "C" void gpu_init_(void)
{

    PRINTDEBUG("BEGIN TO INIT")

    int device = -1;
    int gpuCount = 0;
    hipError_t status;
    hipDeviceProp_t deviceProp;
    status = hipGetDeviceCount(&gpuCount);
    PRINTERROR(status,"hipGetDeviceCount gpu_init failed!");
    if (gpuCount == 0)
    {
        printf("NO CUDA-Enabled GPU FOUND.\n");
        hipDeviceReset();
        exit(-1);
    }
    
    hipGetDeviceProperties(&deviceProp, gpu->gpu_dev_id);
//    if ( (deviceProp.major >=2) || ((deviceProp.major == 1) && (deviceProp.minor == 3))) 
        device = gpu->gpu_dev_id;
/*    else {
        printf("SELECT GPU HAS CUDA SUPPORTING VERSION UNDER 1.3. EXITING. \n");
        hipDeviceReset;
        exit(-1);
    }
  */ 
    if (device == -1) {
        printf("NO CUDA 1.3 SUPPORTED GPU IS FOUND\n");
        gpu_shutdown_();
        exit(-1);
    }

    status = hipSetDevice(device);
    PRINTERROR(status, "hipSetDevice gpu_init failed!");
    hipDeviceSynchronize();
    
    gpu->blocks = deviceProp.multiProcessorCount;
    if (deviceProp.major ==1) {
        switch (deviceProp.minor) {
/*
            case 0:
            case 1:
            case 2:
            case 5:
                printf("GPU SM VERSION SHOULD BE HIGHER THAN 1.3\n");
                gpu_shutdown_();
                exit(-1);
                break;
*/
            default:
                gpu -> sm_version         =   SM_13;
                gpu -> threadsPerBlock    =   SM_13_THREADS_PER_BLOCK;
                break;
        }
    }else {
        gpu -> sm_version               = SM_2X;
        gpu -> threadsPerBlock          = SM_2X_THREADS_PER_BLOCK;
    }

    PRINTDEBUG("FINISH INIT")

    return;
}

//-----------------------------------------------
// shutdonw gpu and terminate gpu calculation part
//-----------------------------------------------
extern "C" void gpu_shutdown_(void)
{
	PRINTDEBUG("BEGIN TO SHUTDOWN")
#ifdef DEBUG
    fclose(debugFile);
#endif
    delete gpu;
    hipDeviceReset();
	PRINTDEBUG("SHUTDOWN NORMALLY")
    return;
}

//-----------------------------------------------
//  Setup up basic infomation of the system
//-----------------------------------------------
extern "C" void gpu_setup_(int* natom, int* nbasis, int* nElec, int* imult, int* molchg, int* iAtomType)
{

    PRINTDEBUG("BEGIN TO SETUP")

    gpu -> natom                    =   *natom;
    gpu -> nbasis                   =   *nbasis;
    gpu -> nElec                    =   *nElec;
    gpu -> imult                    =   *imult;
    gpu -> molchg                   =   *molchg;
    gpu -> iAtomType                =   *iAtomType;
    gpu -> gpu_calculated           =   new gpu_calculated_type;
    gpu -> gpu_basis                =   new gpu_basis_type;
    gpu -> gpu_cutoff               =   new gpu_cutoff_type;
    gpu -> gpu_calculated -> natom  =   *natom;
    gpu -> gpu_basis -> natom       =   *natom;
    gpu -> gpu_calculated -> nbasis =   *nbasis;
    gpu -> gpu_basis -> nbasis      =   *nbasis;
    
    gpu -> gpu_sim.natom            =   *natom;
    gpu -> gpu_sim.nbasis           =   *nbasis;
    gpu -> gpu_sim.nElec            =   *nElec;
    gpu -> gpu_sim.imult            =   *imult;
    gpu -> gpu_sim.molchg           =   *molchg;
    gpu -> gpu_sim.iAtomType        =   *iAtomType;
        
    PRINTDEBUG("FINISH SETUP")
    upload_para_to_const();
}

//-----------------------------------------------
//  upload coordinates
//-----------------------------------------------
extern "C" void gpu_upload_xyz_(QUICKDouble* atom_xyz)
{
    PRINTDEBUG("BEGIN TO UPLOAD COORDINATES")
    gpu -> xyz = new cuda_buffer_type<QUICKDouble>(atom_xyz, 3, gpu->natom);
    gpu -> gpu_basis -> xyz = new cuda_buffer_type<QUICKDouble>(atom_xyz, 3, gpu->natom);
    gpu -> gpu_calculated -> distance = new cuda_buffer_type<QUICKDouble>(gpu->natom, gpu->natom);
    gpu -> xyz -> Upload();

	gpu -> gpu_basis -> xyz ->Upload();
    gpu -> gpu_sim.xyz =  gpu -> gpu_basis -> xyz -> _devData;
/*
    for (int i = 0; i< gpu->natom; i++) {
        for (int j = i; j<gpu->natom; j++) {
            QUICKDouble distance = 0;
            for (int k = 0; k<3; k++) {
                    distance += pow(LOC2(gpu->xyz->_hostData, k, i, gpu->natom, gpu->natom)
                                   -LOC2(gpu->xyz->_hostData, k, j, gpu->natom, gpu->natom),2);
            }
            LOC2(gpu->gpu_calculated->distance->_hostData, i, j, gpu->natom, gpu->natom) = distance;
            LOC2(gpu->gpu_calculated->distance->_hostData, j, i, gpu->natom, gpu->natom) = distance;
        }
    }
    gpu->gpu_calculated->distance->Upload();
    gpu->gpu_sim.distance =  gpu->gpu_calculated->distance->_devData;
*/
    PRINTDEBUG("COMPLETE UPLOADING COORDINATES")

}


//-----------------------------------------------
//  upload molecule infomation
//-----------------------------------------------
extern "C" void gpu_upload_atom_and_chg_(int* atom, QUICKDouble* atom_chg)
{

    PRINTDEBUG("BEGIN TO UPLOAD ATOM AND CHARGE")
/*    
    gpu -> iattype = new cuda_buffer_type<int>(atom, gpu->natom);
    gpu -> chg     = new cuda_buffer_type<QUICKDouble>(atom_chg, gpu->natom);
    gpu -> iattype -> Upload();
    gpu -> chg     -> Upload();
*/
    PRINTDEBUG("COMPLETE UPLOADING ATOM AND CHARGE")
}

extern "C" void gpu_upload_cutoff_(QUICKDouble* cutMatrix, QUICKDouble* YCutoff, QUICKDouble* integralCutoff,\
                                   QUICKDouble* cutPrim, QUICKDouble* primLimit)
{
    PRINTDEBUG("BEGIN TO UPLOAD CUTOFF")
    
    gpu -> gpu_cutoff -> natom      = gpu -> natom;
    gpu -> gpu_cutoff -> cutMatrix  = new cuda_buffer_type<QUICKDouble>(cutMatrix, gpu->nshell, gpu->nshell);
    gpu -> gpu_cutoff -> YCutoff    = new cuda_buffer_type<QUICKDouble>(YCutoff, gpu->nshell, gpu->nshell);
    gpu -> gpu_cutoff -> integralCutoff = *integralCutoff;
    gpu -> gpu_cutoff -> cutPrim    = new cuda_buffer_type<QUICKDouble>(cutPrim, gpu->jbasis, gpu->jbasis);
    gpu -> gpu_cutoff -> primLimit  = *primLimit;
    
    gpu -> gpu_cutoff -> cutMatrix  -> Upload();
    gpu -> gpu_cutoff -> YCutoff    -> Upload();
    gpu -> gpu_cutoff -> cutPrim    -> Upload();
    
    gpu -> gpu_sim.cutMatrix        = gpu -> gpu_cutoff -> cutMatrix -> _devData;
    gpu -> gpu_sim.YCutoff          = gpu -> gpu_cutoff -> YCutoff -> _devData;
    gpu -> gpu_sim.cutPrim          = gpu -> gpu_cutoff -> cutPrim -> _devData;
    gpu -> gpu_sim.integralCutoff   = gpu -> gpu_cutoff -> integralCutoff;
    gpu -> gpu_sim.primLimit        = gpu -> gpu_cutoff -> primLimit;
    PRINTDEBUG("COMPLETE UPLOADING CUTOFF")
}

//-----------------------------------------------
//  upload calculated information,
//  o is the operator matrix,
//  co is the coeffecient matrix,
//  vec is the eigenvector matrix
//  dense it the density matrix.
//-----------------------------------------------
extern "C" void gpu_upload_calculated_(QUICKDouble* o, QUICKDouble* co, QUICKDouble* vec, QUICKDouble* dense)
{
    PRINTDEBUG("BEGIN TO UPLOAD O MATRIX")
    
    gpu -> gpu_calculated -> o        =   new cuda_buffer_type<QUICKDouble>(o,      gpu->nbasis, gpu->nbasis);
//    gpu -> gpu_calculated -> co       =   new cuda_buffer_type<QUICKDouble>(co,     gpu->nbasis, gpu->nbasis);
//    gpu -> gpu_calculated -> vec      =   new cuda_buffer_type<QUICKDouble>(vec,    gpu->nbasis, gpu->nbasis);
    gpu -> gpu_calculated -> dense    =   new cuda_buffer_type<QUICKDouble>(dense,  gpu->nbasis, gpu->nbasis);
    gpu -> gpu_calculated -> oULL     =   new cuda_buffer_type<QUICKULL>(gpu->nbasis, gpu->nbasis);
    gpu -> gpu_calculated -> o        -> Upload();
//    gpu -> gpu_calculated -> co       -> Upload();
//    gpu -> gpu_calculated -> vec      -> Upload();
    gpu -> gpu_calculated -> dense    -> Upload();
    
    for (int i = 0; i<gpu->nbasis; i++) {
        for (int j = 0; j<gpu->nbasis; j++) {
            QUICKULL valUII = (QUICKULL) (fabs ( LOC2( gpu->gpu_calculated->o->_hostData, i, j, gpu->nbasis, gpu->nbasis)*OSCALE + \
                                                 (QUICKDouble)0.5));
            if (LOC2( gpu->gpu_calculated->o->_hostData, i, j, gpu->nbasis, gpu->nbasis)<(QUICKDouble)0.0)
            valUII = 0ull - valUII;
            LOC2( gpu->gpu_calculated->oULL->_hostData, i, j, gpu->nbasis, gpu->nbasis) = valUII;
            
        }
    }
    gpu -> gpu_calculated -> oULL     -> Upload();
    
    gpu -> gpu_sim.o                 =  gpu -> gpu_calculated -> o -> _devData;
//    gpu -> gpu_sim.co                =  gpu -> gpu_calculated -> co -> _devData;
//    gpu -> gpu_sim.vec               =  gpu -> gpu_calculated -> vec -> _devData;
    gpu -> gpu_sim.dense             =  gpu -> gpu_calculated -> dense -> _devData;
    gpu -> gpu_sim.oULL              =  gpu -> gpu_calculated -> oULL -> _devData;
    
    PRINTDEBUG("COMPLETE UPLOADING O MATRIX")
}

/*
//-----------------------------------------------
//  upload calculated information,
//  o is the operator matrix,
//  co is the coeffecient matrix,
//  vec is the eigenvector matrix
//  dense it the density matrix.
//-----------------------------------------------
extern "C" void gpu_download_o_matrix_(QUICKDouble* o)
{
    PRINTDEBUG("BEGIN TO UPLOAD O MATRIX")
    
    gpu -> gpu_calculated -> o        -> Download(o);
    
    PRINTDEBUG("COMPLETE UPLOADING O MATRIX")
}
*/


//-----------------------------------------------
//  upload basis set information
//-----------------------------------------------
extern "C" void gpu_upload_basis_(int* nshell, int* nprim, int* jshell, int* jbasis, int* maxcontract, \
int* ncontract, int* itype,     QUICKDouble* aexp,      QUICKDouble* dcoeff,\
int* first_basis_function, int* last_basis_function, int* first_shell_basis_function, int* last_shell_basis_function, \
int* ncenter,   int* kstart,    int* katom,     int* ktype,     int* kprim,  int* kshell, int* Ksumtype, \
int* Qnumber,   int* Qstart,    int* Qfinal,    int* Qsbasis,   int* Qfbasis,\
QUICKDouble* gccoeff,           QUICKDouble* cons,      QUICKDouble* gcexpo, int* KLMN)
{
    PRINTDEBUG("BEGIN TO UPLOAD BASIS")
    
    gpu -> gpu_basis -> nshell          =   *nshell;
    gpu -> gpu_basis -> nprim           =   *nprim;
    gpu -> gpu_basis -> jshell          =   *jshell;
    gpu -> gpu_basis -> jbasis          =   *jbasis;
    gpu -> gpu_basis -> maxcontract     =   *maxcontract;
    
    gpu -> nshell                       =   *nshell;
    gpu -> nprim                        =   *nprim;
    gpu -> jshell                       =   *jshell;
    gpu -> jbasis                       =   *jbasis;

    gpu -> gpu_sim.nshell                   =   *nshell;
    gpu -> gpu_sim.nprim                    =   *nprim;
    gpu -> gpu_sim.jshell                   =   *jshell;
    gpu -> gpu_sim.jbasis                   =   *jbasis;
    gpu -> gpu_sim.maxcontract              =   *maxcontract;

    gpu -> gpu_basis -> ncontract       =   new cuda_buffer_type<int>(ncontract, gpu->nbasis);
    gpu -> gpu_basis -> itype           =   new cuda_buffer_type<int>(itype, 3, gpu->nbasis);
    gpu -> gpu_basis -> aexp            =   new cuda_buffer_type<QUICKDouble>(aexp, gpu->gpu_basis->maxcontract, gpu->nbasis);
    gpu -> gpu_basis -> dcoeff          =   new cuda_buffer_type<QUICKDouble>(dcoeff, gpu->gpu_basis->maxcontract, gpu->nbasis);

    gpu -> gpu_basis -> first_basis_function        =   new cuda_buffer_type<int>(first_basis_function, gpu->natom);
    gpu -> gpu_basis -> last_basis_function         =   new cuda_buffer_type<int>(last_basis_function,  gpu->natom);

    gpu -> gpu_basis -> first_shell_basis_function  =   new cuda_buffer_type<int>(first_shell_basis_function, gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> last_shell_basis_function   =   new cuda_buffer_type<int>(last_shell_basis_function,  gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> ncenter                     =   new cuda_buffer_type<int>(ncenter,                    gpu->gpu_basis->nbasis);

    gpu -> gpu_basis -> kstart                      =   new cuda_buffer_type<int>(kstart,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> katom                       =   new cuda_buffer_type<int>(katom,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> ktype                       =   new cuda_buffer_type<int>(ktype,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> kprim                       =   new cuda_buffer_type<int>(kprim,    gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> kshell                      =   new cuda_buffer_type<int>(kshell,   93);
    gpu -> gpu_basis -> Ksumtype                    =   new cuda_buffer_type<int>(Ksumtype, gpu->gpu_basis->nshell+1);

    gpu -> gpu_basis -> Qnumber                     =   new cuda_buffer_type<int>(Qnumber,  gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qstart                      =   new cuda_buffer_type<int>(Qstart,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qfinal                      =   new cuda_buffer_type<int>(Qfinal,   gpu->gpu_basis->nshell);
    gpu -> gpu_basis -> Qsbasis                     =   new cuda_buffer_type<int>(Qsbasis,  gpu->gpu_basis->nshell, 4);
    gpu -> gpu_basis -> Qfbasis                     =   new cuda_buffer_type<int>(Qfbasis,  gpu->gpu_basis->nshell, 4);
    gpu -> gpu_basis -> gccoeff                     =   new cuda_buffer_type<QUICKDouble>(gccoeff, 6, gpu->nbasis);

    gpu -> gpu_basis -> cons                        =   new cuda_buffer_type<QUICKDouble>(cons, gpu->nbasis);
    gpu -> gpu_basis -> gcexpo                      =   new cuda_buffer_type<QUICKDouble>(gcexpo, 6, gpu->nbasis);
    gpu -> gpu_basis -> KLMN                        =   new cuda_buffer_type<int>(KLMN, 3, gpu->nbasis);
    
    gpu -> gpu_basis -> Xcoeff                      =   new cuda_buffer_type<QUICKDouble>(4*gpu->jbasis,4*gpu->jbasis);
    
    
    for (int i = 0; i<gpu->jshell; i++) {
        for (int j = 0; j<gpu->jshell; j++) {
            int kAtomI = gpu->gpu_basis->katom->_hostData[i];
            int kAtomJ = gpu->gpu_basis->katom->_hostData[j];
            int KsumtypeI = gpu->gpu_basis->Ksumtype->_hostData[i];
            int KsumtypeJ = gpu->gpu_basis->Ksumtype->_hostData[j];
            int kstartI = gpu->gpu_basis->kstart->_hostData[i];
            int kstartJ = gpu->gpu_basis->kstart->_hostData[j];
            
            QUICKDouble distance = 0;
            for (int k = 0; k<3; k++) {
                    distance += pow(LOC2(gpu->xyz->_hostData, k, kAtomI-1, gpu->natom, gpu->natom)
                                   -LOC2(gpu->xyz->_hostData, k, kAtomJ-1, gpu->natom, gpu->natom),2);
            }
            
            QUICKDouble DIJ = distance;
            
            for (int ii = 0; ii<gpu->gpu_basis->kprim->_hostData[i]; ii++) {
                for (int jj = 0; jj<gpu->gpu_basis->kprim->_hostData[j]; jj++) {
                    
                    QUICKDouble II = LOC2(gpu->gpu_basis->gcexpo->_hostData, ii , KsumtypeI-1, 6, gpu->nbasis);
                    QUICKDouble JJ = LOC2(gpu->gpu_basis->gcexpo->_hostData, jj , KsumtypeJ-1, 6, gpu->nbasis);
                    
                    
                    QUICKDouble X = exp(-II*JJ/(II+JJ)*DIJ)/(II+JJ);
                    for (int itemp = gpu->gpu_basis->Qstart->_hostData[i]; itemp <= gpu->gpu_basis->Qfinal->_hostData[i]; itemp++) {
                        for (int itemp2 = gpu->gpu_basis->Qstart->_hostData[j]; itemp2 <= gpu->gpu_basis->Qfinal->_hostData[j]; itemp2++) {
                            LOC4(gpu->gpu_basis->Xcoeff->_hostData, kstartI+ii-1, kstartJ+jj-1, itemp, itemp2, gpu->jbasis, gpu->jbasis, 4, 4)
                            = X0 * X * LOC2(gpu->gpu_basis->gccoeff->_hostData, ii, KsumtypeI+itemp-1, 6, gpu->nbasis) \
                                     * LOC2(gpu->gpu_basis->gccoeff->_hostData, jj, KsumtypeJ+itemp2-1, 6, gpu->nbasis);
                        }
                    }
                }
            }
        }
    }
    
    
    gpu -> gpu_basis -> Xcoeff   -> Upload();
    
    gpu -> gpu_basis -> upload_all();

    gpu -> gpu_sim.Xcoeff                       =   gpu -> gpu_basis -> Xcoeff -> _devData;
    gpu -> gpu_sim.ncontract                    =   gpu -> gpu_basis -> ncontract -> _devData;
    gpu -> gpu_sim.first_basis_function         =   gpu -> gpu_basis -> first_basis_function -> _devData;
    gpu -> gpu_sim.last_basis_function          =   gpu -> gpu_basis -> last_basis_function -> _devData;
    gpu -> gpu_sim.first_shell_basis_function   =   gpu -> gpu_basis -> first_shell_basis_function -> _devData;
    gpu -> gpu_sim.last_shell_basis_function    =   gpu -> gpu_basis -> last_shell_basis_function -> _devData;
    gpu -> gpu_sim.ncenter                      =   gpu -> gpu_basis -> ncenter -> _devData;
    gpu -> gpu_sim.kstart                       =   gpu -> gpu_basis -> kstart -> _devData;    
    gpu -> gpu_sim.katom                        =   gpu -> gpu_basis -> katom -> _devData;
    gpu -> gpu_sim.ktype                        =   gpu -> gpu_basis -> ktype -> _devData;
    gpu -> gpu_sim.kprim                        =   gpu -> gpu_basis -> kprim -> _devData;
    gpu -> gpu_sim.kshell                       =   gpu -> gpu_basis -> kshell -> _devData;    
    gpu -> gpu_sim.Ksumtype                     =   gpu -> gpu_basis -> Ksumtype -> _devData;
    gpu -> gpu_sim.Qnumber                      =   gpu -> gpu_basis -> Qnumber -> _devData;
    gpu -> gpu_sim.Qstart                       =   gpu -> gpu_basis -> Qstart -> _devData;
    gpu -> gpu_sim.Qfinal                       =   gpu -> gpu_basis -> Qfinal -> _devData;    
    gpu -> gpu_sim.Qsbasis                      =   gpu -> gpu_basis -> Qsbasis -> _devData;
    gpu -> gpu_sim.Qfbasis                      =   gpu -> gpu_basis -> Qfbasis -> _devData;
    gpu -> gpu_sim.gccoeff                      =   gpu -> gpu_basis -> gccoeff -> _devData;
    gpu -> gpu_sim.cons                         =   gpu -> gpu_basis -> cons -> _devData;
    gpu -> gpu_sim.gcexpo                       =   gpu -> gpu_basis -> gcexpo -> _devData;
    gpu -> gpu_sim.KLMN                         =   gpu -> gpu_basis -> KLMN -> _devData;    

    PRINTDEBUG("COMPLETE UPLOADING BASIS")
}


__global__ void test(QUICKDouble* d);
extern "C" void gpu_get2e_(QUICKDouble* o)
{
    PRINTDEBUG("BEGIN TO RUN GET2E")
    upload_sim_to_constant(gpu);
    
    
    
    PRINTDEBUG("BEGIN TO RUN KERNEL") 
#ifndef TEST
    get2e(gpu);
#else
    for (int i = 1; i<= gpu->gpu_basis->jshell; i++) {
        for (int j = i; j<= gpu->gpu_basis->jshell; j++) {
            for (int k = i; k<= gpu->gpu_basis->jshell; k++) {
                for (int l = k; l<= gpu->gpu_basis->jshell; l++) {
                    gpu_shell(i,j,k,l);
                }
            }
        }
    }
#endif

    PRINTDEBUG("COMPLETE KERNEL")
    gpu -> gpu_calculated -> oULL -> Download();
    
    for (int i = 0; i< gpu->nbasis; i++) {
        for (int j = i; j< gpu->nbasis; j++) {
            QUICKULL valULL = LOC2(gpu->gpu_calculated->oULL->_hostData, j, i, gpu->nbasis, gpu->nbasis);
            QUICKDouble valDB;
            
            if (valULL >= 0x8000000000000000ull) {
                valDB  = -(QUICKDouble)(valULL ^ 0xffffffffffffffffull);
            }
            else
            {
                valDB  = (QUICKDouble) valULL;
            }
            LOC2(gpu->gpu_calculated->o->_hostData,i,j,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
            LOC2(gpu->gpu_calculated->o->_hostData,j,i,gpu->nbasis, gpu->nbasis) = (QUICKDouble)valDB*ONEOVEROSCALE;
        }
    }
    
    gpu -> gpu_calculated -> o    -> Download(o);

    PRINTDEBUG("DELETE TEMP VARIABLES")
	delete gpu->gpu_cutoff->cutMatrix;
	delete gpu->gpu_cutoff->YCutoff;
	delete gpu->gpu_cutoff->cutPrim;
	
	delete gpu->gpu_calculated->o;
//	delete gpu->gpu_calculated->co;
//	delete gpu->gpu_calculated->vec;
	delete gpu->gpu_calculated->dense;
	delete gpu->gpu_calculated->oULL;


    PRINTDEBUG("COMPLETE RUNNING GET2E")
}
